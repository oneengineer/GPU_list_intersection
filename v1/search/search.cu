#include "hip/hip_runtime.h"
#include "../common_defines.h"
#include "search.cuh"
#include <cstdio>


	//extern __device__ void scan_a_block_neat(int * src_data,volatile int shared[][WARP_SIZE*2]);

	template<int loops,bool exclusive>
	inline __device__ void scan_warp_neet(volatile int * shared,int value,int id){
		shared[id] = 0;
		id += WARP_SIZE;
		shared[id] = value;

		if ( 1 <= loops ) shared[id] += shared[id - 1];
		if ( 2 <= loops ) shared[id] += shared[id - 2];
		if ( 3 <= loops ) shared[id] += shared[id - 4];
		if ( 4 <= loops ) shared[id] += shared[id - 8];
		if ( 5 <= loops ) shared[id] += shared[id - 16];

		if ( exclusive )
			shared[id] -= value;
	}

	template< int id_range >
	inline __device__ void scan_a_block_neat(volatile int * src_data,volatile int  shared[][WARP_SIZE*2]){
		int id = threadIdx.x;
		int local_id = id&(WARP_SIZE-1);
		int warp_num = id >> LOG_WARP_SIZE ;
		volatile __shared__ int shared_level2[WARP_SIZE*2];

		scan_warp_neet< LOG_WARP_SIZE,false>(shared[warp_num],src_data[id],local_id );//basic level
		syncthreads();
		if ( id < WARP_SIZE){
			int myvalue;
			if ( id < id_range  )
				myvalue = shared[id][WARP_SIZE+WARP_SIZE-1];
			scan_warp_neet<LOG_WARP_SIZE ,true>(shared_level2,myvalue,id);//second level, use calculated 32,in one block
		}
		syncthreads();
		//uniform update
		int a = shared_level2[ warp_num+WARP_SIZE ];
		src_data[id] = shared[warp_num][ local_id + WARP_SIZE ] + a;
	}

	__global__ void scan_buffer_large(int save_indx){

#if DEF_D1*2/WARP_SIZE < 1
		__shared__ volatile int  shared[ 2 ][WARP_SIZE*2];
#else
		__shared__ volatile int  shared[ DEF_D1*2/WARP_SIZE ][WARP_SIZE*2];
#endif

		scan_a_block_neat<2*DEF_D1/WARP_SIZE>(_scan_buffers[save_indx],shared);
		//printf("scaned: <%d> %d\n",threadIdx.x,_scan_buffers[save_indx][threadIdx.x]);
	}



	/**
	 *
	 * To write the program use minimum command
	 * Three things decide the next position:
	 * 1. position out of boundary?
	 * 2. get value, direction? change or not?
	 * 3. length? how l
	 * L: position = what
	 *
	 *
	 * value: the value to search
	 * pos2: starting place, extrapolated position, it need to be return!
	 * return:result 0/1
	 *
	 */

	inline __device__ int gallop_search (int value,int & pos2,volatile int *list,int left,int right){

		int pos = pos2;
		int result = 0;
		int len;
		int value2;

		char last_direction = 3;
		char direction = 0;
		char start_decrease = 0;

		len = 1;
		while ( len > 0){

			if (!( left <= pos2 && pos2 <= right )){
				start_decrease = 1;
			}
			else{
				value2 = list[pos2] - value;
				if ( 0 == value2 ){
					result = 1;break;
				}
				else direction = value2 < 0;
				start_decrease |= (direction + last_direction) ==1;// 0 + 1 or 1 + 0
				pos = pos2;
				last_direction = direction;
			}
			if ( start_decrease )
				len >>=1;
			else len <<=1;

			if ( direction )
				pos2 = pos + len;
			else pos2 = pos - len;
		}
		return result;
	}

	inline __device__ int binary_search(int value,int & pos,volatile int *list,int left,int right){
		int result = 0;
		while ( left <= right ){
			if ( value == list[ pos ] ){
				//printf("block:%d thread: %d INTERSECTION %d\n",blockIdx.x,threadIdx.x,value);
				result = 1;
				break;
			}
			else if ( list[pos] < value )
				left = pos + 1;
			else right = pos - 1;
			pos = (left + right)/2;
		}
		return result;
	}

	/*
	 * because the search_2 is done in one SM in GPU, no divergence occur
	 * begin and end are assigned from blockIdx, one block only have on value => no divergence
	 *
	 */
	inline __device__ int search_one_value(int value,int & pos,volatile int *opposite_addr,const int &left,const int &right){

		//printf("[<%d,%d>] has value  %d search in %d %d\n",blockIdx.x,threadIdx.x,value,opposite_addr[left],opposite_addr[right]);
		int result = binary_search(value,pos,opposite_addr,left,right);

//
//		if (result){
//			int write_pos = atomicAdd(&gpu_result_size,1);
//			printf("[<%d,%d>] found %d\n",blockIdx.x,threadIdx.x,value);
//			_result[write_pos] = value;
//		}
		return result;
	}

	/*
	 * search 4 integer stored in uint4, a,b,c,d
	 * use divide-and-conquer way a,c,b,d order search
	 *
	 */
	inline __device__ void search_uint4(const int & id,const uint4 &myvalue,volatile int *opposite_list,
			volatile int *shared_range,const struct partition_info &info,
			int &mask,int &uint4_sum){
		int pos_A,pos_X,pos_C,pos_D;
		pos_A = (info.left + info.right)/2;
		//pos_A = id<<2;//check search times
		if (search_one_value(myvalue.x,pos_A,opposite_list,info.left,info.right)){
			mask |=1,uint4_sum ++;
		}
		shared_range[id] = pos_A;
		syncthreads();
		int id_addone = id + 1;
		if ( id_addone != info.len )
			pos_D = shared_range[id_addone];
		else pos_D = info.right;

		pos_C = (pos_A + pos_D) /2 ;
		if (search_one_value(myvalue.z,pos_C,opposite_list,pos_A,pos_D)){
			mask |=4,uint4_sum ++;
		}
		pos_X = (pos_A + pos_C) /2 ;
		if (search_one_value(myvalue.y,pos_X,opposite_list,pos_A,pos_C)){
			mask |=2,uint4_sum ++;
		}
		pos_X = (pos_C + pos_D) /2 ;
		if (search_one_value(myvalue.w,pos_X,opposite_list,pos_C,pos_D)){
			mask |=8,uint4_sum ++;
		}
	}

#if 1
	/*
	 * Simple method, not use shared
	 * */
	inline __device__ void search_uint4_2(const int & id,const uint4 &myvalue,volatile int *opposite_list,
			volatile int *shared_range,const struct partition_info &info,
			int &mask,int &uint4_sum){
		int pos_A,pos_X,pos_D;
		//pos_A = id;
		pos_A = (info.left + info.right)/2;
		mask = 0,uint4_sum=0;
		if (search_one_value(myvalue.x,pos_A,opposite_list,info.left,info.right)){
			mask |=1,uint4_sum ++;
		}
		pos_D =  ( pos_A+ info.right)/2;
		if (search_one_value(myvalue.w,pos_D,opposite_list,pos_A,info.right)){
			mask |=8,uint4_sum ++;
		}
		pos_X = (pos_A + pos_D) /2 ;
		if (search_one_value(myvalue.y,pos_X,opposite_list,pos_A,pos_D)){
			mask |=2,uint4_sum ++;
		}
		if (search_one_value(myvalue.z,pos_X,opposite_list,pos_X,pos_D)){
			mask |=4,uint4_sum ++;
		}
	}
#endif



	inline __device__ void scan_and_save_buffer(volatile int *scan_array,volatile int *scan_shared,int * result_buffer,
			const int & id,uint4 &myvalue,int &mask,int &uint4_sum){
		scan_array[id] = uint4_sum;
		syncthreads();
		scan_a_block_neat<DEF_D2/4/WARP_SIZE>((int *)scan_array,(int (*)[WARP_SIZE*2])scan_shared);
		syncthreads();

		int position = scan_array[id] - uint4_sum; // exclusive scan
		//int position_debug = position;

		if ( mask & 1 )
			result_buffer[position++] = myvalue.x;
		if ( mask & 2 )
			result_buffer[position++] = myvalue.y;
		if ( mask & 4 )
			result_buffer[position++] = myvalue.z;
		if ( mask & 8 )
			result_buffer[position++] = myvalue.w;

		//debug bad save method
//		FOR_I(position_debug,position){
//			//int write_pos = atomicAdd(&gpu_result_size,1);
//			if ( result_buffer[i] == 3836)
//			printf("%d is saved at %d by <%d,%d>\n",result_buffer[i],i,blockIdx.x,threadIdx.x);
//		}



	}

#if DEPLETED

	inline __device__ void brute_force(uint4 & myvalue,volatile int * list,const struct partition_info &info){
		FOR_I(info.left,info.right+1){
			if ( myvalue.x == list[i] ){
				int write_pos = atomicAdd(&gpu_result_size,1);
				_result[write_pos] = myvalue.x;
			}
			if ( myvalue.y == list[i] ){
				int write_pos = atomicAdd(&gpu_result_size,1);
				_result[write_pos] = myvalue.y;
			}
			if ( myvalue.z == list[i] ){
				int write_pos = atomicAdd(&gpu_result_size,1);
				_result[write_pos] = myvalue.z;
			}
			if ( myvalue.w == list[i] ){
				int write_pos = atomicAdd(&gpu_result_size,1);
				_result[write_pos] = myvalue.w;
			}
		}
	}
#endif

	inline __device__ void search_in_block(int * V,int search_now,const struct partition_info &info){
		int id = threadIdx.x;
		// begin and end is closed interval [begin , end]

		volatile __shared__ int opposite_list[DEF_D2];//scan use 2/4 times space
		volatile __shared__ int shared_range[DEF_D2/4];// only use once, the right most thread does not use it
		int mask,uint4_sum;
		uint4 myvalue ;
		bool flag1 = false;
		/*
		 * shared_range __shared__ use as src_data for scan
		 * opposite_list __shared__ use as
		 *  */

		// the thread which should copy opposited list elements
		if (id < info.len_opposite){

			myvalue = ((uint4 *)info.opposite_addr)[id];
			((uint4 *)opposite_list)[id] = myvalue;
			if ( id < info.len ){
				myvalue = ((uint4 *)info.addr)[id];
				flag1 = true;

			}
		}
		if ( id < info.warp_len ){
			mask= 0;uint4_sum=0;
			syncthreads();
			if (flag1){
				search_uint4(id,myvalue,opposite_list,shared_range,info,mask,uint4_sum);
			}
			syncthreads();
			scan_and_save_buffer(shared_range,opposite_list,V+DEF_D2*blockIdx.x,id,myvalue,mask,uint4_sum);
			if ( info.warp_len-1 ==id ){
				//printf("<%d %d>'s last  is %d\n",blockIdx.x,threadIdx.x,shared_range[id]);
				_scan_buffers[search_now][blockIdx.x] = shared_range[id];//scan_array
			}
		}
	}

	__global__ void algo2_search(int * V,int search_now,int offset){
		if (partitions_info[search_now][blockIdx.x].len>0){
			search_in_block(V,search_now,partitions_info[search_now][ blockIdx.x ]);
		}
	}



