#include "hip/hip_runtime.h"
/*New feature:
	pre calculate indices, smart adaptive, fully pipeline, not fully hidden
	cudpp have streams, but number of device block/threads can not be controlled
	cudpp prefix sum for my algo
	pipeline ( cudpp pipelined) is employed
	pipeline ( myscan pipelined) is employed
*/

#include <iostream>
#include <fstream>
#include <algorithm>
#include <string>
#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#include <unistd.h>
#include <sys/time.h>

//------ for CUDA ------
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "common_defines.h"
#include "common_functions.h"
#include "others/bsearch.cuh"
#include "others/memory_data.cuh"
#include "indices/cal_indices.cuh"
#include "search/search.cuh"
#include "save/scan.cuh"
#include "save/save_result.h"

#include "others/generate_data.h"

//#include "cudpp.h"


using namespace std;

	template <typename T>
	void debug_a(T * data,int begin,int end){
		for (int i=begin;i<end;i++) cout<<"["<<i<<"]: "<<data[i]<<"\t";cout<<endl;
	}
	template <typename T>
	void debug_a(T * data,int end){
		debug_a(data,0,end);
	}
	template <typename T>
	void debug_a2(T * data,int end1,int end2){
		for (int i=0;i<end1;i++){cout<<"row "<<i<<endl; for (int j=0;j<end2;j++) cout<<"["<<i<<","<<j<<"] "<<data[i][j]<<"\t";cout<<endl;}
	}
	double get_sec(const struct timeval & tval){
		return ((double)(tval.tv_sec*1000*1000 + tval.tv_usec))/1000000.0;
	}

	template <typename T>
	T checkmin(T & data,T value){
		data = min(data,value);
		return data;
	}
struct Watch{
	timeval begin,end;
	void start(){gettimeofday(&begin,NULL);}
	double time(){return get_sec(end)-get_sec(begin);}
	double stop(){gettimeofday(&end,NULL);return time();}
};

	inline double rand_double(double range){
		return  ((double)rand()/(double)RAND_MAX)*range;
	}
//----------- cuda template ------------

struct CudaWatch{
	hipEvent_t start_event, stop_event;
	void start(){
	hipEventCreate(&start_event) ;
	hipEventCreate(&stop_event) ;
	hipEventRecord(start_event,0);
	}
	float stop(){
		float time_cost = -1.0;
		hipEventRecord(stop_event,0);
		hipEventSynchronize(stop_event);
		hipEventElapsedTime(&time_cost,start_event,stop_event);
		return time_cost;
	}
};
	void cu_checkError(){
		hipError_t error = hipGetLastError();
		if (error != hipSuccess){
			printf("ERROR: %s\n",hipGetErrorString(error));
			exit(-1);
		}
	}

	__global__ void cu_print(int *list,int size){
		printf("CUDA SHOW:\n");
		if ( 0 == threadIdx.x && 0 == blockIdx.x ){
			FOR_I(0,size)
				printf("[%d]:%d\t",i,list[i]);
			printf("\n");
		}
	}

	void cu_host_print(int *list,int size){
		hipDeviceSynchronize();
		cu_print<<<1,1>>>(list,size);
		hipDeviceSynchronize();
	}


//-------------------WRITE YOUR PROGRAM FROM HERE



int *host_lists[2];

int *devL1,*devL2;
bool *v;
int *devV[3],*devResult;
int * devIndices;
int n,m;

int Lresult;
int *resultList;
int *cpuResult;

__device__ int *list_p0[2];// the origional   address
__device__ int *list_p[QUEUE_SIZE][2]; // save the starting position of either list
__device__ int calculated_indices_len[QUEUE_SIZE][4];
__device__ int swapped[QUEUE_SIZE];   // save swapped stage for each status
__device__ int *_result;
__device__ int gpu_result_size;
__device__ int _nm[2];
__device__ int partitions[QUEUE_SIZE][512+4][2];
__device__ partition_info partitions_info[QUEUE_SIZE][512+4];

__device__ debug_structure debug1;

	inline void move_pos(int &pos){
		pos = (pos + 1) % QUEUE_SIZE;
	}

//	CUDPPHandle prepare_prefixsum(int size, hipStream_t * stream){
//		CUDPPConfiguration config;
//		config.op = CUDPP_ADD;
//		config.datatype = CUDPP_INT;
//		config.algorithm = CUDPP_SCAN;
//		config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;
//		config.mystream = stream;
//
//		CUDPPHandle theCudpp;
//		cudppCreate(&theCudpp);
//		CUDPPHandle scanplan = 0;
//		CUDPPResult res = cudppPlan(theCudpp,&scanplan,config,size,1,0);
//		if ( CUDPP_SUCCESS != res ){
//			printf("ERROR in prepare_prefixsum\n");
//			exit(-1);
//		}
//		return scanplan;
//	}

	__global__ void save_one_core(int search_now,int *devV){
		int num = 0;
		int len = calculated_indices_len[search_now][0];
		FOR_I(0,len+1)
			if ( devV[i] ){
				_result[ num++ ] = list_p[search_now][0][i];
				//printf("in save : saved :%d\n",list_p[search_now][0][i]);//debug
			}
		_result += num;
	}

	__global__ void help_show_partation(int indices_now,int num_part,bool printout=false){
		FOR_I(0,num_part){
			struct partition_info *info = &partitions_info[indices_now][i];
			printf("Partation: %d\n",i);
			printf("[%llx]: %d --- [%llx]:%d\n",info->addr,*(info->addr),info->opposite_addr,*(info->opposite_addr));
			printf("[%lld]: --- [%lld]:\n",info->addr - list_p0[ info->B2A ],info->opposite_addr - list_p0[!info->B2A]);
			if (printout){
				FOR_I(0,info->len*4) printf("%d \t",info->addr[i]);
				printf("\n |||||||| \n");
				int xx= info->right + (3- info->right%4)%4 +1;
				FOR_I(0, xx) printf("%d \t",info->opposite_addr[i]);
				//FOR_I(info->left,info->right+1) printf("%d \t",info->opposite_addr[i]);
				printf("\n");
			}
			printf("len:%d \t left:%d\t right:%d \t B2A:%d\n",info->len,info->left,info->right,info->B2A);
		}
	}

	__global__ void help_show_partation2(int indices_now,int num_part){
		FOR_I(0,num_part){
			struct partition_info *info = &partitions_info[indices_now][i];
			printf("Partation: %d\n",i);
			printf("len:%d \t left:%d\t right:%d \t B2A:%d\n",info->len,info->left,info->right,info->B2A);
			long long offset1,offset2;
			if (info->B2A){
				offset1 = info->opposite_addr - list_p0[0];
				offset2 = info->addr - list_p0[1];
			}
			else{
				offset1 = info->addr - list_p0[0];
				offset2 = info->opposite_addr - list_p0[1];
			}
			printf("offset : < %lld --- %lld >",offset1,offset2);
			printf("\n---------------------\n");
		}
	}

	__global__ void back__result(){
		_result += gpu_result_size;
	}

	__global__ void show_addr_value(int *devL,int n){
		FOR_I(0,n){
			printf("[%llx]: %d \t",devL+i,devL[i]);
		}
		printf("\n");
	}

	__global__ void help_debug(int loop){
		debug1.num_loop = loop;
	}

	void work(){

		int numStream;
		numStream = 4;
		hipStream_t *streams = (hipStream_t *) malloc(numStream * sizeof(hipStream_t));
		for (int i = 0; i < numStream; i++)
		{
			hipStreamCreate(&(streams[i]));
		}

		int D1,D2,D1save;
		int block_size;
		int block_2_size;
		//------ some settings ----------
		D1save = 16;
		D1 = 128;D2 = 8*32*4;

		//D1 = 16;D2 = 64;
		//D1 = 8;D2 = 64*4;//D2 has to be the multiply of WARP_SIZE, because

		int save_stream = 2;
		int search_stream = 1;

		block_size = D1 * D2;
		block_2_size = block_size;
		//block_2_size = block_size*1000;

		init_data(block_size);
		init_device_variables();
		init_scan(&streams[save_stream],1024);

		int num_small_block = D1;
		dim3 cal_indx_setting(num_small_block,2);

		CudaWatch cudawatch;
		Watch cpuWatch;cpuWatch.start();
		cudawatch.start();
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(algo2_search),hipFuncCachePreferShared);
		cu_checkError();

		//CUDPPHandle prefixsum_plan = prepare_prefixsum(block_size,streams+2);
		//show_addr_value<<<1,1>>>(devL1,n);
		//show_addr_value<<<1,1>>>(devL2,m);

		int len1,len2;
		int cal_pos = 0, search_pos = 0, save_pos  =0; // they are for L1, L2 position
		while (true){
			int devVinc = 0; // use which devV to store data
			back_next_relative_len(len1,len2,cal_pos);
			//cu_checkError();
			outln(len1);outln(len2);//debug
			if ( len1<=0 || len2 <= 0 ) break;
			int loops = min(len1,len2)/block_size;
			if ( min(len1,len2) % block_size != 0 )
				loops ++;

			if ( loops >0 ){
				outln(loops);
				int *saveV; //pointer of saving result

				//-- stage middle
				for ( ;loops> 0 ;loops -- ){
					help_debug<<<1,1>>>(loops);
					outline;outline;outln(loops);
					saveV = devV[ devVinc];
					cal_indx<<<1,cal_indx_setting>>>( block_size,block_2_size,cal_pos);
					//hipDeviceSynchronize();
					if (1 == loops){
						//help_show_partation2<<<1,1>>>(cal_pos,128);//debug
						//return;
					}
					//hipDeviceSynchronize();
					algo2_search<<< 2 * D1 , D2/4 >>>(saveV,search_pos,0);

					move_pos(cal_pos);move_pos(search_pos);move_pos(save_pos);

					//mssleep(10);
					hipDeviceSynchronize();
					//back__result<<<1,1>>>();return;
				}
				//break;//debug
				//hipDeviceSynchronize();
			}

			cu_checkError();
		}
		hipDeviceSynchronize();
		back__result<<<1,1>>>();
		cout<<"MY Algo:"<<cudawatch.stop()/1000.0<<endl;
		cout<<"MY Algo cpu test: "<<cpuWatch.stop()<<endl;
	}

int cpuResultSize = 0;
	int merge_algo(int *array1,int *array2, int begin1,int end1,int begin2,int end2){
		cpuResultSize = 0;
		int i=begin1,j=begin2;
		int lasti,lastj;

		lasti=array1[i];
		lastj=array2[j];
		while ( i<end1 && j<end2){
			if ( lasti == lastj ){
				cpuResult[cpuResultSize++] = lasti;
				i++;j++;
				lasti=array1[i];
				lastj=array2[j];
			}
			else if ( lasti < lastj){
				i++;
				lasti = array1[i];
			}
			else {
				j++;
				lastj = array2[j];
			}
		}
		return cpuResultSize;
	}

	int check_correctness(int cpuResultSize){
			FOR_I(0,cpuResultSize)
				if (cpuResult[i] != resultList[i]){
					printf("Wrong at [%d] R:%d Yours:%d\n",i,cpuResult[i],resultList[i]);
					debug_a( cpuResult+i,-10,10 );
					debug_a( resultList+i,-10,10 );
					return i;
				}
		return -1;
	}

	void prepare_data(int n){
		host_lists[0] = new int [n];
		resultList = new int[n];
		host_lists[1] = new int [n];
		v = new bool [10*n];
		resultList = new int[n];
		cpuResult = new int [n];
	}

#if 0

	extern int * scan_buffer;

	__global__ void set_somethings(int *L,int n){
		calculated_indices_len[0][0] = n;
	}

	void test_scan_save(){
		n = 1024*128;
		generate_random(1.0,2.0,2.0);
		init_data(1024*256);
		init_device_variables();
		int *temp = new int [n];
		int *tempR = new int [n];
		FOR_I(0,n) temp[i] = (rand() % 50 )== 1;
		hipMemcpy(devV[0],temp,sizeof(int)*n,H_T_D);
		int num = 0;
		int count = 0;

		FOR_I(0,n) if ( temp[i]) tempR[num++] = host_lists[0][i];

		set_somethings<<<1,1>>>(devV[0],n);
		//cu_host_print(devV[0],n);
		hipStream_t stream;
		hipStreamCreate(&stream);
		init_scan(&stream,1024);
		ScanSequence seq(1,devV[0],n);
		seq.save_indices_now = 0;
		seq.init();
		seq.run_all();

//		seq.run_scan(0);
//		seq.run_large();
//		seq.run_save(0);

		seq.run_saveLarge();
		hipMemcpy(resultList,devResult,sizeof(int)*num,D_T_H);


		FOR_I(0,num){
			if ( tempR[i] != resultList[i] )
				cout<<"ERROR AT "<<i<<" "<<tempR[i]<<" "<<resultList[i]<<endl;
		}
//		debug_a(tempR,num);
//		debug_a(resultList,num);

		//cu_host_print(devResult,n);
		hipDeviceSynchronize();
		cout<<"Test over "<<endl;
	}



	__global__ void help_test_cal_indices(){
		FOR_I(0,16){
			printf("i: %d (%d %d)\n",i,partitions[0][i][0],partitions[0][i][1]);
		}
	}

	__global__ void show_part(){
		FOR_I(0,16)
			printf("%d  [%d] -- [%d]\n",i,partitions[0][i][0],partitions[0][i][1]);

	}

	void test_cal_indices(){
		n = 4 * 128;
		generate_random(1.2,1.5,1.5);
		//FOR_I(0,n) host_lists[0][i] =2*i ,host_lists[1][i] = i;
		init_data(128);
		init_device_variables();

		int block_size = 128;
//		debug_a(host_lists[0],block_size+10);
//		debug_a(host_lists[1],block_size+10);


		dim3 d(8,2);
		cal_indx<<<1,d>>>(block_size,block_size,0);
		cu_checkError();
		//help_test_cal_indices<<<1,1>>>();
		cu_checkError();
		hipDeviceSynchronize();

		algo2_search<<<16,16>>>(devV[0],0,0);
		cu_checkError();
		hipDeviceSynchronize();

		cpuResultSize = 0;
		merge_algo(host_lists[0],host_lists[1],0,128,0,128);
//		cout<<"Cpu result"<<endl;debug_a(cpuResult,cpuResultSize);//debug
		resultList = new int [200];

		hipMemcpy(resultList,devV[0],sizeof(int)*128,D_T_H);
		int num_dev = 0;
		FOR_I(0,128)
			if (resultList[i]){
//				printf("[%d] %d\t",i,host_lists[0][i]);
				resultList[num_dev++] = host_lists[0][i];
			}printf("\n");

		cout<<"Check Correctness"<<endl;

		if (num_dev != cpuResultSize) { printf("Wrong num!");exit(1);}

		FOR_I(0,cpuResultSize)
			if (  resultList[i] != cpuResult[i] ){
				printf("Wrong at [%d] %d %d",i,resultList[i],cpuResult[i]);
				exit(1);
			}
		cout<<"It is correct"<<endl;
	}

#endif


	__global__ void show_debug_1(){
		printf("wrong 1 times: %d\n",debug1.wrong_1);
		printf("wrong 2 times: %d\n",debug1.wrong_2);
	}

int main(int arg_num,char ** args){
	//hipDeviceReset();

	prepare_data(1024*1024*90);

	FOR_I(155,10000){
		srand(i);
		FOR_J(0,arg_num){
			char ch = args[j][0];
			if ( '0'<= ch && ch<='9'){
				int seed;
				sscanf(args[j],"%d",&seed);
				srand(seed);
				printf("SEED: %d\n",seed);
			}
		}
		//srand(710852);
	n = 1024*1024*40;
	//n = 1024*1024*10;
	//n = 1024*1024;
	//n = 1024*102;
	//n = 5000;
	//n = 660;
	//n = 66;
	//n = 20;

	//generate_case5();
	//generate_random(1.0,2.0,2.0);
	//n = m = 70;
	//host_lists[1][16] = 65;
	//generate_same(2.0);
	generate_shift(2.0,1);

	cout<<"generate data over srand("<<i<<") n="<<n<<" m="<<m<<endl;
	//debug_a(host_lists[0],n);debug_a(host_lists[1],m);//debug
	printf("List 1 ( %d --- %d --- %d )\n",host_lists[0][0],host_lists[0][n/2],host_lists[0][n-1]);
	printf("List 2 ( %d --- %d --- %d )\n",host_lists[1][0],host_lists[1][m/2],host_lists[1][m-1]);

	int sum1=0,sum2=0;
	FOR_K(0,n) if (k%4 ==3) { sum1 += host_lists[0][k],sum2 += host_lists[1][k]; }

	Watch watch;watch.start();
	int cpuResultSize = merge_algo(host_lists[0],host_lists[1],0,n,0,m);
	cout<<"CPU ALGO time: "<<watch.stop()<<endl;

//	work3();//test bsearch
//	return 0;


	work();
	show_debug_1<<<1,1>>>();
	outln(sum1);outln(sum2);
	//cuda_copyResult();cout<<"copied back"<<endl;
	cu_checkError();
	free_device_memory();
	printf(" results / elements = %d / %d %lf\n",Lresult,n,(1.0*Lresult)/(1.0*n));

	sort(resultList,resultList + Lresult);
	outln(Lresult);outln(cpuResultSize);
	//debug_a(resultList,Lresult);
	//debug_a(cpuResult,cpuResultSize);
	//break;

	//continue;// DO NOT CHECK correctness

	int error_index;
	if  ( (error_index = check_correctness(cpuResultSize)) >=0 ){
		outline;
		outln(i);
		printf("all length: cpu:%d gpu:%d  ration: %.5lf\n",cpuResultSize,Lresult,100.0*error_index/(1.0*Lresult));
		break;
	}
		outline;
		outline;
		break; //only execute once
	}//end FOR
	return 0;
}

