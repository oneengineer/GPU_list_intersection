#include "hip/hip_runtime.h"


#include "cal_indices.cuh"
#include "../common_defines.h"
#include <cstdio>

	__device__ int next_queue_pos(int value){
			return (value+1) % QUEUE_SIZE;
	}

	// config should be <1,4>
	__device__ void cal_indx_1(int block_size,int block_2_size,int indices_now){
		int id = threadIdx.x;
		int group_id = id >> 1;
		int sub_id = id & 1;
		int opposite_sub_id = sub_id ^ 1; //!sub_id

		int indices_next = (indices_now + 1) % QUEUE_SIZE;
		int temp_blocks[] = {block_size, block_2_size};

		int *myList = list_p[indices_now][ group_id ^ sub_id ];
		int *oppositeList = list_p[indices_now][ group_id ^ opposite_sub_id ];
		int myLen = temp_blocks[sub_id];
		int oppositeLen = temp_blocks[ opposite_sub_id ];
		int value = myList[ myLen - 1 ];
		int left = 0, right = oppositeLen -1;

		while ( left < right ){
			int mid = (left + right + 1)/2;
			if ( value < oppositeList[mid] )
				right = mid - 1;
			else left = mid;
		}

		//printf("id:%d value %d get to %d\n",id,value,oppositeList[left]);//debug

		int next_opposite_offset = left + ( oppositeList[left] <= value );

		int *save_indices0 = calculated_indices_len[indices_now];
		int *save_indices = save_indices0 + (group_id << 1);
		int *opposite_indices = save_indices0 + ( (group_id^1) << 1);
		save_indices[sub_id] = next_opposite_offset ;

		int mysum = save_indices[ sub_id ] + save_indices[ opposite_sub_id ];
		int oppositeSum = opposite_indices[ sub_id ] + opposite_indices[ opposite_sub_id ];

		//decide opposite list offset and length
		if ( mysum + group_id > oppositeSum){

			//printf("cal TURN: id %d len: %d  \n",id,next_opposite_offset);//debug
			save_indices0[ opposite_sub_id ] = next_opposite_offset - 1;
			list_p[indices_now][sub_id] = myList;

			list_p[indices_next][ opposite_sub_id ] = oppositeList + next_opposite_offset;//set opposite pointer
			swapped[ indices_next ] = swapped[ indices_now ] ^ group_id;
		}
	}

	__device__ void bitonic_merge(bool up, volatile int * data,int id, int n){
		if ( n <1 ) return ;
		int n2 = n >> 1;

		syncthreads();
		if  ((data[id] > data[id + n]) ^ up ){
			int t = data[id];
			data[id] = data[id+n];
			data[id+n] = t;
		}
		// change part of the thread
		if ( id >= n2 )
			id -= n2,data += n;
		bitonic_merge(up, data,id, n2);
	}

	//config should be dim3 ths(16,2)

#define ALIGN_MUL 4 // 4 int
#define ALIGN_ADDR_MUL (4*ALIGN_MUL-1) // int = 4 byte
#define ALIGN_MOD (ALIGN_MUL-1)


	inline __device__ int complement(int value,int mod){
		return (mod - ( value &mod) )& mod;
	}

	/*
	 * begin ,end : the data range. may larger than the calculation needs
	 * left, right: the search range. or the real data range
	 */
	inline __device__ void make_memory_alignment(int *addr1,int *addr2
									,int &begin1, int &begin2
									,int &end1, int &end2
									,int &left1, int &left2
									,int &right1, int &right2
									,int &len1, int &len2){
//		int offset1 = ((long long)(addr1) & ALIGN_ADDR_MUL) / 4;
//		int offset2 = ((long long)(addr2) & ALIGN_ADDR_MUL) / 4;
//
//		printf("vof1: %d vof2: %d\n",*addr1,*addr2);
//		printf("[ %llx  %llx ] of1: %d of2: %d\n",addr1,addr2,offset1,offset2);



		left1 = begin1 & ALIGN_MOD;
		left2 = begin2 & ALIGN_MOD; // === %4 for the memory alignment
		right1 = complement(end1,ALIGN_MOD);
		right2 = complement(end2,ALIGN_MOD);
		begin1 -= left1,begin2 -=left2;
		end1 += right1,end2 += right2; // === %4 for the memory alignment

		len1 = end1 - begin1,len2 = end2 - begin2;
		right1 = len1 - right1,right2 = len2 - right2;

//		if ( (len1+1) % 4!=0 || (len2+1) %4 !=0 ) printf("!!wrong\n");
//		if ( begin1 < 0 || begin2 < 0) printf("!!very wrong\n");

	}

	__device__ bool monotone_check(int *list,int n){
		FOR_I(1,n)
				if ( !(list[i-1] <= list[i]) )
					return false;
		return true;
	}

	__device__ void cal_indx_2(int parts,int part_size,int block_size,int indices_now){
		int id = threadIdx.x;
		int myside = threadIdx.y;
		int opposite_side = !myside;
		int idx = (id+1)*part_size-1;
		int opposite_idx;
		int *myList = list_p[indices_now][ myside ];
		int *oppositeList = list_p[indices_now][ opposite_side ];

		int myValue = myList[ idx ];

		//------ bsearch upper bound part----------
		int left = 0,right = block_size;
		//TODO the logical might be wrong
		while ( left < right ){
			int mid = (left + right + 1)/2;
			if ( myValue < oppositeList[mid] )
				right = mid - 1;
			else left = mid;
		}


		//------ END bsearch upper bound part----------
		opposite_idx = left + ( oppositeList[left] <= myValue ) - 1;

		__shared__ volatile int shared[2][2*DEF_D1+1];
		shared[myside][ parts - id - 1 ] = idx; //reverse save
		shared[opposite_side][ parts + id ] = opposite_idx;

		syncthreads();
		bitonic_merge(false,shared[myside],id,parts);
		syncthreads();

//		if (!id && !monotone_check( (int *)(shared[myside]),parts*2))
//			printf("WRONG!!");

		//debug print out
//		if ( 12 == debug1.num_loop )
//		if ( !myside && !id ){
//			printf("[%d]:%d ==== [%d]:%d\n",0,myList[0],0,oppositeList[0]);
//			FOR_I(0,parts*2){
//				int a = shared[0][ i ];
//				int b = shared[1][ i ];
//				printf("[%d]:%d --- [%d]:%d\n",a,myList[a],b,oppositeList[b]);
//			}
//		}

		syncthreads();
		int whole_id = blockDim.x * myside + id;
		__shared__ volatile int shared_decide_next_addr[2*DEF_D1+1];//default may be not 0!!
		shared_decide_next_addr[whole_id] = max( shared[0][whole_id], shared[1][whole_id] ) < block_size;
		syncthreads();
		int indices_next = (indices_now + 1) % QUEUE_SIZE;

		//decide partitions address, which is list 1 and which is list 2
		int begin1,end1,begin2,end2,len1,len2;
		if ( whole_id == 0 ){
			begin1 = calculated_indices_len[indices_now][0];
			begin2 = calculated_indices_len[indices_now][1];
			//begin1 = 0;
			//begin2 = 0;
		}
		else {
			begin1 = shared[0][whole_id-1]+1;
			begin2 = shared[1][whole_id-1]+1;
		}
		int left1,left2,right1,right2;
		end1 = shared[0][whole_id], end2 = shared[1][whole_id];

		make_memory_alignment(list_p[indices_now][ 0 ],list_p[ indices_now ][ 1 ]
							,begin1, begin2,end1 ,end2, left1, left2,right1, right2,len1, len2);


		if ( 0 == shared_decide_next_addr[whole_id] ){
			len1 = len2 = -999;//do not do calculation
		}
		struct partition_info *info = & partitions_info[indices_now][whole_id];
		if ( len1 <= len2 ){
			// A ---> B
			info->B2A = false;
			info->addr = list_p[indices_now][0]+begin1;
			info->opposite_addr = list_p[indices_now][1]+begin2;
			info->left = left2;
			info->right = right2;
			info->len = (len1+1)>>2;
			info->len_opposite = (len2+1)>>2;
		}
		else{
			// B ---> A
			info->B2A = true;
			info->addr = list_p[indices_now][1]+begin2;
			info->opposite_addr = list_p[indices_now][0]+begin1;
			info->left = left1;
			info->right = right1;
			info->len = (len2+1)>>2;
			info->len_opposite = (len1+1)>>2;
		}

		info->warp_len = info->len - info->len % WARP_SIZE;
		_scan_buffers[indices_now][whole_id] = 0;

		if ( info->len % WARP_SIZE ){
			info->warp_len += WARP_SIZE;
		}

		syncthreads();
		if (( whole_id == 2*parts - 1 && shared_decide_next_addr[ whole_id ] == 1 ) ||
			( shared_decide_next_addr[ whole_id ] && !shared_decide_next_addr[whole_id+1] ) ){
			int begin_new1 = (shared[0][whole_id]+1 )& ALIGN_MOD;
			int begin_new2 = (shared[1][whole_id]+1 )& ALIGN_MOD;
			swapped[ indices_next ] = 0;
			swapped[ indices_now ] = 0;

			list_p[indices_next][ 0 ] = list_p[indices_now][ 0 ] + shared[0][whole_id]+1 - begin_new1;
			list_p[indices_next][ 1 ] = list_p[indices_now][ 1 ] + shared[1][whole_id]+1 - begin_new2;
			calculated_indices_len[indices_next][0] = begin_new1; //changed meaning
			calculated_indices_len[indices_next][1] = begin_new2; //changed meaning

			//printf("Next real start: [%d]:%d [%d]:%d\n",begin_new1,list_p[indices_next][0][begin_new1],begin_new2,list_p[indices_next][1][begin_new2]);
		}

	}

	__device__ void cal_indx_3(int block_size,int indices_now){
		int id = threadIdx.x;
		int side = threadIdx.y;
		int n = blockDim.x;
		int part_size = block_size / n;
		int x,y;
		int left = 0,right;
		int *myList = list_p[indices_now][ 0];
		int *oppositeList = list_p[indices_now][ 1];

		_scan_buffers[indices_now][threadIdx.x+threadIdx.y*blockDim.x] = 0;

		if ( 0 == id && 1 == side ) return;

		if ( 1 == side){
			x = block_size-1;
			y = id * part_size ;
			right = (n-id)*part_size;
		}
		else {
			x = (id+1) * part_size-1;
			y = 0;
			right = (id+1)*part_size;
		}

		//printf("<%d %d> x:%d y:%d  value1:%d,%d\n",side,id,x,y,myList[x],oppositeList[y]);

		int x2,y2;
		while (left<right){
			int mid = (left + right + 1)/2;
			x2 =  x - mid;//hash back
			y2 =  y + mid;
			if (  myList[x2] < oppositeList[y2] )
				right = mid - 1;
			else left = mid;
		}

		x2 =  x - left;//hash back
		y2 =  y + left;

		//printf("<%d %d> found %d <%d,%d> (%d,%d)\n",side,id,left,x2,y2,myList[x2] ,oppositeList[y2] );

		if ( x2 >=0  ){
			if (myList[x2] > oppositeList[y2] ){
				x2 --;
			}
//			else if ((myList[x2] < oppositeList[y2] ))
//				y2 --;
		}

		//printf("<%d %d> found %d <%d,%d> (%d,%d)\n",side,id,left,x2,y2,myList[x2] ,oppositeList[y2] );
		__shared__ volatile int shared[2][2*DEF_D1+1];
		int whole_id = id;
		if ( side == 1 )
			whole_id = id-1 + blockDim.x;
		shared[0][whole_id+1] = x2;
		shared[1][whole_id+1] = y2;
		if ( 0 == whole_id &&  0 == blockIdx.x)
			shared[0][0] = -1,shared[1][0] = -1;
		syncthreads();

//		if ( IS_FIRST){
//			printf("[%d]:%d ==== [%d]:%d\n",0,myList[0],0,oppositeList[0]);
//			FOR_I(1,part_size){
//				int a = shared[0][ i ];
//				int b = shared[1][ i ];
//				printf("[%d]:%d --- [%d]:%d\n",a,myList[a],b,oppositeList[b]);
//			}
//		}

		__shared__ volatile int shared_decide_next_addr[2*DEF_D1+1];//default may be not 0!!

		if ( shared[0][whole_id+1] == shared[0][whole_id] ){
			shared_decide_next_addr[whole_id] = -1;//vertical line
		}
		else if ( shared[1][whole_id+1] == shared[1][whole_id] ){
			shared_decide_next_addr[whole_id] = 1; //line
		}
		else shared_decide_next_addr[whole_id] = 0; //OK
		int indices_next = (indices_now + 1) % QUEUE_SIZE;


		int begin1,end1,begin2,end2,len1,len2;
		if ( whole_id == 0 ){

			begin1 = calculated_indices_len[indices_now][0];
			begin2 = calculated_indices_len[indices_now][1];
			//begin1 = 0;
			//begin2 = 0;
		}
		else {
			begin1 = shared[0][whole_id]+1;
			begin2 = shared[1][whole_id]+1;
		}
		int left1,left2,right1,right2;
		end1 = shared[0][whole_id+1], end2 = shared[1][whole_id+1];

//		printf("wid: wholeID %d <%d %d> begin [%d]%d <---> [%d]%d\n",whole_id,threadIdx.y,threadIdx.x,begin1,myList[begin1],begin2,oppositeList[begin2]);
//		printf("wid: wholeID %d <%d %d> end [%d]%d <---> [%d]%d\n",whole_id,threadIdx.y,threadIdx.x,end1,myList[end1],end2,oppositeList[end2]);

		len1 = end1 - begin1+1;
		len2 = end2 - begin2+1;

		make_memory_alignment(list_p[indices_now][ 0 ],list_p[ indices_now ][ 1 ],begin1, begin2,end1 ,end2, left1, left2,right1, right2,len1, len2);

		if ( 0 != shared_decide_next_addr[whole_id] ){
			len1 = len2 = -999;//do not do calculation
		}
		struct partition_info *info = & partitions_info[indices_now][whole_id];
		if ( len1 <= len2 ){
			// A ---> B
			info->B2A = false;
			info->addr = list_p[indices_now][0]+begin1;
			info->opposite_addr = list_p[indices_now][1]+begin2;
			info->left = left2;
			info->right = right2;
			info->len = (len1+1)>>2;
			info->len_opposite = (len2+1)>>2;

			//info->len = len1;

		}
		else{
			// B ---> A
			info->B2A = true;
			info->addr = list_p[indices_now][1]+begin2;
			info->opposite_addr = list_p[indices_now][0]+begin1;
			info->left = left1;
			info->right = right1;
			info->len = (len2+1)>>2;
			info->len_opposite = (len1+1)>>2;


			//info->len = len2;

		}

		info->warp_len = info->len - info->len % WARP_SIZE;


		if ( info->len % WARP_SIZE ){
			info->warp_len += WARP_SIZE;
		}


		__shared__ int ending;
		if ( whole_id == 0 ){
			int start = shared_decide_next_addr[blockDim.x*2-2];
			if ( 0 == start ){
				ending = blockDim.x*2-3;
			}
			else {
				for (int i = blockDim.x*2-3;i>=0;i--)
					if ( start != shared_decide_next_addr[i] ){
						ending = i;
						break;
					}
			}
		}

		syncthreads();
		if ( ending+1 == whole_id ){

			int begin_new1 = (shared[0][whole_id]+1 )& ALIGN_MOD;
			int begin_new2 = (shared[1][whole_id]+1 )& ALIGN_MOD;
			swapped[ indices_next ] = 0;
			swapped[ indices_now ] = 0;

			list_p[indices_next][ 0 ] = list_p[indices_now][ 0 ] + shared[0][whole_id]+1 - begin_new1;
			list_p[indices_next][ 1 ] = list_p[indices_now][ 1 ] + shared[1][whole_id]+1 - begin_new2;
			calculated_indices_len[indices_next][0] = begin_new1; //changed meaning
			calculated_indices_len[indices_next][1] = begin_new2; //changed meaning
			//printf("movement: %d,%d\n",shared[0][whole_id]+1,shared[1][whole_id]+1);

			//printf("Next real start: [%d]:%d [%d]:%d\n",begin_new1,list_p[indices_next][0][begin_new1],begin_new2,list_p[indices_next][1][begin_new2]);
		}
//		if ( whole_id > ending +1 )
//			info->len = -999;


	}

	__global__ void cal_indx (int block_size,int block_2_size,int indices_now){
		//cal_indx_1(block_size,block_2_size,indices_now);
		int n = blockDim.x;
		cal_indx_2(n,block_size/n,block_size,indices_now);
		//cal_indx_3(block_size,indices_now);
	}



	//special case for cal_indices
	__global__ void move_indices(int len1,int len2,int block_size,int indices_now){
		int indices_next = (indices_now + 1) % QUEUE_SIZE;
		int swapflag = 0;
		//make sure that len1 is always the shorter one
		if ( len1 > len2 ){
			int *temp = list_p[indices_now][0];
			list_p[indices_now][0] = list_p[indices_now][1];
			list_p[indices_now][1] = temp;
			int temp2 = len1;
			len1 = len2;
			len2 = temp2;
			swapflag = 1;
		}
		int b1 = min(len1,block_size);
		list_p[ indices_next ][ 0 ] = list_p[ indices_now ][ 0 ] + b1;
		list_p[ indices_next ][ 1 ] = list_p[ indices_now ][ 1 ] ;//keep the same
		swapped[ indices_next ] = swapped[ indices_now ] ^ swapflag;
		calculated_indices_len[ indices_now ][ 0 ] = b1-1;// or b1
		calculated_indices_len[ indices_now ][ 1 ] = len2-1;// or b1
	}

