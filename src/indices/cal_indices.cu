#include "hip/hip_runtime.h"


#include "cal_indices.cuh"
#include "../common_defines.h"
#include <cstdio>

	__device__ int next_queue_pos(int value){
			return (value+1) % QUEUE_SIZE;
	}

	// config should be <1,4>
	__device__ void cal_indx_1(int block_size,int block_2_size,int indices_now){
		int id = threadIdx.x;
		int group_id = id >> 1;
		int sub_id = id & 1;
		int opposite_sub_id = sub_id ^ 1; //!sub_id

		int indices_next = (indices_now + 1) % QUEUE_SIZE;
		int temp_blocks[] = {block_size, block_2_size};

		int *myList = list_p[indices_now][ group_id ^ sub_id ];
		int *oppositeList = list_p[indices_now][ group_id ^ opposite_sub_id ];
		int myLen = temp_blocks[sub_id];
		int oppositeLen = temp_blocks[ opposite_sub_id ];
		int value = myList[ myLen - 1 ];
		int left = 0, right = oppositeLen -1;

		while ( left < right ){
			int mid = (left + right + 1)/2;
			if ( value < oppositeList[mid] )
				right = mid - 1;
			else left = mid;
		}

		//printf("id:%d value %d get to %d\n",id,value,oppositeList[left]);//debug

		int next_opposite_offset = left + ( oppositeList[left] <= value );

		int *save_indices0 = calculated_indices_len[indices_now];
		int *save_indices = save_indices0 + (group_id << 1);
		int *opposite_indices = save_indices0 + ( (group_id^1) << 1);
		save_indices[sub_id] = next_opposite_offset ;

		int mysum = save_indices[ sub_id ] + save_indices[ opposite_sub_id ];
		int oppositeSum = opposite_indices[ sub_id ] + opposite_indices[ opposite_sub_id ];

		//decide opposite list offset and length
		if ( mysum + group_id > oppositeSum){

			//printf("cal TURN: id %d len: %d  \n",id,next_opposite_offset);//debug
			save_indices0[ opposite_sub_id ] = next_opposite_offset - 1;
			list_p[indices_now][sub_id] = myList;

			list_p[indices_next][ opposite_sub_id ] = oppositeList + next_opposite_offset;//set opposite pointer
			swapped[ indices_next ] = swapped[ indices_now ] ^ group_id;
		}
	}

	__device__ void bitonic_merge(bool up, volatile int * data,int id, int n){
		if ( n <1 ) return ;
		int n2 = n >> 1;

		syncthreads();
		if  ((data[id] > data[id + n]) ^ up ){
			int t = data[id];
			data[id] = data[id+n];
			data[id+n] = t;
		}
		//syncthreads();
		// change part of the thread
		if ( id >= n2 )
			id -= n2,data += n;
		bitonic_merge(up, data,id, n2);
	}

	//config should be dim3 ths(16,2)
	__device__ void cal_indx_2(int parts,int part_size,int block_size,int block2_size,int indices_now){
		int id = threadIdx.x;
		int myside = threadIdx.y;
		int opposite_side = !myside;
		int idx = (id+1)*part_size-1;
		int opposite_idx;
		int *myList = list_p[indices_now][ myside ];
		int *oppositeList = list_p[indices_now][ opposite_side ];


		int myValue = myList[ idx ];

		//printf("%d %d parts:%d\n",idx,myside,opposite_side);


		int temp_len[] ={ block_size,block2_size };
		//------ bsearch upper bound part----------
		int left = 0,right = temp_len[opposite_side];
		while ( left < right ){
			int mid = (left + right + 1)/2;
			if ( myValue < oppositeList[mid] )
				right = mid - 1;
			else left = mid;
		}
		//------ END bsearch upper bound part----------
		opposite_idx = left + ( oppositeList[left] <= myValue ) - 1;

		__shared__ volatile int shared[2][64];
		__shared__ volatile int shared2[64];
		shared[myside][ parts - id - 1 ] = idx; //reverse save
		shared[opposite_side][ parts + id ] = opposite_idx;

		syncthreads();
		bitonic_merge(false,shared[myside],id,parts);
		syncthreads();

		partitions[indices_now][ id + 1 ][ myside ] = shared[myside][ id ];
		partitions[indices_now][ id + parts + 1 ][ myside ] = shared[myside][ id + parts ];

		//print out
//		if ( !myside && !id ){
//			FOR_I(0,parts*2){
//				int a = shared[0][ i ];
//				int b = shared[1][ i ];
//				printf("[%d]:%d --- [%d]:%d\n",a,myList[a],b,oppositeList[b]);
//			}
//		}
	}

	__global__ void cal_indx (int block_size,int block_2_size,int indices_now){
		//cal_indx_1(block_size,block_2_size,indices_now);
		int n = blockDim.x;
		cal_indx_2(n,block_size/n,block_size,block_2_size,indices_now);
	}



	//special case for cal_indices
	__global__ void move_indices(int len1,int len2,int block_size,int indices_now){
		int indices_next = (indices_now + 1) % QUEUE_SIZE;
		int swapflag = 0;
		//make sure that len1 is always the shorter one
		if ( len1 > len2 ){
			int *temp = list_p[indices_now][0];
			list_p[indices_now][0] = list_p[indices_now][1];
			list_p[indices_now][1] = temp;
			int temp2 = len1;
			len1 = len2;
			len2 = temp2;
			swapflag = 1;
		}
		int b1 = min(len1,block_size);
		list_p[ indices_next ][ 0 ] = list_p[ indices_now ][ 0 ] + b1;
		list_p[ indices_next ][ 1 ] = list_p[ indices_now ][ 1 ] ;//keep the same
		swapped[ indices_next ] = swapped[ indices_now ] ^ swapflag;
		calculated_indices_len[ indices_now ][ 0 ] = b1-1;// or b1
		calculated_indices_len[ indices_now ][ 1 ] = len2-1;// or b1
	}

