#include "hip/hip_runtime.h"
#include "read_data.cuh"
#include "../common_defines.h"
#include <cstdio>
#include <fstream>
#include <iostream>
#include <vector>

//using namespace std;

const char *dir = "/home/sudakentemp/data1.dat";
const int MAX_BLOCK_SIZE = 1024 * 3072;

extern struct list_info *cpuData;
/*
 * Store in int 4 byte
 * Every list start from a len, the id,id,id...
 *
 * */

	struct list_info read_one_list(int * datapool,int file_offset,std::ifstream &reader){
		struct list_info result;
		reader.seekg(file_offset,reader.beg);
		int list_len = -1;
		reader.read((char *)&list_len,4);
		reader.read((char *)datapool,4*list_len);
		result.start_addr = datapool;
		result.len = list_len;
		return result;
	}

	bool filter(int len){
		if (len < 1000*100)
			return false;
		return true;
	}

	void readStructure(std::ifstream &reader,int *lens,int &availiable_list_num,int &data_len_sum){
		int fileLen;
		reader.seekg(0,reader.end);
		fileLen = reader.tellg();
		reader.seekg(0,reader.beg);
		std::cout<<"file length = "<<fileLen<<std::endl;

		int current_offset = 0;
		int list_num = 0;
		availiable_list_num = 0;
		data_len_sum = 0;
		while (current_offset < fileLen){
			reader.seekg(current_offset,reader.beg);
			int list_len = -1;
			reader.read((char *)&list_len,4);
			if (filter(list_len)){
				lens[availiable_list_num ++] = current_offset;
				data_len_sum += list_len;
			}
			current_offset += list_len*4 +4;
			list_num ++;
		}
	}

	__global__ void show_data(struct list_info *list_info){
		printf("len is %d\n",list_info->len);
		FOR_I(0,10){
			printf("%d\t",list_info->start_addr[i]);
		}
	}

	__global__ void show_raw_data(int *addr){
		FOR_I(0,10){
			printf("%d\t",addr[i]);
		}
	}

	__global__ void correct_address(struct list_info *list_infos,int size,int *cpu_addr0,int *gpu_addr0){
		int id = threadIdx.x;
		while (id < size){
			list_infos[id].start_addr = list_infos[id].start_addr - cpu_addr0 + gpu_addr0;
			//printf("<%d> %lld\n",id,list_infos[id].start_addr);
			id += blockDim.x;
		}

	}

void read_gov2(){

	std::ifstream reader;
	reader.open(dir,std::fstream::binary | std::fstream::in);
	int list_num;
	int array[10000];
	int sum;
	readStructure(reader,array,list_num,sum);
	printf("SUM = %d, list_num %d\n",sum,list_num);//debug
	data_pool = new int[sum];
	int* current_pool_pos = data_pool;
	cpuData = new struct list_info[sum];
	FOR_I(0,list_num){
		struct list_info temp = read_one_list(current_pool_pos,array[i],reader);
		current_pool_pos += temp.len;
		cpuData[i] = temp;
	}

	hipMalloc(&gpu_data_pool,4*sum);
	hipMemcpy(gpu_data_pool,data_pool,4*sum,H_T_D);
	hipMalloc(&gpuData,sizeof(struct list_info) * list_num);
	hipMemcpy(gpuData,cpuData,sizeof(struct list_info) * list_num,H_T_D);

	correct_address<<<1,400>>>(gpuData,list_num,data_pool,gpu_data_pool);
	hipDeviceSynchronize();

}



