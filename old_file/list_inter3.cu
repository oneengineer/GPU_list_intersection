#include "hip/hip_runtime.h"
/*New feature:
	pre calculate indices is employed
	smart adaptive: swap or not is decided by lengths
	cudpp prefix sum for naive bsearch
	cudpp prefix sum for my algo
*/

#include <iostream>
#include <fstream>
#include <algorithm>
//#include <string>
//#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#include <unistd.h>
#include <sys/time.h>
#include <cudpp.h>
//#include <memory.h>

#define PI acos(-1)
#define eps 1e-9

#define out(x) (cout<<#x<<":"<<x<<" ")
#define outln(x) (cout<<#x<<":"<<x<<endl)
#define outs(x) (cout<<x)
#define outline (cout<<endl)
#define mssleep(time) usleep((time)*(10*1000))

#define FOR_I(begin,end) for (int i=begin;i<end;i++)
#define FOR_J(begin,end) for (int j=begin;j<end;j++)
#define FOR_K(begin,end) for (int k=begin;k<end;k++)
#define FOR_I_J(B1,E1,B2,E2) FOR_I(B1,E1) FOR_J(B2,E2)
#define FOR_I_J_K(B1,E1,B2,E2,B3,E3) FOR_I_J(B1,E1,B2,E2) FOR_K(B3,E3)
#define FOR(begin,end) FOR_I(begin,end)
#define FORN(end) FOR_I(0,end)

#define SWAP(a,b) { a=(a)-(b);b=(b)+(a);a=(b)-(a);}

//------ for CUDA ------
#include <hip/hip_runtime.h>
//#include <cutil.h>
#include <hip/hip_runtime.h>
#define H_T_D hipMemcpyHostToDevice
#define D_T_H hipMemcpyDeviceToHost
#define D_T_D hipMemcpyDeviceToDevice
#define CUID (threadIdx.x + blockIdx.x*blockDim.x)
#define CUSIZE (blockDim.x*gridDim.x)

using namespace std;

//for mpi
#define MCW MPI_COMM_WORLD

	template <typename T>
	void debug_a(T * data,int begin,int end){
		for (int i=begin;i<end;i++) cout<<"["<<i<<"]: "<<data[i]<<"\t";cout<<endl;
	}
	template <typename T>
	void debug_a(T * data,int end){
		debug_a(data,0,end);
	}
	template <typename T>
	void debug_a2(T * data,int end1,int end2){
		for (int i=0;i<end1;i++){cout<<"row "<<i<<endl; for (int j=0;j<end2;j++) cout<<"["<<i<<","<<j<<"] "<<data[i][j]<<"\t";cout<<endl;}
	}
	double get_sec(const struct timeval & tval){
		return ((double)(tval.tv_sec*1000*1000 + tval.tv_usec))/1000000.0;
	}

	template <typename T>
	T checkmin(T & data,T value){
		data = min(data,value);
		return data;
	}
struct Watch{
	timeval begin,end;
	void start(){gettimeofday(&begin,NULL);}
	double time(){return get_sec(end)-get_sec(begin);}
	double stop(){gettimeofday(&end,NULL);return time();}
};

	inline double rand_double(double range){
		return  ((double)rand()/(double)RAND_MAX)*range;
	}
//----------- cuda template ------------

struct CudaWatch{
	hipEvent_t start_event, stop_event;
	void start(){
	hipEventCreate(&start_event) ;
	hipEventCreate(&stop_event) ;
	hipEventRecord(start_event,0);
	}
	float stop(){
		float time_cost = -1.0;
		hipEventRecord(stop_event,0);
		hipEventSynchronize(stop_event);
		hipEventElapsedTime(&time_cost,start_event,stop_event);
		return time_cost;
	}
};
	void cu_checkError(){
		hipError_t error = hipGetLastError();
		if (error != hipSuccess){
			printf("ERROR: %s\n",hipGetErrorString(error));
			exit(-1);
		}
	}

	__global__ void cu_print(int *list,int size){
		printf("CUDA SHOW:\n");
		if ( 0 == threadIdx.x && 0 == blockIdx.x ){
			FOR_I(0,size)
				printf("[%d]:%d\t",i,list[i]);
			printf("\n");
		}
	}

	void cu_host_print(int *list,int size){
		hipDeviceSynchronize();
		cu_print<<<1,1>>>(list,size);
		hipDeviceSynchronize();
	}


//-------------------WRITE YOUR PROGRAM FROM HERE


const int N = 55000100;
const int M = 1024*1024;

int list[2][N];
int *devL1,*devL2;

bool v[N*10];

int n,m;

int V[M];
int resultList[N];
int cpuResult[N],cpuResultSize;

int *devV,*devResult, *devMark;

int L2index,L1index,Lresult;
int *devL2index,*devL1index,*devResultIndex;

int block_size;

const int QUEUE_SIZE= 5;

__device__ int *list_p0[2];
__device__ int *list_p[QUEUE_SIZE][2]; // save the starting position of either list
__device__ int calculated_length[QUEUE_SIZE];
__device__ int calculated_indices_len[QUEUE_SIZE][4];
__device__ int swapped[QUEUE_SIZE];   // save swapped stage for each status
__device__ int queue_size = QUEUE_SIZE;
__device__ int indices_now=0, search_now=0, prefix_now=0;
__device__ int _nm[2];
__device__ int *_result;

__device__ double ratio_avg;
__device__ int cal_times;

int * devIndices;

__shared__ int share[2048*2];

	void generate_different(int * array,int n,int mod){
		FOR_I(0,mod) v[i] = false;
		for (int i=0;i<n;i++){
			int j=-1;
			do
			{
				j = rand() % mod;
			}while ( v[j] );
			v[j] = true;
			array[i] = j;
		}
	}

	void generate_random(double alpha=1.0,double scala1=1.0,double scala2=1.0){
		int mod = n*4;
		m = (int)((double)n*alpha + eps);
		int mod1 = (int)((double)mod*scala1 + eps);
		int mod2 = (int)((double)mod*scala2 + eps);
		generate_different(list[0],n,mod1);
		sort(list[0],list[0]+n);
		generate_different(list[1],m,mod2);
		sort(list[1],list[1]+m);
	}

	void generate_case2(){
		printf("a general test. mainly for the search bound\nPlease use D1 = D2 = 2,block2size = 5\n\n");
		int temp1[] = {1,2,3,4,           20,            29, 34,36,37,38 ,39,30,66
					,100,110,120,130,140,200,211,230,231,540 };
		int temp2[] = {1,2,3,4,6,7,10,12, 20,25,26,27,28 ,29                   ,60
				,100,110,120,130,140,200,210,220,230,540};

		n = sizeof(temp1)/sizeof(int);
		m = sizeof(temp2)/sizeof(int);
		FOR_I(0,n) list[0][i] = temp1[i];
		FOR_I(0,m) list[1][i] = temp2[i];
	}

	void generate_case3(){
		printf("a generate test. mainly for the search swap\nPlease use D1 = 1; D2 = 3,block2size = 5\n\n");
		int temp1[] = {1,  2,  9,   11,15,16,17,26,27
					,100,110,120,130,140,200,211,230,231,540 };
		int temp2[] = {1,2,5,6,9,          21,25,  27
					,100,110,120,130,140,200,210,220,230,540 };

		n = sizeof(temp1)/sizeof(int);
		m = sizeof(temp2)/sizeof(int);
		FOR_I(0,n) list[0][i] = temp1[i];
		FOR_I(0,m) list[1][i] = temp2[i];
	}

	void generate_case4(){
		printf("a end point test. Please use D1 = 2; D2 = 2,block2size = 6\n\n");
		int temp1[] = {196};
		int temp2[] = {195,196,200};

		n = sizeof(temp1)/sizeof(int);
		m = sizeof(temp2)/sizeof(int);
		FOR_I(0,n) list[0][i] = temp1[i];
		FOR_I(0,m) list[1][i] = temp2[i];
	}

	void init_data(){
		int sizeV = 2*block_size + 32;
		int *temp = new int [sizeV];
		FOR_I(0,sizeV) temp[i] = 0;

		hipMalloc( (void **)&devL1, sizeof(int)*n );
		hipMalloc( (void **)&devL2, sizeof(int)*m );
		hipMalloc( (void **)&devV, sizeof(int)*sizeV );
		hipMalloc( (void **)&devResult, sizeof(int)*n );
		hipMalloc( (void **)&devMark, sizeof(int)*(n+1) );
		cu_checkError();

		hipMalloc( (void **)&devL2index, sizeof(int) );
		hipMalloc( (void **)&devL1index, sizeof(int) );
		hipMalloc( (void **)&devResultIndex, sizeof(int) );
		cu_checkError();

		hipMemcpy( devL1,list[0],sizeof(int)*n,H_T_D );
		hipMemcpy( devL2,list[1],sizeof(int)*m,H_T_D );
		hipMemcpy( devMark,temp,sizeof(int),H_T_D );
		hipMemcpy( devV,temp,sizeof(int)*sizeV,H_T_D );
		devV = devV + 32 + block_size;
		cu_checkError();
	}

	void show_lists(int index1,int index2,int len1=n,int len2=n){
		printf("Two lists:\n");
		printf("L1 :\t");debug_a(list[0]+index1,len1);
		printf("L2 :\t");debug_a(list[1]+index2,len2);
		outline;
	}

	void cuda_copyResult(){
		int *end;
		hipMemcpyFromSymbol(&end,HIP_SYMBOL(_result),sizeof(int *),0,D_T_H );
		Lresult = (int)(end - devResult );
		hipMemcpy( resultList,devResult, sizeof(int)*Lresult,D_T_H );
	}

	__device__ int next_queue_pos(int value){
			return (value+1) % queue_size;
	}

	__global__ void prefix_sum_oneCore(int *V,int block_size){
		FOR_I(1,block_size)
			V[i] += V[i-1];
	}

//------------------ prefix sum over ---------------

	//binary search lower bound in list2 for each element in list1
	//algo2 should not care about swap or not
	__global__ void algo2_search(int * V){
		int id = CUID;

		if (id > calculated_indices_len[indices_now][0]){
//			printf("id %d quit!\n",id);
			return;
		}
		int left = 0;
		int right = calculated_indices_len[indices_now][1];

		int *list1 = list_p[ indices_now ][ 0 ];
		int *list2 = list_p[ indices_now ][ 1 ];
		int value = list1[ id ];

//		if(!id) { printf("R:%d\n",right) ;FOR_I(0,right+1) printf("!%d\t",list2[i]); printf("\n"); }

		int result = 0;
		while ( left <= right ){
			int mid = (left + right)/2;
			if ( value == list2[ mid ] ){
//				printf("id: %d INTERSECTION %d\n",id,value);
				result = 1;
				break;
			}
			else if ( list2[mid] < value )
				left = mid + 1;
			else right = mid - 1;
		}
		V[id] = result;
		//if ( value == 196 )
//		printf("  algo2 search id: %d  %d get [%d] %d\n",id, value,right,list2[right]);
	}

	__global__ void save_result(int *V){
		int _size = calculated_indices_len[indices_now][0];
		int id = CUID;
		if ( id > _size )
			return;
		int *list1 = list_p[indices_now][0];

		if (V[id] - V[id-1] >0){
			_result[ V[id-1] ] = list1[ id ];

		}


	}

	//can only be executed by one core
	__global__ void move_result_addr(int *V){
		int _size = calculated_indices_len[indices_now][0];
		_result += V[_size];
	}

	__global__ void init_device_var_kernel(){

		int id = CUID;
		if (id) return;
		list_p[0][0] = list_p0[0];
		list_p[0][1] = list_p0[1];
		swapped[0] = 0;//false
		ratio_avg = 0.0;//for debug
		cal_times = 0;//for debug
		indices_now = search_now = prefix_now= -1;
	}


	void init_device_variables(){
		hipMemcpyToSymbol(HIP_SYMBOL(list_p0),&devL1,sizeof(int *),0,H_T_D);
		cu_checkError();
		hipMemcpyToSymbol(HIP_SYMBOL(list_p0),&devL2,sizeof(int *),0+sizeof(int *),H_T_D);
		int temp[] = {n,m};
		hipMemcpyToSymbol(HIP_SYMBOL(_nm),temp,sizeof(int)*2,0,H_T_D);

		hipMemcpyToSymbol(HIP_SYMBOL( _result),&devResult,sizeof(int*),0,H_T_D);
		cu_checkError();
		init_device_var_kernel<<<1,1>>>();
		cu_checkError();
		hipDeviceSynchronize();
		hipMalloc( (void **)&devIndices , sizeof(int )*2);
	}

	void free_device_memory(){
		hipFree(devL1);
		hipFree(devL2);
		hipFree(devResult);
		hipFree(devMark);
	}


	//calculate the start point of next block ( pointer might overflow )
	// two threads calculate and write opposite lowerbound (each other)
	// id my index !id opposite index
	__global__ void calculate_indices(int block_size,int block_2_size,int isSwap){

		int id = CUID;
		if ( id > 1 ) return;

		int opposite = !id;

		if ( 0 == id)
			indices_now = (indices_now + 1) % queue_size;

		int temp_arr[] = {block_2_size,block_size};
		int left = 0;
		int right = temp_arr[id] - 1;

		int value = list_p[indices_now][ id ][ temp_arr[opposite] -1];

		//printf("CI: id %d| [ %d ] %d search (%d %d)\n",id, (int)(list_p[indices_now][0] - list_p0[0]) , value,list2[left],list2[right]);//debug

		while ( left < right ){
			int mid = (left + right + 1)/2;
			if ( value < list_p[indices_now][ opposite ][mid] )
				right = mid - 1;
			else left = mid;
		}

		//printf("CI: id %d|   %d search to [%d] %d\n",id,value,left,list2[left]);//debug
		int next = next_queue_pos(indices_now);
		//list_p[next][ (0+isSwap) %2 ] = list_p[indices_now][0]+block_size;

		int next_opposite_index = (opposite)^isSwap;

		int next_offset = left+ ( list_p[indices_now][ opposite ][left] <= value );
		//Attention <= is the lower bound condition for edge condition
		list_p[next][ next_opposite_index ] = list_p[indices_now][ opposite ] + next_offset;

		if (0 == id){
			calculated_length[ indices_now ] = next_offset - 1;
			swapped[next] = swapped[indices_now] ^ isSwap;
		}
	}

	__global__ void cal_indx (int block_size,int block_2_size){
		int id = threadIdx.x;
		int group_id = id >> 1;
		int sub_id = id & 1;
		int opposite_sub_id = sub_id ^ 1; //!sub_id

		indices_now = (indices_now + 1) % queue_size;
		int indices_next = (indices_now + 1) % queue_size;
		int temp_blocks[] = {block_size, block_2_size};

		int *myList = list_p[indices_now][ group_id ^ sub_id ];
		int *oppositeList = list_p[indices_now][ group_id ^ opposite_sub_id ];
		int myLen = temp_blocks[sub_id];
		int oppositeLen = temp_blocks[ opposite_sub_id ];
		int value = myList[ myLen - 1 ];
		int left = 0, right = oppositeLen -1;

		while ( left < right ){
			int mid = (left + right + 1)/2;
			if ( value < oppositeList[mid] )
				right = mid - 1;
			else left = mid;
		}

		//printf("id:%d value %d get to %d\n",id,value,oppositeList[left]);

		int next_opposite_offset = left + ( oppositeList[left] <= value );

		int *save_indices0 = calculated_indices_len[indices_now];
		int *save_indices = save_indices0 + (group_id << 1);
		int *opposite_indices = save_indices0 + ( (group_id^1) << 1);
		//syncthreads(); // UNNECESSARY
		save_indices[sub_id] = next_opposite_offset ;
		//if (!id) FOR_I(0,4) printf("[ %d ] %d\t",i,save_indices0[i]);
		//syncthreads(); // UNNECESSARY

		int mysum = save_indices[ sub_id ] + save_indices[ opposite_sub_id ];
		int oppositeSum = opposite_indices[ sub_id ] + opposite_indices[ opposite_sub_id ];

		//decide opposite list offset and length
		if ( mysum + group_id > oppositeSum){

//			if (!sub_id){
//				FOR_I(0,block_2_size) printf("%d\t",myList[i]);printf("\n");
//				FOR_I(0,block_2_size) printf("%d\t",oppositeList[i]);printf("\n");
//			}

			save_indices0[ opposite_sub_id ] = next_opposite_offset - 1;
			//attention may swap current list
			//also save_indices is to save current indices
			list_p[indices_now][sub_id] = myList;

			//In fact there is no need to swap or record whether it is swapped or not,
			//Because it always do double direction check(4 threads), the only need is to record the current start position
			//But copy indices is based on current next position, just record it
			list_p[indices_next][ opposite_sub_id ] = oppositeList + next_opposite_offset;//set opposite pointer
			swapped[ indices_next ] = swapped[ indices_now ] ^ group_id;

//			printf("id:%d value:%d --> [%d] %d \n",id,value,next_opposite_offset,oppositeList[next_opposite_offset]);//debug

			if (!sub_id ){ //debug part
//				printf("id:%d group_id:%d say: mysum %d \topposum2 %d   swapped: %d\n",id,group_id,mysum,oppositeSum,swapped[ indices_next ]);
//				FOR_I(0,4) printf("[ %d ] %d\t",i,save_indices0[i]);
//				printf("\n-------------\n");
			}

		}
	}

	//special case for cal_indices
	__global__ void move_indices(int len1,int len2,int block_size){
		indices_now = (indices_now + 1) % queue_size;
		int indices_next = (indices_now + 1) % queue_size;
		int swapflag = 0;
		//make sure that len1 is always the shorter one
		if ( len1 > len2 ){
			int *temp = list_p[indices_now][0];
			list_p[indices_now][0] = list_p[indices_now][1];
			list_p[indices_now][1] = temp;
			SWAP(len1,len2);
			swapflag = 1;
		}
		int b1 = min(len1,block_size);
		list_p[ indices_next ][ 0 ] = list_p[ indices_now ][ 0 ] + b1;
		list_p[ indices_next ][ 1 ] = list_p[ indices_now ][ 1 ] ;//keep the same
		swapped[ indices_next ] = swapped[ indices_now ] ^ swapflag;
		calculated_indices_len[ indices_now ][ 0 ] = b1-1;// or b1
		calculated_indices_len[ indices_now ][ 1 ] = len2-1;// or b1
	}


	// configure : <<<1,2>>> only and but only 2 threads
	__global__ void helpLen_relative(int *memory){
			int temp = next_queue_pos(indices_now);
			int id = threadIdx.x;
			int isSwapped = swapped[ temp ];
			int myindex = id ^ isSwapped;// id:1  !swapped

			memory[id] = list_p[temp][ id ]-list_p0[ myindex ];
			memory[id] = _nm[myindex] - memory[id];

		}

	void back_next_relative_len(int & len1,int & len2){
		int temp[2];
		helpLen_relative<<<1,2>>>(devIndices);
		hipMemcpy(temp,devIndices,sizeof(int)*2, D_T_H);
		len1 = temp[0];len2 = temp[1];
	}

	__global__ void show_cur_lists(int len1,int len2){
		FOR_I(0, calculated_indices_len[indices_now][0] )
				printf("{%d} %d\t",i,list_p[indices_now][0][i]);printf("\n");
		FOR_I(0, calculated_indices_len[indices_now][1])
				printf("{%d} %d\t",i,list_p[indices_now][1][i]);printf("\n");
	}

	CUDPPHandle prepare_prefixsum(int size){
		CUDPPConfiguration config;
		config.op = CUDPP_ADD;
		config.datatype = CUDPP_INT;
		config.algorithm = CUDPP_SCAN;
		config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;

		CUDPPHandle theCudpp;
		cudppCreate(&theCudpp);
		CUDPPHandle scanplan = 0;
		CUDPPResult res = cudppPlan(theCudpp,&scanplan,config,size,1,0);
		if ( CUDPP_SUCCESS != res ){
			printf("ERROR in prepare_prefixsum\n");
			exit(-1);
		}
		return scanplan;
	}

	void work(){
//		int numStream;
//		hipStream_t *streams = (hipStream_t *) malloc(numStream * sizeof(hipStream_t));
//		for (int i = 0; i < numStream; i++)
//		{
//			hipStreamCreate(&(streams[i]));
//		}

		int D1,D2;
		int block_2_size;
		//------ some settings ----------
		D1 = 64*4;
		D2 = 64*2;

		//D1 = 14;
		//D2 = 10;

		block_size = D1 * D2;
		block_2_size = block_size;
		//block_2_size = block_size*1000;

		init_data();
		init_device_variables();
		CudaWatch cudawatch;
		cudawatch.start();

		CUDPPHandle prefixsum_plan = prepare_prefixsum(block_size);

		int len1,len2;
		while (true){

			back_next_relative_len(len1,len2);

			//outln(len1);outln(len2);//debug
			if ( len1<=0 || len2 <= 0 ) break;

			int loops = min(len1,len2)/block_2_size;

			if ( loops >0 ){
				//outln(loops);//debug
				FOR_I(0,loops){
					cal_indx<<<1,4>>>(block_size,block_2_size);
					algo2_search<<<D1,D2>>>(devV);
					//hipMemcpy(V,devV,sizeof(int)*block_size,D_T_H);debug_a(V-1,block_size+1);
					cudppScan(prefixsum_plan,devV,devV,block_size);
					//hipMemcpy(V,devV,sizeof(int)*block_size,D_T_H);debug_a(V-1,block_size+1);
					save_result<<<D1,D2>>>(devV);
					move_result_addr<<<1,1>>>(devV);

				}//end loops
			}
			else {
				//cout<<"small seg "<<len1<<" "<<len2<<endl;
				move_indices<<<1,1>>>(len1,len2,block_size);
				algo2_search<<<D1,D2>>>(devV);
				cudppScan(prefixsum_plan,devV,devV,block_size);
				save_result<<<D1,D2>>>(devV);
				move_result_addr<<<1,1>>>(devV);
			}
		}
		cout<<"MY Algo:"<<cudawatch.stop()/1000.0<<endl;
		free_device_memory();
	}

	void merge_algo(int *array1,int *array2, int begin1,int end1,int begin2,int end2){
		//return ;
		int i=begin1,j=begin2;
		int lasti,lastj;
		cpuResultSize = 0;
		lasti=array1[i];
		lastj=array2[j];
		while ( i<end1 && j<end2){
			if ( lasti == lastj ){
				//V2[i] = 1;
				cpuResult[cpuResultSize++] = lasti;
				i++;j++;
				lasti=array1[i];
				lastj=array2[j];
			}
			else if ( lasti < lastj){
				i++;
				lasti = array1[i];
			}
			else {
				j++;
				lastj = array2[j];
			}
		}
	}

	int check_correctness(){
			FOR_I(0,cpuResultSize)
				if (cpuResult[i] != resultList[i]){
					printf("Wrong at [%d] R:%d Yours:%d\n",i,cpuResult[i],resultList[i]);
					debug_a( cpuResult+i,-10,10 );
					debug_a( resultList+i,-10,10 );
					return i;
				}
		return -1;
	}

	__global__ void algo_bsearch(
			int *list1,int *list2
			,int *mark
			,int n,int m){
		int i = CUID;
		int size = CUSIZE;
		while ( i < n){
			int value = list1[i];
			int left = 0,right = m-1;
			mark[i] = 0;
			while (left <= right){
				int mid = (left+right)/2;
				int difference = value-list2[mid];
				if ( difference <0 )
					right = mid-1;
				else if ( difference >0 )
					left = mid +1;
				else {
					mark[i] = 1;
					break;
				}
			}
			i += size;
		}
	}

	__global__ void bsearch_copy_result(int * list1,int *mark, int *result,int n){
		int id = CUID;
		int size = CUSIZE;
		while ( id < n ){
			if (mark[id] - mark[id-1] >0){
				result[ mark[id-1] ] = list1[ id ];
			}
			id += size;
		}
	}

	void work3(){
		init_data();
		//outln(n);
		CUDPPHandle prefixsum_plan = prepare_prefixsum(n+1);
		CudaWatch cudawatch;cudawatch.start();
		int * markFrom = devMark+1;

		algo_bsearch<<<32,512>>>(devL1,devL2,markFrom,n,m);
//		cu_host_print(devMark,n+1);//debug

		cudppScan(prefixsum_plan,devMark,devMark,n+1);
//		cu_host_print(devMark,n+1);//debug

		//cu_checkError();
		bsearch_copy_result<<<32,512>>>(devL1,markFrom,devResult,n+1);
		//cu_checkError();
		cout<<"NAIVE Bsearch:"<<cudawatch.stop()/1000.0<<endl;
		hipMemcpy(&Lresult,devMark+n,sizeof(int),D_T_H);cu_checkError();
		hipMemcpy(resultList,devResult,sizeof(int)*Lresult,D_T_H);cu_checkError();
		free_device_memory();
	}




int main(){
	cout<<"PROGRAM START "<<endl;
	//test_cal_indices();return 0;
	//test_prefix_sum();

	//test_bsearch();return 0;

	int r;
	FOR_I(103,10000){
	r = rand() % 10013123 ;
	//r =1344532745 ;
	srand(i);
	n = 1024*1024*40;
	//n = 1024*1024;

	//generate_case2();
	generate_random(1.0,2.0,2.0);

	cout<<"generate data over srand("<<i<<") n="<<n<<" m="<<m<<endl;
	printf("List 1 ( %d --- %d --- %d )\n",list[0][0],list[0][n/2],list[0][n-1]);
	printf("List 2 ( %d --- %d --- %d )\n",list[1][0],list[1][m/2],list[1][m-1]);

	//	debug_a(list[0],n);outline;debug_a(list[1],m);

	memset(V,0,sizeof(V));

	Watch watch;watch.start();
	merge_algo(list[0],list[1],0,n,0,m);
	cout<<"CPU ALGO time: "<<watch.stop()<<endl;

	work();
	//cuda_copyResult();cout<<"copied back"<<endl;
	cu_checkError();
	//work3();//test bsearch

	//outln(Lresult);outln(cpuResultSize);
//	debug_a(resultList,Lresult);
//	debug_a(cpuResult,cpuResultSize);
	break;


	//continue;// DO NOT CHECK correctness

	int error_index;
	if  ( (error_index = check_correctness()) >=0 ){
		outline;
		outln(i);
		printf("all length: cpu:%d gpu:%d  ration: %.5lf\n",cpuResultSize,Lresult,100.0*i/(1.0*Lresult));
		break;
	}
		outline;
		outline;
		outline;
		outline;

		break; //only execute once
	}//end FOR
	return 0;
}
