#include "hip/hip_runtime.h"
/*New feature:
 pre calculate indices, smart adaptive, fully pipeline, not fully hidden
 cudpp have streams, but number of device block/threads can not be controlled
 cudpp prefix sum for my algo
 pipeline ( cudpp pipelined) is employed
 pipeline ( myscan pipelined) is employed
 */

#include <iostream>
#include <fstream>
#include <algorithm>
#include <string>
#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#include <unistd.h>
#include <sys/time.h>

//------ for CUDA ------
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "common_defines.h"
#include "common_functions.h"
#include "others/bsearch.cuh"
#include "others/memory_data.cuh"
#include "indices/cal_indices.cuh"
#include "search/search.cuh"
#include "save/scan.cuh"
#include "save/save_result.h"
#include "others/read_data.cuh"

#include "others/generate_data.h"

//#include "cudpp.h"

using namespace std;

template<typename T>
void debug_a(T * data, int begin, int end) {
	for (int i = begin; i < end; i++)
		cout << "[" << i << "]: " << data[i] << "\t";
	cout << endl;
}
template<typename T>
void debug_a(T * data, int end) {
	debug_a(data, 0, end);
}
template<typename T>
void debug_a2(T * data, int end1, int end2) {
	for (int i = 0; i < end1; i++) {
		cout << "row " << i << endl;
		for (int j = 0; j < end2; j++)
			cout << "[" << i << "," << j << "] " << data[i][j] << "\t";
		cout << endl;
	}
}
double get_sec(const struct timeval & tval) {
	return ((double) (tval.tv_sec * 1000 * 1000 + tval.tv_usec)) / 1000000.0;
}

template<typename T>
T checkmin(T & data, T value) {
	data = min(data, value);
	return data;
}
struct Watch {
	timeval begin, end;
	void start() {
		gettimeofday(&begin, NULL);
	}
	double time() {
		return get_sec(end) - get_sec(begin);
	}
	double stop() {
		gettimeofday(&end, NULL);
		return time();
	}
};

inline double rand_double(double range) {
	return ((double) rand() / (double) RAND_MAX) * range;
}
//----------- cuda template ------------

struct CudaWatch {
	hipEvent_t start_event, stop_event;
	void start() {
		hipEventCreate(&start_event);
		hipEventCreate(&stop_event);
		hipEventRecord(start_event, 0);
	}
	float stop() {
		float time_cost = -1.0;
		hipEventRecord(stop_event, 0);
		hipEventSynchronize(stop_event);
		hipEventElapsedTime(&time_cost, start_event, stop_event);
		return time_cost;
	}
};
void cu_checkError() {
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("ERROR: %s\n", hipGetErrorString(error));
		exit(-1);
	}
}

__global__ void cu_print(int *list, int size) {
	printf("CUDA SHOW:\n");
	if (0 == threadIdx.x && 0 == blockIdx.x) {
		FOR_I(0,size)
			printf("[%d]:%d\t", i, list[i]);
		printf("\n");
	}
}

void cu_host_print(int *list, int size) {
	hipDeviceSynchronize();
	cu_print<<<1, 1>>>(list, size);
	hipDeviceSynchronize();
}

//-------------------WRITE YOUR PROGRAM FROM HERE

int *host_lists[2];

int *devL1, *devL2;
bool *v;
int *devV[QUEUE_SIZE], *devResult;
int * devIndices;
int n, m;

int Lresult = 0;
int *resultList;
int *cpuResult;

struct list_info *cpuData;
struct list_info *gpuData;
int *data_pool;
int *gpu_data_pool;

__device__ int *list_p0[2]; // the origional   address
__device__ int *list_p[QUEUE_SIZE][2]; // save the starting position of either list
__device__ int calculated_indices_len[QUEUE_SIZE][2];
__device__ int swapped[QUEUE_SIZE]; // save swapped stage for each status
__device__ int *_result;
__device__ int gpu_result_size;
__device__ int _nm[2];
__device__ partition_info partitions_info[QUEUE_SIZE][DEF_D1*2 + 4];
__device__ int *_result_addr[QUEUE_SIZE];
__device__ debug_structure debug1;

__device__ int o_scan_buffers[QUEUE_SIZE][4+DEF_D1*2+32*2];
__device__ int *_scan_buffers[QUEUE_SIZE];

inline void move_pos(int &pos) {
	pos = (pos + 1) % QUEUE_SIZE;
}

__global__ void help_show_partation(int indices_now, int num_part,
		bool printout = false) {
	FOR_I(0,num_part)
	{
		struct partition_info *info = &partitions_info[indices_now][i];
		if (info->len <=0) continue;

		printf("Partation: %d\n", i);
		printf("[%llx]: %d --- [%llx]:%d\n", info->addr, *(info->addr),
				info->opposite_addr, *(info->opposite_addr));
		printf("[%lld]: --- [%lld]:\n", info->addr - list_p0[info->B2A],
				info->opposite_addr - list_p0[!info->B2A]);
		if (printout) {
			FOR_I(0,info->len*4)
				printf("%d \t", info->addr[i]);
			printf("\n |||||||| \n");
			int xx = info->right + (3 - info->right % 4) % 4 + 1;
			FOR_I(0, xx)
				printf("%d \t", info->opposite_addr[i]);
			//FOR_I(info->left,info->right+1) printf("%d \t",info->opposite_addr[i]);
			printf("\n");
		}
		printf("len:%d \t oppt_len:%d left:%d\t right:%d \t B2A:%d\n", info->len,info->len_opposite,
				info->left, info->right, info->B2A);
	}
}

__global__ void help_show_partation2(int indices_now, int num_part) {
	FOR_I(0,num_part)
	{
		struct partition_info *info = &partitions_info[indices_now][i];
		printf("Partation: %d\n", i);
		printf("len:%d \t left:%d\t right:%d \t B2A:%d warp_len:%d\n",
				info->len, info->left, info->right, info->B2A, info->warp_len);
		long long offset1, offset2;
		if (info->B2A) {
			offset1 = info->opposite_addr - list_p0[0];
			offset2 = info->addr - list_p0[1];
		} else {
			offset1 = info->addr - list_p0[0];
			offset2 = info->opposite_addr - list_p0[1];
		}
		printf("offset : < %lld --- %lld >", offset1, offset2);
		printf("\n---------------------\n");
	}
}



__global__ void show_list2(){
	FOR_I(0,10)
			printf("L1[%d] = %d\n",i,list_p0[0][i]);
	FOR_I(0,10)
			printf("L1[%d] = %d\n",i,list_p0[1][i]);

}

	void work(int l1_id,int l2_id) {

		int numStream;
		numStream = 4;
		hipStream_t *streams = (hipStream_t *) malloc(
				numStream * sizeof(hipStream_t));
		for (int i = 0; i < numStream; i++) {
			hipStreamCreate(&(streams[i]));
		}

		int D1, D2 ,D_save1,D_save2;
		int block_size;
		int block_2_size;
		//------ some settings ----------
		D1 = DEF_D1;
		D2 = DEF_D2;

		printf("CASE config:<%d %d>\n",D1,D2);

		D_save1 = max(1,2*D1/16); // one block deal 4 parts
		D_save2 = 64; // 64 threads per block for compacting

		//D1 = 16;D2 = 64;
		//D1 = 8;D2 = 64*4;//D2 has to be the multiply of WARP_SIZE, because

		hipStream_t &cal_index_stream= streams[0];
		hipStream_t &search_stream= streams[0];
		hipStream_t &save_stream= streams[0];

		block_size = D1 * D2;
		block_2_size = block_size;
		//block_2_size = block_size*1000;



		//init_data(block_size);
		init_real_data(block_size,l1_id,l2_id);
		init_device_variables();

		dim3 cal_indx_setting(D1, 2);

		CudaWatch cudawatch;
		Watch cpuWatch;
		cpuWatch.start();
		cudawatch.start();
		cu_checkError();

		//printf("addr : %llx %llx",devL1,devL2);

		int len1, len2;
		int cal_pos = 0, search_pos = 0, save_pos = 0; // they are for L1, L2 position
		while (true) {
			back_next_relative_len(len1, len2, cal_pos);
			//cu_checkError();

			if (len1 <= 0 || len2 <= 0) break;
			int loops = min(len1, len2) / block_size;
			if ( min(len1,len2) % block_size != 0 ) loops ++;

			printf(" left n:%d  left m: %d loops:%d\n",len1,len2,loops);
			if (loops > 0) {
				//-- first stage
				hipDeviceSynchronize();

				cal_indx<<<1, cal_indx_setting,0,cal_index_stream>>>(block_size, block_2_size,cal_pos);
				move_pos(cal_pos);
				//-- second stage
				if (loops >= 2){
					cal_indx<<<1, cal_indx_setting,0,cal_index_stream>>>(block_size, block_2_size,cal_pos);
					move_pos(cal_pos);
				}
				else hipStreamSynchronize(cal_index_stream);
				algo2_search<<<2 * D1, D2 / 4,0,search_stream>>>(devV[search_pos], search_pos, 0);
				move_pos(search_pos);
				hipStreamSynchronize(search_stream);

				//-- stage middle
				for (int i=3;i<loops;i++) {
					//help_debug<<<1,1>>>(loops);
					//outline;outline;outln(i);
					cal_indx<<<1, cal_indx_setting,0,cal_index_stream>>>(block_size, block_2_size,cal_pos);
					algo2_search<<<2 * D1, D2 / 4,0,search_stream>>>(devV[search_pos], search_pos, 0);
					scan_buffer_large<<<1, max( 2*D1,32),0,save_stream>>>(save_pos);
					compact<<<D_save1, D_save2,0,save_stream>>>(2*D1/D_save1,devV[save_pos],save_pos);
					move_pos(cal_pos);move_pos(search_pos);move_pos(save_pos);
					//hipDeviceSynchronize();
				}
				hipStreamSynchronize(save_stream);
				if (loops >=2 ){
					algo2_search<<<2 * D1, D2 / 4,0,search_stream>>>(devV[search_pos], search_pos, 0);
					scan_buffer_large<<<1, max( 2*D1,32),0,save_stream>>>(save_pos);
					compact<<<D_save1, D_save2,0,save_stream>>>(2*D1/D_save1,devV[save_pos],save_pos);
					move_pos(search_pos);move_pos(save_pos);
				}
				hipStreamSynchronize(search_stream);
				scan_buffer_large<<<1, max( 2*D1,32),0,save_stream>>>(save_pos);
				compact<<<D_save1, D_save2,0,save_stream>>>(2*D1/D_save1,devV[save_pos],save_pos);
				move_pos(save_pos);
			}
			//return;
		}
		hipDeviceSynchronize();
		//back__result<<<1,1>>>();
		cout << "MY Algo:" << cudawatch.stop() / 1000.0 << endl;
		cout << "MY Algo cpu test: " << cpuWatch.stop() << endl;
	}

int cpuResultSize = 0;
int merge_algo(int *array1, int *array2, int begin1, int end1, int begin2,
		int end2) {
	cpuResultSize = 0;
	int i = begin1, j = begin2;
	int lasti, lastj;

	lasti = array1[i];
	lastj = array2[j];
	while (i < end1 && j < end2) {
		if (lasti == lastj) {
			cpuResult[cpuResultSize++] = lasti;
			i++;
			j++;
			lasti = array1[i];
			lastj = array2[j];
		} else if (lasti < lastj) {
			i++;
			lasti = array1[i];
		} else {
			j++;
			lastj = array2[j];
		}
	}
	return cpuResultSize;
}

int check_correctness(int cpuResultSize) {
	FOR_I(0,cpuResultSize)
		if (cpuResult[i] != resultList[i]) {
			printf("Wrong at [%d] R:%d Yours:%d\n", i, cpuResult[i],
					resultList[i]);
			debug_a(cpuResult + i, -10, 10);
			debug_a(resultList + i, -10, 10);
			return i;
		}
	return -1;
}

void prepare_data(int n) {
	host_lists[0] = new int[n];
	resultList = new int[n];
	host_lists[1] = new int[n];
	v = new bool[20 * n];
	resultList = new int[n];
	cpuResult = new int[n];
}

__global__ void help_show_cal3(int indices_now, int num_part,bool printout = false) {
	FOR_I(0,num_part)
	{
		struct partition_info *info = &partitions_info[indices_now][i];
		printf("Partation: %d %d\n", i,info->len);
		if (printout) {
			FOR_I(0,info->len)
				printf("%d \t", info->addr[i]);
		}
		printf("len:%d \t left:%d\t right:%d \t B2A:%d\n", info->len,info->left, info->right, info->B2A);
	}
}

	void test_cal_idx3(){
		n = 16;
		generate_random(1.0, 0.5, 0.5);
		//generate_case_cal3();
		debug_a(host_lists[0],n);debug_a(host_lists[1],m);//debug
		int block_size = 8;
		init_data(block_size);
		init_device_variables();
		dim3 config(2,2);

		FOR_I(0,2){
//			cal_indx<<<1,config>>>(16,block_size,i);
//			//help_show_cal3<<<1,1>>>(0,4,true);
//			help_show_partation<<<1,1>>>(i,3,true);
//			algo2_search<<<2 * 2, 2>>>(devV[0], i, 0);

		}
		hipDeviceSynchronize();
	}

	void performance_read(char **args,int &l1,int &l2){
		sscanf(args[1],"%d",&l1);
		sscanf(args[2],"%d",&l2);
	}

	void test_read(){
		read_gov2();
		exit(0);
	}

	void cpu_work(int l1_id,int l2_id){

		list_info a,b;
		a = cpuData[l1_id];
		b = cpuData[l2_id];

		printf("L1 use [%d] L2 use[%d]\n",l1_id,l2_id);
		printf("List 1 ( %d --- %d --- %d ) %d\n",a.start_addr[0],a.start_addr[ a.len / 2], a.start_addr[a.len-1],a.len);
		printf("List 1 ( %d --- %d --- %d ) %d\n",b.start_addr[0],b.start_addr[ b.len / 2], b.start_addr[b.len-1],b.len);

		Watch watch;
		watch.start();
		cpuResultSize = merge_algo(a.start_addr, b.start_addr, 0, a.len, 0,b.len);
		cout << "CPU ALGO time: " << watch.stop() << endl;
		cout << "CPU find " << cpuResultSize<< endl;
	}

int main(int arg_num, char ** args) {

	prepare_data(1024 * 1024 * 90);

	read_gov2();
	int l1,l2;
	performance_read(args,l1,l2);
	//l1 = 171,l2=164;

//	test_cal_idx3();
//	return 0;

	FOR_I(155,10000)
	{
		srand(time(0) % 1234567);


//	work3();//test bsearch
//	return 0;

		cpu_work(l1,l2);

		work(l1,l2);
		cuda_copyResult();cout<<"copied back"<<endl;
		cu_checkError();
		free_device_memory();
		printf(" results / elements = %d / %d %lf\n", Lresult, n,(1.0 * Lresult) / (1.0 * n));

		//sort(resultList, resultList + Lresult);
		outln(Lresult);
		outln(cpuResultSize);
//		debug_a(resultList,Lresult+20);
//		debug_a(cpuResult,cpuResultSize);

		//continue;// DO NOT CHECK correctness

		int error_index;
		if ((error_index = check_correctness(cpuResultSize)) >= 0) {
			outline;
			outln(i);
			printf("all length: cpu:%d gpu:%d  ration: %.5lf\n", cpuResultSize,Lresult, 100.0 * error_index / (1.0 * Lresult));
			break;
		}
		outline;
		outline;
		break; //only execute once
	} //end FOR
	hipDeviceReset();
	return 0;
}

