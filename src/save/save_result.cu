#include "hip/hip_runtime.h"
#include "../common_defines.h"
#include "save_result.h"
#include "scan.cuh"
#include <cstdio>

extern int *scan_buffer;
extern hipStream_t *scan_stream;

	__device__ void save_result1(int *V,int indices_now){
			int _size = calculated_indices_len[indices_now][0];
			int id = CUID;
			int cusize = CUSIZE;
			int *list1 = list_p[indices_now][0];

			while (id <= _size ){
				int diff = V[id] - V[id-1];
				if ( diff >0){
					_result[ V[id-1] ] = list1[ id ];
				}
				id += cusize;
			}
		}

	__global__ void save_result(int *V,int indices_now){
		save_result1(V,indices_now);
	}


	inline __device__ void block_update_and_save(const int &id, int *V, int partial_addup,int indices_now){
		int *L1 = list_p[indices_now][0];

		if ( V[id] - V[id-1] > 0 ){
			_result[ V[id-1]+partial_addup ] = L1[ id ];
			//printf("in block_update_and_save (id:%d) %d is saved at [%d]\n",id,L1[ id ],V[id-1]+partial_addup);//debug
		}
	}

	__device__ void inner_move_result_addr(int *V,const int & indices_now){
		int _size = calculated_indices_len[indices_now][0];
		_result += V[_size];
	}

	__global__ void block_update_level2(int *buffer, int *V, int small_size,int indices_now){
		int *L1 = list_p[indices_now][0];
		int id = threadIdx.x;
		int idx = id * small_size ;
		int _size = calculated_indices_len[indices_now][0];

		if ( idx > _size ) return ;

		int a = buffer[ id];
		if ( V[idx] >0 ){
			_result[ a ] = L1[ idx ];
		}
		int part_id = _size / small_size;
		syncthreads();
		if ( part_id == id ){ // relative last part
			V[ _size ] += a;
			_result += V[ _size ];
		}
	}


	__global__ void update_and_save(int *buffer,int *V,int indices_now,int offset = 0){
		int _size = calculated_indices_len[indices_now][0];
		int partial_addup = buffer[ blockIdx.x ];
		int id = CUID + offset;

		if (id <= _size){
			block_update_and_save( id,V,partial_addup,indices_now);
		}
	}

	__global__ void move_result_addr(int *V,int indices_now){
		//inner_move_result_addr(V,indices_now);
	}

	void ScanSequence::run_save(int step){
		//dim3 threads(32,32);
		dim3 threads(small_size);
		int offset = step*step_size;
		int b_offset = offset / small_size;
		update_and_save<<< D1,small_size,0,*scan_stream>>>(scan_buffer+3 + b_offset,devL,save_indices_now,offset);
	}

	void ScanSequence::run_saveLarge(){
		block_update_level2<<<1,n/small_size,0,*scan_stream>>>(scan_buffer+3,devL,small_size,save_indices_now);
	}

	inline __device__ void one_block_compact(const int &compact_blocks,int *buffer,int *result,const int & save_idx){
		int id = threadIdx.x;
		int bsize = blockDim.x;
		int end;
		int *save_addr = _result_addr[save_idx];

		FOR_I( 0 ,compact_blocks){
			int j = buffer[i-1];
			end = buffer[i];
			int id2 = id;
			j += id;

			while ( j < end ){
				save_addr[ j ] = result[id2];
				j += bsize;
				id2 += bsize;
			}
			result += DEF_D2;
		}
		if ( 0 == id && blockIdx.x == gridDim.x -1 ){
			int next_idx = (save_idx + 1)%QUEUE_SIZE;
			//printf("End move: %d\n",end);//debug
			_result_addr[next_idx] = end + save_addr;
			_result = _result_addr[next_idx];//atmoic
		}
	}

	__global__ void compact(int compact_blocks,int *result,int save_idx){
		int offset1 = blockIdx.x * compact_blocks;
		int offset2 = offset1 * DEF_D2;
		one_block_compact(compact_blocks,_scan_buffers[save_idx] + offset1,result+offset2,save_idx);
	}




