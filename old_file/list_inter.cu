#include "hip/hip_runtime.h"
/*New feature:
	pre calculate indices is employed
	adaptive search is tried.( not pipeline yet )
	foolish adaptive
*/

#include <iostream>
#include <fstream>
#include <algorithm>
//#include <string>
//#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#include <unistd.h>
#include <sys/time.h>
//#include <memory.h>

#define PI acos(-1)
#define eps 1e-9

#define out(x) (cout<<#x<<":"<<x<<" ")
#define outln(x) (cout<<#x<<":"<<x<<endl)
#define outs(x) (cout<<x)
#define outline (cout<<endl)
#define mssleep(time) usleep((time)*(10*1000))

#define FOR_I(begin,end) for (int i=begin;i<end;i++)
#define FOR_J(begin,end) for (int j=begin;j<end;j++)
#define FOR_K(begin,end) for (int k=begin;k<end;k++)
#define FOR_I_J(B1,E1,B2,E2) FOR_I(B1,E1) FOR_J(B2,E2)
#define FOR_I_J_K(B1,E1,B2,E2,B3,E3) FOR_I_J(B1,E1,B2,E2) FOR_K(B3,E3)
#define FOR(begin,end) FOR_I(begin,end)
#define FORN(end) FOR_I(0,end)

#define SWAP(a,b) { a=(a)-(b);b=(b)+(a);a=(b)-(a);}

//------ for CUDA ------
#include <hip/hip_runtime.h>
//#include <cutil.h>
#include <hip/hip_runtime.h>
#define H_T_D hipMemcpyHostToDevice
#define D_T_H hipMemcpyDeviceToHost
#define D_T_D hipMemcpyDeviceToDevice
#define CUID (threadIdx.x + blockIdx.x*blockDim.x)
#define CUSIZE (blockDim.x*gridDim.x)

#define BANK_SIZE 32
#define BANK_SIZE_LOG 5

#define SHARE_OFFSET(id) ( (id) >> (BANK_SIZE_LOG))
using namespace std;

//for mpi
#define MCW MPI_COMM_WORLD

	template <typename T>
	void debug_a(T * data,int begin,int end){
		for (int i=begin;i<end;i++) cout<<"["<<i<<"]: "<<data[i]<<"\t";cout<<endl;
	}
	template <typename T>
	void debug_a(T * data,int end){
		debug_a(data,0,end);
	}
	template <typename T>
	void debug_a2(T * data,int end1,int end2){
		for (int i=0;i<end1;i++){cout<<"row "<<i<<endl; for (int j=0;j<end2;j++) cout<<"["<<i<<","<<j<<"] "<<data[i][j]<<"\t";cout<<endl;}
	}
	double get_sec(const struct timeval & tval){
		return ((double)(tval.tv_sec*1000*1000 + tval.tv_usec))/1000000.0;
	}

	template <typename T>
	T checkmin(T & data,T value){
		data = min(data,value);
		return data;
	}
struct Watch{
	timeval begin,end;
	void start(){gettimeofday(&begin,NULL);}
	double time(){return get_sec(end)-get_sec(begin);}
	double stop(){gettimeofday(&end,NULL);return time();}
};

	inline double rand_double(double range){
		return  ((double)rand()/(double)RAND_MAX)*range;
	}
//----------- cuda template ------------

struct CudaWatch{
	hipEvent_t start_event, stop_event;
	void start(){
	hipEventCreate(&start_event) ;
	hipEventCreate(&stop_event) ;
	hipEventRecord(start_event,0);
	}
	float stop(){
		float time_cost = -1.0;
		hipEventRecord(stop_event,0);
		hipEventSynchronize(stop_event);
		hipEventElapsedTime(&time_cost,start_event,stop_event);
		return time_cost;
	}
};
	void cu_checkError(){
		hipError_t error = hipGetLastError();
		if (error != hipSuccess){
			printf("ERROR: %s\n",hipGetErrorString(error));
			exit(-1);
		}
	}

	__global__ void cu_print(int *list,int size){
		printf("CUDA SHOW:\n");
		if ( 0 == threadIdx.x && 0 == blockIdx.x ){
			FOR_I(0,size)
				printf("[%d]:%d\t",i,list[i]);
			printf("\n");
		}
	}

	void cu_host_print(int *list,int size){
		hipDeviceSynchronize();
		cu_print<<<1,1>>>(list,size);
		hipDeviceSynchronize();
	}


//-------------------WRITE YOUR PROGRAM FROM HERE


const int N = 54100100;
const int M = 1024*1024;

int list[2][N];
int *devL1,*devL2;

bool v[N*10];

int n,m;

int P[M],V[M],V2[M];
int resultList[N];
int cpuResult[N],cpuResultSize;

int *devP,*devV,*devResult;

int L2index,L1index,Lresult;
int *devL2index,*devL1index,*devResultIndex;

int block_size;

const int QUEUE_SIZE= 5;

__device__ int *list_p0[2];
__device__ int *list_p[QUEUE_SIZE][2]; // save the starting position of either list
__device__ int calculated_length[QUEUE_SIZE];
__device__ int swapped[QUEUE_SIZE];   // save swapped stage for each status
__device__ int queue_size = QUEUE_SIZE;
__device__ int indices_now=0, search_now=0, prefix_now=0;
__device__ int _nm[2];
__device__ int *_result;

__device__ double ratio_avg;
__device__ int cal_times;

int * devIndices;

__shared__ int share[2048*2];

	void generate_different(int * array,int n,int mod){
		FOR_I(0,mod) v[i] = false;
		for (int i=0;i<n;i++){
			int j=-1;
			do
			{
				j = rand() % mod;
			}while ( v[j] );
			v[j] = true;
			array[i] = j;
		}
	}

	void generate_random(double alpha=1.0,double scala1=1.0,double scala2=1.0){
		int mod = n*4;
		m = (int)((double)n*alpha + eps);
		int mod1 = (int)((double)mod*scala1 + eps);
		int mod2 = (int)((double)mod*scala2 + eps);
		generate_different(list[0],n,mod1);
		sort(list[0],list[0]+n);
		generate_different(list[1],m,mod2);
		sort(list[1],list[1]+m);
	}

	void generate_case2(){
		int temp1[] = {1,7,8,9,12};
		int temp2[] = {3,5,7,9,11};
		n = m = 5;
		FOR_I(0,n) list[0][i] = temp1[i];
		FOR_I(0,n) list[1][i] = temp2[i];
	}

	void generate_case3(){
		int temp1[] = {1,7,8,9,12};
		int temp2[] = {13,15,17,19,21};
		n = m = 5;
			FOR_I(0,n) list[0][i] = temp1[i];
			FOR_I(0,n) list[1][i] = temp2[i];
	}

	void generate_case4(){
		int temp1[] = {1,7,8,11,12,  25,29,30,55,90};
		int temp2[] = {13,15,17,19,21, 22,25,30,33,34};
		n = m = 10;
		FOR_I(0,n) list[0][i] = temp1[i];
		FOR_I(0,n) list[1][i] = temp2[i];
	}

	void generate_case5(){
		int temp1[] = {1,7,8,19,20, 21,29,30,35,40};
		int temp2[] = {13,15,17,19,21, 24,25,30,33,40};
		n = m = 10;
		FOR_I(0,n) list[0][i] = temp1[i];
		FOR_I(0,n) list[1][i] = temp2[i];
	}

	void generate_case6(){
		int temp1[] = {13,15,17,19,21, 24,25,30,33,40};
		int temp2[] = {71,77,78,79,80, 91,92,93,94,95};
		n = m = 10;
		FOR_I(0,n) list[0][i] = temp1[i];
		FOR_I(0,n) list[1][i] = temp2[i];
	}

	void init_data(){
		int sizeV = 2*block_size + 32;
		int *temp = new int [sizeV];
		FOR_I(0,sizeV) temp[i] = 0;

		hipMalloc( (void **)&devL1, sizeof(int)*n );
		hipMalloc( (void **)&devL2, sizeof(int)*m );
		hipMalloc( (void **)&devV, sizeof(int)*sizeV );
		hipMalloc( (void **)&devP, sizeof(int)*block_size );
		hipMalloc( (void **)&devResult, sizeof(int)*n );

		hipMalloc( (void **)&devL2index, sizeof(int) );
		hipMalloc( (void **)&devL1index, sizeof(int) );
		hipMalloc( (void **)&devResultIndex, sizeof(int) );

		hipMemcpy( devL1,list[0],sizeof(int)*n,H_T_D );
		hipMemcpy( devL2,list[1],sizeof(int)*m,H_T_D );
		hipMemcpy( devV,temp,sizeof(int)*sizeV,H_T_D );
		hipMemcpy( devP,temp,sizeof(int)*block_size,H_T_D );
		devV = devV + 32 + block_size;
	}

	void show_lists(int index1,int index2,int len1=n,int len2=n){
		printf("Two lists:\n");
		printf("L1 :\t");debug_a(list[0]+index1,len1);
		printf("L2 :\t");debug_a(list[1]+index2,len2);
		outline;
	}

	void cuda_copyResult(){
		int *end;
		hipMemcpyFromSymbol(&end,HIP_SYMBOL(_result),sizeof(int *),0,D_T_H );
		Lresult = (int)(end - devResult );
		outln(Lresult);
		hipMemcpy( resultList,devResult, sizeof(int)*Lresult,D_T_H );
	}

	__device__ int next_queue_pos(int value){
			return (value+1) % queue_size;
	}

//----------------- prefix sum part ------------------
	__global__ void prefix_sum_cuda(int *list){
		int id = threadIdx.x;
		int block_size = blockDim.x << 1;
		int offset = blockIdx.x * block_size;
		list += offset;

		share[ 2*id + SHARE_OFFSET(2*id) ] = list[ 2*id ];
		share[ 2*id + 1 + SHARE_OFFSET(2*id+1) ] = list[ 2*id+1 ];
		//printf("%d --> %d\n",2*id,2*id + SHARE_OFFSET(2*id));

		//reduce stage
		int len = 1;
		int threads = 1;
		for (threads = blockDim.x ; threads >0  ;len <<=1,threads >>= 1 ){
			syncthreads();
			if ( id < threads ){
				int from = len -1 + 2*len*id;
				int addTo =  from + len;
				from += SHARE_OFFSET(from);
				addTo += SHARE_OFFSET(addTo);
				//list[ addTo ] += list[from];
				share[addTo] += share[from];
				//printf("id %d to[%d] %d\n",id,addTo,share[addTo]);
			}
		}
		//map stage
		len >>= 2;
		for ( threads = 2;len >=1 ;len >>=1,threads <<= 1 ){
			syncthreads();
			if ( id < threads -1 ){
				int from = 2*len -1 + 2*len*id;
				int addTo = from + len;
				from += SHARE_OFFSET(from);
				addTo += SHARE_OFFSET(addTo);
				//list[ addTo ] += list[from];
				share[addTo] += share[from];
			}
		}

			syncthreads();
		list[ 2*id ] = share[ 2*id + SHARE_OFFSET(2*id) ] ;
		list[ 2*id+1 ] = share[ 2*id + 1 + SHARE_OFFSET(2*id+1) ] ;
	}


	//add the value to all elements in list
	__global__ void addup(int *list,int *valueList){
		int id = threadIdx.x;
		int block_size = blockDim.x << 1;
		int offset = (blockIdx.x+1) * block_size;
		list += offset;
		// the length of list should be double of CUSIZE

		int value = *(valueList + blockIdx.x);
		int temp = id << 1;
		list[ temp ] += value;
		list[ temp+1 ] += value;
	}

	inline int block_multiple(int value,int block_size){
		if ( value % block_size != 0 )
			return block_size - value % block_size + value;
		return value;
	}

	//do prefix sum for inputed list
	void prefix_sum_algo(int *devList,int size, int block_size,int deep){
		int num_blocks = size / block_size;
		prefix_sum_cuda<<< num_blocks ,block_size/2>>>( devList );
		if ( num_blocks <=1 ) return;
		int size2 = block_multiple(num_blocks,block_size);
		//prepare data stage
			//ATTENTION this part is sequential part
		int i=0;
		int *nextList = devList - size2 - 1;
		for (i=0;i < num_blocks ;i++){
			int *dst = nextList + i;/// NOTATION from the end and a zero between
			int *src = devList + block_size*i + block_size - 1;
			hipMemcpy( dst, src, sizeof(int) , D_T_D );//only copy last element
		}
		for (;i < size2 ;i++){
			int *dst = nextList + i;
			int temp = 0;
			hipMemcpy(dst,&temp,sizeof(int),H_T_D);
		}
		//cal higher level block
		prefix_sum_algo(nextList,size2,block_size,deep+1);
		addup<<< num_blocks-1 ,block_size/2>>>(devList,nextList);
	}

	__global__ void prefix_sum_oneCore(int *V,int block_size){
		FOR_I(1,block_size)
			V[i] += V[i-1];
	}

//------------------ prefix sum over ---------------

	//binary search lower bound in list2 for each element in list1
	//algo2 should not care about swap or not
	__global__ void algo2_search(int * V){
		bool swap_now = swapped[indices_now];
		swap_now = false;
		bool swap_next = swapped[next_queue_pos(indices_now)];
		int *list1 = list_p[ indices_now ][ swap_now ];
		int *list2 = list_p[ indices_now ][ !swap_now ];
		//int *list2_end = list_p[ next_queue_pos(indices_now) ][ !swap_next ];

		int id = CUID;
		//if (!id) printf("now %d\n",indices_now);

		int block_size = CUSIZE;
		int left = 0;
		int right = calculated_length[indices_now] ;
		int value = list1[ id ];

			//if (!id)
		//printf("search %d in (%d <-- %d --> %d)\n",list1[id],list2[0],right+1,list2[right]);//debug
		V[id] = 0;
		while ( left <= right ){
			int mid = (left + right)/2;
			if ( value == list2[ mid ] ){
				V[ id ] = 1;
				return ;
			}
			else if ( list2[mid] < value )
				left = mid + 1;
			else right = mid - 1;
		}

		//printf("id: %d  %d get [%d] %d\n",id, value,left,list2[left]);
	}

	__global__ void save_result(int *V){
		int block_size = CUSIZE;
		int id = CUID;
		int *list1 = list_p[indices_now][0];

		if (V[id] - V[id-1] >0){
			_result[ V[id-1] ] = list1[ id ];
		}
	}

	__global__ void move_result(int block_size,int * V){
		//muse wait for others
		int id = CUID;
		if ( id == 0){
			_result += V[  block_size - 1 ];
		}
	}

	__global__ void init_device_var_kernel(){
		int id = CUID;
		if (id) return;
		list_p[0][0] = list_p0[0];
		list_p[0][1] = list_p0[1];
		swapped[0] = 0;//false
		ratio_avg = 0.0;//for debug
		cal_times = 0;//for debug
		indices_now = search_now = prefix_now= -1;
	}


	void init_device_variables(){
		hipMemcpyToSymbol(HIP_SYMBOL(list_p0),&devL1,sizeof(int *),0,H_T_D);
		cu_checkError();
		hipMemcpyToSymbol(HIP_SYMBOL(list_p0),&devL2,sizeof(int *),0+sizeof(int *),H_T_D);
		int temp[] = {n,m};
		hipMemcpyToSymbol(HIP_SYMBOL(_nm),temp,sizeof(int)*2,0,H_T_D);
		hipMemcpyToSymbol(HIP_SYMBOL( _result),&devResult,sizeof(int*),0,H_T_D);
		cu_checkError();
		init_device_var_kernel<<<1,1>>>();
		cu_checkError();
		hipDeviceSynchronize();
		hipMalloc( (void **)&devIndices , sizeof(int )*2);
	}


	//calculate the start point of next block ( pointer might overflow )
	// two threads calculate and write opposite lowerbound (each other)
	// id my index !id opposite index
	__global__ void calculate_indices(int block_size,int block_2_size,int isSwap){

		int id = CUID;
		if ( id > 1 ) return;

		int opposite = !id;

		if (0 == id)
			indices_now = (indices_now + 1) % queue_size;

		syncthreads();

		int temp_arr[] = {block_2_size,block_size};
		int left = 0;
		int right = temp_arr[id] - 1;

		int *list2 = list_p[indices_now][ opposite ];
		int value = list_p[indices_now][ id ][ temp_arr[opposite] -1];

		//printf("CI: id %d| [ %d ] %d search (%d %d)\n",id, (int)(list_p[indices_now][0] - list_p0[0]) , value,list2[left],list2[right]);//debug

		while ( left < right ){
			int mid = (left + right + 1)/2;
			if ( value < list_p[indices_now][ opposite ][mid] )
				right = mid - 1;
			else left = mid;
		}

		//printf("CI: id %d|   %d search to [%d] %d\n",id,value,left,list2[left]);//debug
		int next = next_queue_pos(indices_now);
		//list_p[next][ (0+isSwap) %2 ] = list_p[indices_now][0]+block_size;

		int next_opposite_index = (opposite)^isSwap;

		int next_offset = left+ ( list_p[indices_now][ opposite ][left] <= value );
		//Attention <= is the lower bound condition for edge condition
		list_p[next][ next_opposite_index ] = list_p[indices_now][ opposite ] + next_offset;

		if (0 == id){
			calculated_length[ indices_now ] = next_offset - 1;
			swapped[next] = swapped[indices_now] ^ isSwap;
		}
	}

	__global__ void helpIndex(int *indices,bool isNext){
		int temp = indices_now;
		if (isNext) temp = next_queue_pos(temp);
		indices[0] = list_p[temp][swapped[temp] ]-list_p0[0];
		indices[1] = list_p[temp][!swapped[temp] ]-list_p0[1];
	}

	__global__ void helpIndex_relative(int *indices,bool isNext){
		int temp = indices_now;
		if (isNext) temp = next_queue_pos(temp);
		indices[0] = list_p[temp][ 0 ]-list_p0[0];
		indices[1] = list_p[temp][ 1 ]-list_p0[1];
	}

	void back_next_relative_indices(int & index1,int & index2){
		int temp[2];
		helpIndex_relative<<<1,1>>>(devIndices,true);
		hipMemcpy(temp,devIndices,sizeof(int)*2, D_T_H);
		index1 = temp[0];index2 = temp[1];
	}

	// configure : <<<1,2>>> only and but only 2 threads
	__global__ void helpLen_relative(int *memory){
			int temp = next_queue_pos(indices_now);
			int id = threadIdx.x;
			int isSwapped = swapped[ temp ];
			int myindex = id ^ isSwapped;// id:1  !swapped

			memory[id] = (int)(list_p[temp][ id ]-list_p0[ myindex ]);
			memory[id] = _nm[myindex] - memory[id];

		}

	void back_next_relative_len(int & len1,int & len2){
		int temp[2];
		helpLen_relative<<<1,2>>>(devIndices);
		hipMemcpy(temp,devIndices,sizeof(int)*2, D_T_H);
		len1 = temp[0];len2 = temp[1];
	}

	void back_next_indices(int & index1,int & index2){
		int temp[2];
		helpIndex<<<1,1>>>(devIndices,true);
		hipMemcpy(temp,devIndices,sizeof(int)*2, D_T_H);
		index1 = temp[0];index2 = temp[1];
	}

	void back_indices(int & index1,int & index2){
		int temp[2];
		helpIndex<<<1,1>>>(devIndices,false);
		hipMemcpy(temp,devIndices,sizeof(int)*2, D_T_H);
		index1 = temp[0];index2 = temp[1];
	}

	void work(){
		int D1,D2,Dprefix;// Dprefix is the cuda dimension of prefix sum
		int block_2_size;

		//------ some settings ----------
		D1 = 64*4;
		D2 = 64*2;
		Dprefix = 4;

		block_size = D1 * D2;
		//block_size is only the length of looking list
		block_2_size = block_size*4;
		int block_sum = block_size + block_2_size;

		int switch_mode = 1;// foolish adaptive mode

		//------ END some settings --------
//		list[0][n++] = 1 <<10;list[1][m++] = 1<< 10;

		init_data();
		init_device_variables();

		L1index = L2index = 0;
//		n--;m--;

		int times=0;
		CudaWatch cudawatch;
		cudawatch.start();

		bool DEB_VP = false;

		int len1 = n,len2 = m;
		while (true){

			back_next_relative_len(len1,len2);
			//outln(len1);outln(len2);//debug
			if ( len1<=0 || len2 <= 0 ) break;

			int loops = 2*(min(len1,len2)-block_2_size)/block_sum;

			if ( loops >0 ){
				outln(loops);//debug
				FOR_I(0,loops){
					calculate_indices<<<1,2>>>(block_size,block_2_size,switch_mode);
					algo2_search<<<D1,D2>>>(devV);
					//prefix_sum_oneCore<<<1,1>>>(devV,block_size);
					//save_result<<<D1,D2>>>(devV);
					//move_result<<<1,1>>>(block_size,devV);

					//back_next_indices(L1index,L2index);//debug
					//printf("Next [%d] %d --> [%d] %d\n",L1index,list[0][L1index],L2index,list[1][L2index]);

					//back_indices(L1index,L2index);//debug
					//show_lists(L1index,L2index,10,10);//debug
					//getchar();//debug
					//TODO prefix sum
				}//end for
			}
			else {

				if ( len1 < 1024 ) {
					cout<<"Small segment:";out(len1);outln(len2);out(L1index);outln(L2index);
					calculate_indices<<<1,2>>>(len1,len2,switch_mode);
					algo2_search<<<1,len1>>>(devV);
					//prefix_sum_oneCore<<<1,1>>>(devV,len1);
					//save_result<<<1,len1>>>(devV);
					//move_result<<<1,1>>>(len1,devV);


				}
				else {
					//cout<<"go to 1024"<<endl;
					len2 = min(block_2_size,len2);
					calculate_indices<<<1,2>>>(32*32,len2,switch_mode);
					algo2_search<<<32,32>>>(devV);
					//prefix_sum_oneCore<<<1,1>>>(devV,32*32);
					//save_result<<<32,32>>>(devV);
					//move_result<<<1,1>>>(32*32,devV);

				}
			}

		}
		outln(cudawatch.stop()/1000.0);
	}

	void merge_algo(int *array1,int *array2, int begin1,int end1,int begin2,int end2){
		//return ;
		int i=begin1,j=begin2;
		int lasti,lastj;
		cpuResultSize = 0;
		lasti=array1[i];
		lastj=array2[j];
		while ( i<end1 && j<end2){
			if ( lasti == lastj ){
				//V2[i] = 1;
				cpuResult[cpuResultSize++] = lasti;
				i++;j++;
				lasti=array1[i];
				lastj=array2[j];
			}
			else if ( lasti < lastj){
				i++;
				lasti = array1[i];
			}
			else {
				j++;
				lastj = array2[j];
			}
		}
	}

	int check_correctness(){
			FOR_I(0,cpuResultSize)
				if (cpuResult[i] != resultList[i]){
					double x = (double)i / (double)cpuResultSize;
					printf("Wrong at [%d] R:%d Yours:%d   len:%lf\n",i,cpuResult[i],resultList[i],x);

					FOR_J(-10,10)
					printf("(%d,%d)",cpuResult[i+j],resultList[i+j]);

					return i;
				}
		return -1;
	}

	__global__ void algo_bsearch(
			int *list1,int *list2
			,int *V
			,int n,int m){
		int i = threadIdx.x + blockIdx.x*blockDim.x;
		while ( i < n){
			int value = list1[i];
			int left = 0,right = m-1;
			while (left <= right){
				int mid = (left+right)/2;
				int difference = value-list2[mid];
				if ( difference <0 )
					right = mid-1;
				else if ( difference >0 )
					left = mid +1;
				else {
					V[i] = 8;break;
				}
			}
			i += blockDim.x * gridDim.x;
		}
	}

	void work3(){
		init_data();
		CudaWatch cudawatch;
		cudawatch.start();
		//best achieved at 128,128 64, 32, 16
		// which is strange for 16 !
		algo_bsearch<<<32,512>>>(devL1,devL2,devResult,n,m);
		cout<<"NAIVE Bsearch:"<<cudawatch.stop()<<endl;
	}


	void test_indices(){
		int block_size = 64;
		int block_2_size = 128;
		n = 1024*1024;
		srand(45);
		generate_random();
		init_data();
		init_device_variables();

		//show_lists(50,50,70,70);

		int i;
		int t1,t2;
		for ( i=1;;i++ ){
			calculate_indices<<<1,1>>>(block_size,block_2_size,true);
			back_indices(t1,t2);
			printf("%d %d | %d %d\n",t1,t2, t1-i*block_size,t2-i*block_size);
			if ( n - max(t1,t2) < block_2_size )
				break;
		}
		outln(t1);outln(t2);
		outln(i);//debug

		hipDeviceSynchronize();
		double temp;
		int times;
		//hipMemcpyFromSymbol( &temp, HIP_SYMBOL("ratio_avg"), sizeof(double),0,D_T_H );
		hipDeviceSynchronize();
		//hipMemcpyFromSymbol( &times, HIP_SYMBOL("cal_times"), sizeof(int),0,D_T_H );
		cu_checkError();
		//printf(" %.1lf / %d = %lf\n",temp,times,temp/(1.0*times));
	}

	void test_prefix_sum(){
		n = 1024*2;
		generate_random();
		int m = n*2 +32;
		hipMalloc((void **)& devL1, sizeof(int)*m);
		hipMemcpy( devL1+n+32 ,list[0],sizeof(int)*n,H_T_D);
		hipMemcpy( devL1,list[0]+n,sizeof(int)*n,H_T_D);
		prefix_sum_algo(devL1+n+32,n,2,0);
		cu_checkError();
		hipDeviceSynchronize();
		cu_host_print(devL1,2*n);//debug
		hipDeviceSynchronize();
		hipMemcpy( list[1],devL1+n+32,sizeof(int)*n,D_T_H);
		FOR_I(1,n) {
			list[0][i] += list[0][i-1];
			if (list[0][i] != list[1][i]){
				cout<<"ERROR at"<<i<<endl;
				exit(123);
			}
		}

		cu_checkError();
		exit(0);
	}

int main(){
	//test_indices();return 0;
	//test_prefix_sum();

	//test_bsearch();return 0;

	int r;
	FOR_I(6,10000){
	r = rand() % 10013123 ;
	//r =1344532745 ;
	srand(i);
	n = 1024*1024*20;
	//n = 1024*16;

	//generate_case2();
	generate_random(2.0,2.0,1.0);


	cout<<"generate data over srand("<<i<<") n="<<n<<" m="<<m<<endl;
	printf("List 1 ( %d --- %d --- %d )\n",list[0][0],list[0][n/2],list[0][n-1]);
	printf("List 2 ( %d --- %d --- %d )\n",list[1][0],list[1][m/2],list[1][m-1]);

		//debug_a(list[0],n);
		//debug_a(list[1],n);

	memset(V2,0,sizeof(V2));
	memset(V,0,sizeof(V));

	Watch watch;watch.start();
	merge_algo(list[0],list[1],0,n,0,m);
	cout<<"CPU ALGO time: "<<watch.stop()<<endl;

	work();
	cu_checkError();

	work3();//test bsearch

	//cuda_copyResult();
	//outln(Lresult);outln(cpuResultSize);
	//debug_a(resultList,Lresult);
	cout<<"copied back"<<endl;
	//debug_a(cpuResult,10);
	//break;

	continue;

	int error_index;
	if  ( (error_index = check_correctness()) >=0 ){
		outline;
		outln(i);
		break;
	}
		outline;
		outline;
		outline;
		outline;

		//break; //debug
	}//end FOR
	return 0;
}
