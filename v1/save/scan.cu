#include "hip/hip_runtime.h"
#include "scan.cuh"
#include "../common_defines.h"
#include <assert.h>
#include <iostream>
#include <cstdio>

int *scan_buffer;
int SCAN_SMALL_SIZE = 2048;

hipStream_t *scan_stream;


	__device__ void scan_one(int *array,int n){
		int id = threadIdx.x;
		int id2 = id;
		__shared__ volatile int shared[ 2048*2  ];
		shared[ id ] = 0;
		id += n;
		shared[ id ] = array[ id2 ];
		for (int len = 1; len < n;len <<= 1){
			syncthreads();
			int a = shared[ id ] + shared[ id-len ];
			syncthreads();
			shared[ id ] = a;
		}
		array[ id2 ] = shared[id];
	}

	__global__ void scan_large(int *array,int *buffer,int blocks,int block_n){
		int id = threadIdx.x;
		int start = block_n-1;
		buffer[id] = array[start + id*block_n];
		scan_one(buffer,blocks);
	}









