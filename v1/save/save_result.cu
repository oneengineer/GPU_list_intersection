#include "hip/hip_runtime.h"
#include "../common_defines.h"
#include "save_result.h"
#include "scan.cuh"
#include <cstdio>

extern int *scan_buffer;
extern hipStream_t *scan_stream;

	inline __device__ void one_block_compact(const int &compact_blocks,int *buffer,int *result,const int & save_idx){
		int id = threadIdx.x;
		int bsize = blockDim.x;
		int end;
		int *save_addr = _result_addr[save_idx];

		FOR_I( 0 ,compact_blocks){
			int j = buffer[i-1];
			end = buffer[i];
			int id2 = id;
			j += id;

			while ( j < end ){
				save_addr[ j ] = result[id2];
				j += bsize;
				id2 += bsize;
			}
			result += DEF_D2;
		}
		if ( 0 == id && blockIdx.x == gridDim.x -1 ){
			int next_idx = (save_idx + 1)%QUEUE_SIZE;
			//printf("End move: %d\n",end);//debug
			_result_addr[next_idx] = end + save_addr;
			_result = _result_addr[next_idx];//atmoic
		}
	}

	__global__ void compact(int compact_blocks,int *result,int save_idx){
		int offset1 = blockIdx.x * compact_blocks;
		int offset2 = offset1 * DEF_D2;
		one_block_compact(compact_blocks,_scan_buffers[save_idx] + offset1,result+offset2,save_idx);
	}




