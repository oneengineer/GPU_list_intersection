#include "hip/hip_runtime.h"
#include "../common_defines.h"
#include "search.cuh"
#include <cstdio>


#if DEPLETED

	//binary search lower bound in list2 for each element in list1
	//algo2 should not care about swap or not


	__device__ void search_1(int * V,int search_now,int offset){
		int id = CUID;
		id += offset;
		if (id > calculated_indices_len[search_now][0]){
//			printf("id %d quit!\n",id);
			return;
		}
		int left = 0;
		int right = calculated_indices_len[search_now][1];

		int *list1 = list_p[ search_now ][ 0 ];
		int *list2 = list_p[ search_now ][ 1 ];
		int value = list1[ id ];
//		if(!id) { printf("R:%d\n",right) ;FOR_I(0,right+1) printf("!%d\t",list2[i]); printf("\n"); }
		int result = 0;
		while ( left <= right ){
			int mid = (left + right)/2;
			if ( value == list2[ mid ] ){
//				printf("id: %d INTERSECTION %d\n",id,value);
				result = 1;
				break;
			}
			else if ( list2[mid] < value )
				left = mid + 1;
			else right = mid - 1;
		}
		V[id] = result;
//		printf("  algo2 search id: %d  %d get [%d] %d\n",id, value,right,list2[right]);
	}

	//Too many command
	__global__ void gallop_search_stage2(int *devL1,int *devL,int n){
		int id = threadIdx.x;
		int value = devL1[id];
		int pos = id;
		int left = 0,right = n;
		int bound;
		int result = 0;
		int len;
		//printf("id:%d get value:%d\n",id,value);
		int value2;

		int debugid = 20;
		int add_to_len;

		bool direction;
		//decide direction
		if ( devL[pos] <= value ) // right -->> direction
			direction = true,bound = right,len=1,add_to_len=0;
		else direction = false,bound = left,len=-1,add_to_len=1;
		printf(" (%d) value: %d  direction:%d  len:%d\n",id,value,direction,len);

		//len increase only one direction
		int pos2;
		while ( true ){
			value2 = devL[pos];
			if (id == debugid) printf(" compared with [%d]:%d,it move to %d for next time %d\n",pos,value2,pos,len);
			if ( value2 == value ) break;
			else if ( direction == ( value2 < value ) ){
				pos2 = pos + len;
			}
			else {
				len = (len+add_to_len)>>1;
				break;}
			if (!( left <= pos2 && pos2 <=right )) break;
			pos = pos2;
			len <<= 1;
		}

		if (!id) printf(" stage 1 over\n");
		printf(" (%d) value: %d  pos:%d  len:%d\n",id,value,pos,len);

		//len = 1;
		//len decrease, possible two directions
		while ( len!=0 ){
			len = (len+add_to_len)>>1;
			if (id == debugid) printf(" compared with %d,it move to %d for next time %d\n",value2,pos,len);
			if ( value2 == value ){
				result = 1;
				//add a break?
				break;
			}
			else if ( direction == (value2 < value )){
				pos += len;
			}
			else pos -= len;

			value2 = devL[pos];
		}

		if ( result )
			printf("(%d)  found %d at [%d]\n",id,value,pos);
		syncthreads();
		devL1[id] = result;
	}

	void test_gallop_search(){
		n = 1001;
		generate_random(1.0,1,1);
		//FOR_I(4,n)
//		host_lists[0][i] *=10;
		//host_lists[0][1] = 33;
		//host_lists[0][0] = 17;
		//generate_same(2);
		//generate_case3();
		debug_a(host_lists[0],n);outline;debug_a(host_lists[1],m);//debug
		init_data(128);
		init_device_variables();
		hipDeviceSynchronize();
		HERE;
		//n = 32;

		int * cpuresult = new int[n];
		FOR_I(0,n){
			cpuresult[i] = 0;
			FOR_J(0,n)
			if ( host_lists[0][i] == host_lists[1][j] ){
				cpuresult[i] = 1;
				break;
			}
		}
		gallop_search<<<1,n>>>(devL1,devL2,n-1);
		hipMemcpy(resultList,devL1,sizeof(int)*n,D_T_H);

		hipDeviceSynchronize();

		debug_a(resultList,n);
		debug_a(cpuresult,n);

		cu_checkError();

		FOR_I(0,n)
			if ( resultList[i] != cpuresult[i] ){
				printf("WRONG AT [%d]:%d\n",i,host_lists[0][i]);
				exit(0);
			}
	}

#endif



	//extern __device__ void scan_a_block_neat(int * src_data,volatile int shared[][WARP_SIZE*2]);

	template<int loops,bool exclusive>
	inline __device__ void scan_warp_neet(volatile int * shared,int value,int id){
		shared[id] = 0;
		id += WARP_SIZE;
		shared[id] = value;

		if ( 1 <= loops ) shared[id] += shared[id - 1];
		if ( 2 <= loops ) shared[id] += shared[id - 2];
		if ( 3 <= loops ) shared[id] += shared[id - 4];
		if ( 4 <= loops ) shared[id] += shared[id - 8];
		if ( 5 <= loops ) shared[id] += shared[id - 16];

		if ( exclusive )
			shared[id] -= value;
	}

	__device__ void scan_a_block_neat(volatile int * src_data,volatile int shared[][WARP_SIZE*2]){
		int id = threadIdx.x;
		int local_id = id&(WARP_SIZE-1);
		int warp_num = id >> LOG_WARP_SIZE ;
		volatile __shared__ int shared_level2[WARP_SIZE*2];

		//printf("id:%d src: %llx  shared:%llx\n",id,src_data+id,shared[warp_num]+id);//debug
		scan_warp_neet< LOG_WARP_SIZE,false>(shared[warp_num],src_data[id],local_id );//basic level
		syncthreads();
		if ( id < WARP_SIZE){
			int myvalue = shared[id][WARP_SIZE+WARP_SIZE-1];
			scan_warp_neet<LOG_SCAN_BLOCK_SIZE - LOG_WARP_SIZE ,true>(shared_level2,myvalue,id);//second level, use calculated 32,in one block
		}
		syncthreads();
		//uniform update
		int a = shared_level2[ warp_num+WARP_SIZE ];
		src_data[id] = shared[warp_num][ local_id + WARP_SIZE ] + a;
	}

	/**
	 *
	 * To write the program use minimum command
	 * Three things decide the next position:
	 * 1. position out of boundary?
	 * 2. get value, direction? change or not?
	 * 3. length? how l
	 * L: position = what
	 *
	 *
	 * value: the value to search
	 * pos2: starting place, extrapolated position, it need to be return!
	 * return:result 0/1
	 *
	 */

	inline __device__ int gallop_search (int value,int & pos2,volatile int *list,int left,int right){

		int pos = pos2;
		int result = 0;
		int len;
		int value2;

		char last_direction = 3;
		char direction = 0;
		char start_decrease = 0;

		len = 1;
		while ( len > 0){

			if (!( left <= pos2 && pos2 <= right )){
				start_decrease = 1;
			}
			else{
				value2 = list[pos2] - value;
				if ( 0 == value2 ){
					result = 1;break;
				}
				else direction = value2 < 0;
				start_decrease |= (direction + last_direction) ==1;// 0 + 1 or 1 + 0
				pos = pos2;
				last_direction = direction;
			}
			if ( start_decrease )
				len >>=1;
			else len <<=1;

			if ( direction )
				pos2 = pos + len;
			else pos2 = pos - len;
		}
		return result;
	}

	inline __device__ int binary_search(int value,int & pos,volatile int *list,int left,int right){
		int result = 0;
		while ( left <= right ){
			if ( value == list[ pos ] ){
				//printf("block:%d thread: %d INTERSECTION %d\n",blockIdx.x,threadIdx.x,value);
				result = 1;
				break;
			}
			else if ( list[pos] < value )
				left = pos + 1;
			else right = pos - 1;
			pos = (left + right)/2;
		}
		return result;
	}

	/*
	 * because the search_2 is done in one SM in GPU, no divergence occur
	 * begin and end are assigned from blockIdx, one block only have on value => no divergence
	 *
	 */
	inline __device__ int search_one_value(int value,int & pos,volatile int *opposite_addr,const int &left,const int &right){

		int result = binary_search(value,pos,opposite_addr,left,right);
		//printf("<%d,%d>search %d in (%d %d) to %d\n",blockIdx.x,threadIdx.x,value,left,right,pos);//debug
//
		if (result){
			int write_pos = atomicAdd(&gpu_result_size,1);
			_result[write_pos] = value;
		}

//		if (result)
//		printf("B<%d> %d was found at [%d]\n",blockIdx.x,value,pos);

		return result;
	}

	/*
	 * search 4 integer stored in uint4, a,b,c,d
	 * use divide-and-conquer way a,c,b,d order search
	 *
	 */
	inline __device__ void search_uint4(const int & id,const uint4 &myvalue,volatile int *opposite_list,
			volatile int *shared_range,const struct partition_info &info,
			int &mask,int &uint4_sum){
		int pos_A,pos_X,pos_C,pos_D;
		//pos_A = (info.left + info.right)/2;
		pos_A = id;
		mask = 0,uint4_sum=0;
		if (search_one_value(myvalue.x,pos_A,opposite_list,info.left,info.right)){
			mask |=1,uint4_sum ++;
		}
		shared_range[id] = pos_A;
		if ( pos_A <0 || pos_A > info.right ) printf("!!!");
		syncthreads();
		int id_addone = id + 1;
		if ( id_addone != info.len )
			pos_D = shared_range[id_addone];
		else pos_D = info.right;

		pos_C = (pos_A + pos_D) /2 ;
		if (search_one_value(myvalue.z,pos_C,opposite_list,pos_A,pos_D)){
			mask |=4,uint4_sum ++;
		}
		if ( pos_C <0 || pos_C > info.right ) printf("!!!");
		pos_X = (pos_A + pos_C) /2 ;
		if (search_one_value(myvalue.y,pos_X,opposite_list,pos_A,pos_C)){
			mask |=2,uint4_sum ++;
		}
		if ( pos_X <0 || pos_X > info.right ) printf("!!!");
		pos_X = (pos_C + pos_D) /2 ;
		if (search_one_value(myvalue.w,pos_X,opposite_list,pos_C,pos_D)){
			mask |=8,uint4_sum ++;
		}
		if ( pos_X <0 || pos_X > info.right ) printf("!!!");
	}

	/*
	 * Simple method, not use shared
	 * */
//	inline __device__ void search_uint4_2(const int & id,const uint4 &myvalue,volatile int *opposite_list,
//			volatile int *shared_range,const struct partition_info &info,
//			int &mask,int &uint4_sum){
//		int pos_A,pos_X,pos_D;
//		pos_A = id;
//		mask = 0,uint4_sum=0;
//		if (search_one_value(myvalue.x,pos_A,opposite_list,info.left,info.right)){
//			mask |=1,uint4_sum ++;
//		}
//		pos_D =  ( pos_A+ info.right)/2;
//		if (search_one_value(myvalue.z,pos_D,opposite_list,pos_A,info.right)){
//			mask |=8,uint4_sum ++;
//		}
//		pos_X = (pos_A + pos_D) /2 ;
//		if (search_one_value(myvalue.y,pos_X,opposite_list,pos_A,pos_D)){
//			mask |=2,uint4_sum ++;
//		}
//		if (search_one_value(myvalue.w,pos_X,opposite_list,pos_X,pos_D)){
//			mask |=4,uint4_sum ++;
//		}
//	}

	inline __device__ void scan_and_save_buffer(volatile int *scan_array,volatile int *scan_shared,int * save_buffer,
			int id,uint4 &myvalue,int &mask,int &uint4_sum,const struct partition_info & info){
		scan_array[id] = uint4_sum;
		//printf("<%d %d> position:%d %d\n",blockIdx.x,threadIdx.x,scan_array[id],info.len);
		syncthreads();

//		if (!id){
//			FOR_I(0,blockDim.x)
//				printf("[%llx] %d\t",scan_array+i,scan_array[i]);
//			printf("\n");
//			FOR_I(0,blockDim.x)
//				printf("[%llx] %d\t",scan_shared+i,scan_shared[i]);
//			printf("\n");
//		}
		scan_a_block_neat((int *)scan_array,(int (*)[WARP_SIZE*2])scan_shared);

//		if (mask & 1)
//				printf("B<%d> %d was found at\n",blockIdx.x,myvalue.x);
//		if (mask & 2)
//				printf("B<%d> %d was found at\n",blockIdx.x,myvalue.y);
//		if (mask & 4)
//				printf("B<%d> %d was found at\n",blockIdx.x,myvalue.z);
//		if (mask & 8)
//				printf("B<%d> %d was found at\n",blockIdx.x,myvalue.w);


		int position = scan_array[id] - uint4_sum; // inclusive scan
		//printf("<%d %d> position:%d\n",blockIdx.x,threadIdx.x,position);

//		if ( mask & 1 )
//			save_buffer[position] = myvalue.x;
//		if ( mask & 2 )
//			save_buffer[position+1] = myvalue.y;
//		if ( mask & 4 )
//			save_buffer[position+2] = myvalue.z;
//		if ( mask & 8 )
//			save_buffer[position+3] = myvalue.w;

	}

	inline __device__ void check_address(const struct partition_info &info,int * shared,uint4 myvalue){
		int *abs_add1,*abs_opposite;
		bool flag1 = false;
		if ( info.addr - list_p0[0] <0 && info.opposite_addr - list_p0[1] <0 ){
			printf("ONLY APPEAR ONCE\n");
		}

		if ( info.addr - list_p0[0] >=0 && info.opposite_addr - list_p0[1] >=0 ){
			abs_add1 = list_p0[0];
			abs_opposite = list_p0[1];
		}
		else {
			abs_add1 = list_p0[1];
			abs_opposite = list_p0[0];
			flag1 = true;
		}
		//FOR_I(0,4*info.len){
			LL offset = info.addr + (LL)(4*threadIdx.x) - abs_add1;
/*			if ( offset <0 || offset >= _nm[0] + 128 *  8*32*4 ){
				atomicAdd(&debug1.wrong_2,1);
				return;
			}*/
			atomicAdd(&debug1.wrong_2,myvalue.x);
			return;
			if ( myvalue.y != abs_add1[offset+1] ){
				atomicAdd(&debug1.wrong_2,1);
			}
		//}

//		FOR_I(0,4*info.len_opposite){
//			LL offset = info.opposite_addr + (LL)i - abs_opposite;
//
//			if ( abs_opposite[offset] != shared[i] ){
//				atomicAdd(&debug1.wrong_1,1);
//				//printf("wrong copy %d %d\n",info.opposite_addr[offset],shared[i]);
//			}
//			if (335544314 == abs_opposite[offset] || 335544319 == abs_opposite[offset]){
//				printf("END : %d \n",offset);
//			}
//
//			if ( offset <0 || offset>= _nm[0] + 128 *  8*32*4 )
//				printf("wrong %d offset",offset-_nm[0]);
//		}

	}

	inline __device__ void brute_force(uint4 & myvalue,volatile int * list,const struct partition_info &info){
		FOR_I(info.left,info.right+1){
			if ( myvalue.x == list[i] ){
				int write_pos = atomicAdd(&gpu_result_size,1);
				_result[write_pos] = myvalue.x;
			}
			if ( myvalue.y == list[i] ){
				int write_pos = atomicAdd(&gpu_result_size,1);
				_result[write_pos] = myvalue.y;
			}
			if ( myvalue.z == list[i] ){
				int write_pos = atomicAdd(&gpu_result_size,1);
				_result[write_pos] = myvalue.z;
			}
			if ( myvalue.w == list[i] ){
				int write_pos = atomicAdd(&gpu_result_size,1);
				_result[write_pos] = myvalue.w;
			}
		}
	}

	inline __device__ void search_in_block(int * V,int search_now,const struct partition_info &info){
		int id = threadIdx.x;
		// begin and end is closed interval [begin , end]

		volatile __shared__ int opposite_list[10*32*4];
		//volatile __shared__ int shared_range[1024];// only use once, the right most thread does not use it
		int mask,uint4_sum;
		uint4 myvalue ;
		bool flag1 = false;
		/*
		 * shared_range __shared__ use as src_data for scan
		 * opposite_list __shared__ use as
		 *  */

		// the thread which should copy opposited list elements

		if (id < info.len_opposite){

			myvalue = ((uint4 *)info.opposite_addr)[id];
			//atomicAdd(&debug1.wrong_1,myvalue.w);
			((uint4 *)opposite_list)[id] = myvalue;
			if ( id < info.len ){
				myvalue = ((uint4 *)info.addr)[id];
				//atomicAdd(&debug1.wrong_2,myvalue.w);
				flag1 = true;
				//search_uint4(id,myvalue,opposite_list,shared_range,info,mask,uint4_sum);
			}
		}
		syncthreads();

//		if ( !threadIdx.x ){
//			FOR_I(0,info.len_opposite)
//				printf("%d\t",info.opposite_addr[i]);
//			FOR_I(0,info.len_opposite)
//				printf("%d\t",info.opposite_addr[i]);
//		}

		if (flag1){
			//check_address(info,(int *)opposite_list,myvalue);
			brute_force(myvalue, info.opposite_addr ,info);
			//search_uint4(id,myvalue,opposite_list,shared_range,info,mask,uint4_sum);
		}
		//

		//if ( id < info.warp_len )
		//scan_and_save_buffer(shared_range,opposite_list,V,id,myvalue,mask,uint4_sum,info);
	}

	__global__ void algo2_search(int * V,int search_now,int offset){
		if (partitions_info[search_now][blockIdx.x].len>0){
			search_in_block(V,search_now,partitions_info[search_now][ blockIdx.x ]);
		}
	}

#if DEPLETED

	template<bool use1>
	__device__ void search_2(int * V,int search_now,int offset,int begin1,int end1,int begin2,int end2){
		int id = threadIdx.x;
		// begin and end is closed interval [begin , end]

		int *list1 ;
		int *list2 ;
		int value;

		volatile __shared__ int opposite_list[2048];

		int left(0),right;
		if ( use1 ){
			list2 = list_p[ search_now ][ 1 ];
			opposite_list[ id ] = list2[ begin2 + id ];
			if ( begin1 + id > end1 ) return;
			list1 = list_p[ search_now ][ 0 ];
			//left = begin2;
			//right = end2;
			right = end2 - begin2;
			value = list1[ begin1 + id ];
		}
		if ( !use1 ){
			list2 = list_p[ search_now ][ 0 ];
			if ( begin1 + id <= end1 ){
				opposite_list[ id ] = list2[begin1 + id];
				V[ begin1 + id ] = 0;
			}
			if ( begin2 + id > end2 ) return;
			list1 = list_p[ search_now ][ 1 ];
			//left = begin1;
			//right = end1;
			right = end1 - begin1;
			value = list1[ begin2 + id ];
		}
		syncthreads();

//		if ( !id )printf("block:%d flag:%d { %d  -> %d } [%d] -- [%d]\n",
//					blockIdx.x,use1,end1-begin1,end2 - begin2,list_p[ search_now ][ 0 ][end1],list_p[ search_now ][ 1 ][end2]);//debug


		int result = 0;
		int pos = id;
		//result = binary_search(value,pos,opposite_list,left,right);
		result = gallop_search(value,pos,opposite_list,left,right);


		if (use1){
			//printf("(%d,%d) set V[%d] %d to zero \n",blockIdx.x,threadIdx.x,begin1 + id,V[ begin1 + id ]);
			V[ begin1 + id ] = result;
			if (result){
				//printf("(%d,%d) set V[%d] %d to 1\n",blockIdx.x,threadIdx.x,begin1 + id,value);
			}
		}
		if (!use1){
			//printf("(%d,%d) set V[%d] %d to real_zero \n",blockIdx.x,threadIdx.x,begin1 + id,V[ begin1 + id ]);
			//V[ begin1 + id ] = 0;
			if (result){
				V[ begin1 + pos ] = 1;
				//printf("(%d,%d) set oV[%d] %d to 1\n",blockIdx.x,threadIdx.x,begin1 + mid,value);
			}
		}

	}

	__global__ void algo2_search(int * V,int search_now,int offset){
		//search_1(V,search_now,offset);
		int begin1 = partitions[search_now][blockIdx.x][0]+1;
		int end1 = partitions[search_now][blockIdx.x+1][0];
		int begin2 = partitions[search_now][blockIdx.x][1]+1;
		int end2 = partitions[search_now][blockIdx.x+1][1];
		if (end1 - begin1 <= end2 - begin2){
			search_2<true>(V,search_now,offset,begin1,end1,begin2,end2);
		}
		else {
			search_2<false>(V,search_now,offset,begin1,end1,begin2,end2);
		}
	}
#endif




