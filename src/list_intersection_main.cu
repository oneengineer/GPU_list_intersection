#include "hip/hip_runtime.h"
/*New feature:
	pre calculate indices, smart adaptive, fully pipeline, not fully hidden
	cudpp have streams, but number of device block/threads can not be controlled
	cudpp prefix sum for my algo
	pipeline ( cudpp pipelined) is employed
	pipeline ( myscan pipelined) is employed
*/

#include <iostream>
#include <fstream>
#include <algorithm>
#include <string>
#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#include <unistd.h>
#include <sys/time.h>

//------ for CUDA ------
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "common_defines.h"
#include "common_functions.h"
#include "others/bsearch.cuh"
#include "others/memory_data.cuh"
#include "indices/cal_indices.cuh"
#include "search/search.cuh"
#include "save/scan.cuh"
#include "save/save_result.h"

#include "others/generate_data.h"

//#include "cudpp.h"


using namespace std;

	template <typename T>
	void debug_a(T * data,int begin,int end){
		for (int i=begin;i<end;i++) cout<<"["<<i<<"]: "<<data[i]<<"\t";cout<<endl;
	}
	template <typename T>
	void debug_a(T * data,int end){
		debug_a(data,0,end);
	}
	template <typename T>
	void debug_a2(T * data,int end1,int end2){
		for (int i=0;i<end1;i++){cout<<"row "<<i<<endl; for (int j=0;j<end2;j++) cout<<"["<<i<<","<<j<<"] "<<data[i][j]<<"\t";cout<<endl;}
	}
	double get_sec(const struct timeval & tval){
		return ((double)(tval.tv_sec*1000*1000 + tval.tv_usec))/1000000.0;
	}

	template <typename T>
	T checkmin(T & data,T value){
		data = min(data,value);
		return data;
	}
struct Watch{
	timeval begin,end;
	void start(){gettimeofday(&begin,NULL);}
	double time(){return get_sec(end)-get_sec(begin);}
	double stop(){gettimeofday(&end,NULL);return time();}
};

	inline double rand_double(double range){
		return  ((double)rand()/(double)RAND_MAX)*range;
	}
//----------- cuda template ------------

struct CudaWatch{
	hipEvent_t start_event, stop_event;
	void start(){
	hipEventCreate(&start_event) ;
	hipEventCreate(&stop_event) ;
	hipEventRecord(start_event,0);
	}
	float stop(){
		float time_cost = -1.0;
		hipEventRecord(stop_event,0);
		hipEventSynchronize(stop_event);
		hipEventElapsedTime(&time_cost,start_event,stop_event);
		return time_cost;
	}
};
	void cu_checkError(){
		hipError_t error = hipGetLastError();
		if (error != hipSuccess){
			printf("ERROR: %s\n",hipGetErrorString(error));
			exit(-1);
		}
	}

	__global__ void cu_print(int *list,int size){
		printf("CUDA SHOW:\n");
		if ( 0 == threadIdx.x && 0 == blockIdx.x ){
			FOR_I(0,size)
				printf("[%d]:%d\t",i,list[i]);
			printf("\n");
		}
	}

	void cu_host_print(int *list,int size){
		hipDeviceSynchronize();
		cu_print<<<1,1>>>(list,size);
		hipDeviceSynchronize();
	}


//-------------------WRITE YOUR PROGRAM FROM HERE



int *host_lists[2];

int *devL1,*devL2;
bool *v;
int *devV[3],*devResult, *devMark;
int * devIndices;
int n,m;

int Lresult;
int *resultList;
int *cpuResult;

__device__ int *list_p0[2];// the origional   address
__device__ int *list_p[QUEUE_SIZE][2]; // save the starting position of either list
__device__ int calculated_indices_len[QUEUE_SIZE][4];
__device__ int swapped[QUEUE_SIZE];   // save swapped stage for each status
__device__ int *_result;
__device__ int _nm[2];
__device__ int partitions[QUEUE_SIZE][256+4][2];

	inline void move_pos(int &pos){
		pos = (pos + 1) % QUEUE_SIZE;
	}

//	CUDPPHandle prepare_prefixsum(int size, hipStream_t * stream){
//		CUDPPConfiguration config;
//		config.op = CUDPP_ADD;
//		config.datatype = CUDPP_INT;
//		config.algorithm = CUDPP_SCAN;
//		config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;
//		config.mystream = stream;
//
//		CUDPPHandle theCudpp;
//		cudppCreate(&theCudpp);
//		CUDPPHandle scanplan = 0;
//		CUDPPResult res = cudppPlan(theCudpp,&scanplan,config,size,1,0);
//		if ( CUDPP_SUCCESS != res ){
//			printf("ERROR in prepare_prefixsum\n");
//			exit(-1);
//		}
//		return scanplan;
//	}

	__global__ void save_one_core(int search_now,int *devV){
		int num = 0;
		int len = calculated_indices_len[search_now][0];
		FOR_I(0,len+1)
			if ( devV[i] ){
				_result[ num++ ] = list_p[search_now][0][i];
				//printf("in save : saved :%d\n",list_p[search_now][0][i]);//debug
			}
		_result += num;
	}

	__global__ void help_show1(int indices_now){
		printf("Moved %d %d\n",calculated_indices_len[indices_now][0],calculated_indices_len[indices_now][1]);

	}

	void work(){
		int numStream;
		numStream = 4;
		hipStream_t *streams = (hipStream_t *) malloc(numStream * sizeof(hipStream_t));
		for (int i = 0; i < numStream; i++)
		{
			hipStreamCreate(&(streams[i]));
		}

		int D1,D2,D1save;
		int block_size;
		int block_2_size;
		//------ some settings ----------
		D1save = 16;
		D1 = 128;
		D2 = 512;

		int save_stream = 2;
		int search_stream = 1;

		block_size = D1 * D2;
		block_2_size = block_size;
		//block_2_size = block_size*1000;

		init_data(block_size);
		init_device_variables();
		init_scan(&streams[save_stream],1024);
		ScanSequence scanSeq(1,devV[0],block_size);
		scanSeq.init();

		SearchSettingQueue searchQue(2,&(streams[search_stream]));
		int searchConfig[][2] = { {64,512},{64,512},{64,512},{64,512} };
		int searchConfig8[][2] = { {32,512},{32,512},{32,512},{32,512},{32,512},{32,512},{32,512},{32,512} };
		int searchConfig1[][2] = { {256,512}};
		int searchConfig2[][2] = { {128,512},{128,512}};
		int searchConfig_small[][2] = { {1,16}};
		searchQue.setSettings(searchConfig2);
		searchQue.init();
		outln(searchQue.length());

		int num_small_block = D1;
		dim3 cal_indx_setting(num_small_block,2);

		CudaWatch cudawatch;
		Watch cpuWatch;cpuWatch.start();
		cudawatch.start();

		//CUDPPHandle prefixsum_plan = prepare_prefixsum(block_size,streams+2);

		int len1,len2;
		int cal_pos = 0, search_pos = 0, save_pos  =0; // they are for L1, L2 position
		while (true){
			int devVinc = 0; // use which devV to store data
			back_next_relative_len(len1,len2,cal_pos);
			cu_checkError();
			outln(len1);outln(len2);//debug
			if ( len1<=0 || len2 <= 0 ) break;
			int loops = min(len1,len2)/block_size;
			if ( min(len1,len2) % block_size != 0 )
				loops ++;

			if ( loops >0 ){
				outln(loops);
				bool lastButOne = loops > 1;
				int *saveV; //pointer of saving result

				//-- stage middle
				for ( ;loops> 0 ;loops -- ){
					//outln(loops);
					saveV = devV[ devVinc];
					cal_indx<<<1,cal_indx_setting>>>( block_size,block_2_size,cal_pos);
					//hipDeviceSynchronize();
					algo2_search<<< 2 * D1 , D2 >>>(saveV,search_pos,0);
					//cu_host_print(saveV,block_size);//debug
					//save_one_core<<<1,1>>>(search_pos,devV[0]);
					scanSeq.set(saveV,save_pos);
					scanSeq.run_all();
					//cu_host_print(saveV,block_size);//debug

					//cu_checkError();
//					if ( loops < 10)
//						help_show1<<<1,1>>>(save_pos);
					move_pos(cal_pos);move_pos(search_pos);move_pos(save_pos);


					//mssleep(10);
					//hipDeviceSynchronize();
					//return ;
				}
				hipDeviceSynchronize();
			}

			cu_checkError();
		}
		hipDeviceSynchronize();
		cout<<"MY Algo:"<<cudawatch.stop()/1000.0<<endl;
		cout<<"MY Algo cpu test: "<<cpuWatch.stop()<<endl;
	}

int cpuResultSize = 0;
	int merge_algo(int *array1,int *array2, int begin1,int end1,int begin2,int end2){
		cpuResultSize = 0;
		int i=begin1,j=begin2;
		int lasti,lastj;

		lasti=array1[i];
		lastj=array2[j];
		while ( i<end1 && j<end2){
			if ( lasti == lastj ){
				cpuResult[cpuResultSize++] = lasti;
				i++;j++;
				lasti=array1[i];
				lastj=array2[j];
			}
			else if ( lasti < lastj){
				i++;
				lasti = array1[i];
			}
			else {
				j++;
				lastj = array2[j];
			}
		}
		return cpuResultSize;
	}

	int check_correctness(int cpuResultSize){
			FOR_I(0,cpuResultSize)
				if (cpuResult[i] != resultList[i]){
					printf("Wrong at [%d] R:%d Yours:%d\n",i,cpuResult[i],resultList[i]);
					debug_a( cpuResult+i,-10,10 );
					debug_a( resultList+i,-10,10 );
					return i;
				}
		return -1;
	}

	void prepare_data(int n){
		host_lists[0] = new int [n];
		resultList = new int[n];
		host_lists[1] = new int [n];
		v = new bool [10*n];
		resultList = new int[n];
		cpuResult = new int [n];
	}

#if 0

	extern int * scan_buffer;

	__global__ void set_somethings(int *L,int n){
		calculated_indices_len[0][0] = n;
	}

	void test_scan_save(){
		n = 1024*128;
		generate_random(1.0,2.0,2.0);
		init_data(1024*256);
		init_device_variables();
		int *temp = new int [n];
		int *tempR = new int [n];
		FOR_I(0,n) temp[i] = (rand() % 50 )== 1;
		hipMemcpy(devV[0],temp,sizeof(int)*n,H_T_D);
		int num = 0;
		int count = 0;

		FOR_I(0,n) if ( temp[i]) tempR[num++] = host_lists[0][i];

		set_somethings<<<1,1>>>(devV[0],n);
		//cu_host_print(devV[0],n);
		hipStream_t stream;
		hipStreamCreate(&stream);
		init_scan(&stream,1024);
		ScanSequence seq(1,devV[0],n);
		seq.save_indices_now = 0;
		seq.init();
		seq.run_all();

//		seq.run_scan(0);
//		seq.run_large();
//		seq.run_save(0);

		seq.run_saveLarge();
		hipMemcpy(resultList,devResult,sizeof(int)*num,D_T_H);


		FOR_I(0,num){
			if ( tempR[i] != resultList[i] )
				cout<<"ERROR AT "<<i<<" "<<tempR[i]<<" "<<resultList[i]<<endl;
		}
//		debug_a(tempR,num);
//		debug_a(resultList,num);

		//cu_host_print(devResult,n);
		hipDeviceSynchronize();
		cout<<"Test over "<<endl;
	}



	__global__ void help_test_cal_indices(){
		FOR_I(0,16){
			printf("i: %d (%d %d)\n",i,partitions[0][i][0],partitions[0][i][1]);
		}
	}

	__global__ void show_part(){
		FOR_I(0,16)
			printf("%d  [%d] -- [%d]\n",i,partitions[0][i][0],partitions[0][i][1]);

	}

	void test_cal_indices(){
		n = 4 * 128;
		generate_random(1.2,1.5,1.5);
		//FOR_I(0,n) host_lists[0][i] =2*i ,host_lists[1][i] = i;
		init_data(128);
		init_device_variables();

		int block_size = 128;
//		debug_a(host_lists[0],block_size+10);
//		debug_a(host_lists[1],block_size+10);


		dim3 d(8,2);
		cal_indx<<<1,d>>>(block_size,block_size,0);
		cu_checkError();
		//help_test_cal_indices<<<1,1>>>();
		cu_checkError();
		hipDeviceSynchronize();

		algo2_search<<<16,16>>>(devV[0],0,0);
		cu_checkError();
		hipDeviceSynchronize();

		cpuResultSize = 0;
		merge_algo(host_lists[0],host_lists[1],0,128,0,128);
//		cout<<"Cpu result"<<endl;debug_a(cpuResult,cpuResultSize);//debug
		resultList = new int [200];

		hipMemcpy(resultList,devV[0],sizeof(int)*128,D_T_H);
		int num_dev = 0;
		FOR_I(0,128)
			if (resultList[i]){
//				printf("[%d] %d\t",i,host_lists[0][i]);
				resultList[num_dev++] = host_lists[0][i];
			}printf("\n");

		cout<<"Check Correctness"<<endl;

		if (num_dev != cpuResultSize) { printf("Wrong num!");exit(1);}

		FOR_I(0,cpuResultSize)
			if (  resultList[i] != cpuResult[i] ){
				printf("Wrong at [%d] %d %d",i,resultList[i],cpuResult[i]);
				exit(1);
			}
		cout<<"It is correct"<<endl;
	}

#endif


int main(){

	prepare_data(1024*1024*50);


	FOR_I(154,10000){
	//r =1344532745 ;
	srand(i);
	n = 1024*1024*40;
	//n = 1024*1024;
	//n = 66;

	//generate_case5();
	//generate_random(1.0,2.0,2.0);
	generate_same(2.0);

	cout<<"generate data over srand("<<i<<") n="<<n<<" m="<<m<<endl;
	//debug_a(host_lists[0],n);debug_a(host_lists[1],m);//debug
	printf("List 1 ( %d --- %d --- %d )\n",host_lists[0][0],host_lists[0][n/2],host_lists[0][n-1]);
	printf("List 2 ( %d --- %d --- %d )\n",host_lists[1][0],host_lists[1][m/2],host_lists[1][m-1]);

	Watch watch;watch.start();
	int cpuResultSize = merge_algo(host_lists[0],host_lists[1],0,n,0,m);
	cout<<"CPU ALGO time: "<<watch.stop()<<endl;

//	work3();//test bsearch
//	return 0;


	work();
	cuda_copyResult();cout<<"copied back"<<endl;
	cu_checkError();
	free_device_memory();
	printf(" results / elements = %d / %d %lf\n",Lresult,n,(1.0*Lresult)/(1.0*n));
	outln(Lresult);outln(cpuResultSize);

	//outln(Lresult);outln(cpuResultSize);
	//debug_a(resultList,Lresult);
	//debug_a(cpuResult,cpuResultSize);
	//break;

	//continue;// DO NOT CHECK correctness

	int error_index;
	if  ( (error_index = check_correctness(cpuResultSize)) >=0 ){
		outline;
		outln(i);
		printf("all length: cpu:%d gpu:%d  ration: %.5lf\n",cpuResultSize,Lresult,100.0*i/(1.0*Lresult));
		break;
	}
		outline;
		outline;
		break; //only execute once
	}//end FOR
	return 0;
}

