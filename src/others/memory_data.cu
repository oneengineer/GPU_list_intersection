#include "hip/hip_runtime.h"

#include "../common_defines.h"
#include "memory_data.cuh"
#include <cstdio>

	void init_data(int block_size){
		int sizeV = block_size + 10;
		int *temp = new int [sizeV];
		FOR_I(0,sizeV) temp[i] = 0;

		hipMalloc( (void **)&devL1, sizeof(int)*n );
		hipMalloc( (void **)&devL2, sizeof(int)*m );
		hipMalloc( (void **)&(devV[0]), sizeof(int)*sizeV );
		hipMalloc( (void **)&(devV[1]), sizeof(int)*sizeV );
		hipMalloc( (void **)&(devV[2]), sizeof(int)*sizeV );
		hipMalloc( (void **)&devResult, sizeof(int)*n );
		hipMalloc( (void **)&devMark, sizeof(int)*(n+1) );

		hipMemcpy( devL1,host_lists[0],sizeof(int)*n,H_T_D );
		hipMemcpy( devL2,host_lists[1],sizeof(int)*m,H_T_D );
		hipMemcpy( devMark,temp,sizeof(int),H_T_D );
		hipMemcpy( devV[0],temp,sizeof(int)*sizeV,H_T_D );
		hipMemcpy( devV[1],temp,sizeof(int)*sizeV,H_T_D );
		hipMemcpy( devV[2],temp,sizeof(int)*sizeV,H_T_D );
		devV[0] +=4; // memeory allign for cudpp
		devV[1] +=4; // memeory allign for cudpp
		devV[2] +=4; // memeory allign for cudpp

	}


	__global__ void init_device_var_kernel(){
		int id = CUID;
		if (id) return;
		printf("init_device_var_kernel addr of list_p :%llx \t list_p0: %llx\n",list_p,list_p0);
		list_p[0][0] = list_p0[0];
		list_p[0][1] = list_p0[1];
		swapped[0] = 0;//false
	}


	void init_device_variables(){
		hipMemcpyToSymbol(HIP_SYMBOL(list_p0),&devL1,sizeof(int *),0,H_T_D);
		hipMemcpyToSymbol(HIP_SYMBOL(list_p0),&devL2,sizeof(int *),0+sizeof(int *),H_T_D);
		int temp[] = {n,m};
		hipMemcpyToSymbol(HIP_SYMBOL(_nm),temp,sizeof(int)*2,0,H_T_D);
		hipMemcpyToSymbol(HIP_SYMBOL( _result),&devResult,sizeof(int*),0,H_T_D);
		init_device_var_kernel<<<1,1>>>();
		hipDeviceSynchronize();
		hipMalloc( (void **)&devIndices , sizeof(int )*2);
	}

	void free_device_memory(){
		hipFree(devL1);
		hipFree(devL2);
		hipFree(devResult);
		hipFree(devMark);
	}

	// configure : <<<1,2>>> only and but only 2 threads
	__global__ void helpLen_relative(int *memory,int indices_now){
			int temp = indices_now; //next_queue_pos(indices_now);
			int id = threadIdx.x;
			int isSwapped = swapped[ temp ];
			int myindex = id ^ isSwapped;// id:1  !swapped
			memory[id] = list_p[temp][ id ]-list_p0[ myindex ];
			memory[id] = _nm[myindex] - memory[id];
		}

	void back_next_relative_len(int & len1,int & len2,int indices_now){
		int temp[2];
		helpLen_relative<<<1,2>>>(devIndices, indices_now);
		hipMemcpy(temp,devIndices,sizeof(int)*2, D_T_H);
		len1 = temp[0];len2 = temp[1];
	}

	void cuda_copyResult(){
		int *end;
		hipMemcpyFromSymbol(&end,HIP_SYMBOL(_result),sizeof(int *),0,D_T_H );
		Lresult = (int)(end - devResult );
		hipMemcpy( resultList,devResult, sizeof(int)*Lresult,D_T_H );
	}



