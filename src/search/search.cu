#include "hip/hip_runtime.h"
#include "../common_defines.h"
#include "search.cuh"
#include <cstdio>


#if DEPLETED

	//binary search lower bound in list2 for each element in list1
	//algo2 should not care about swap or not


	__device__ void search_1(int * V,int search_now,int offset){
		int id = CUID;
		id += offset;
		if (id > calculated_indices_len[search_now][0]){
//			printf("id %d quit!\n",id);
			return;
		}
		int left = 0;
		int right = calculated_indices_len[search_now][1];

		int *list1 = list_p[ search_now ][ 0 ];
		int *list2 = list_p[ search_now ][ 1 ];
		int value = list1[ id ];
//		if(!id) { printf("R:%d\n",right) ;FOR_I(0,right+1) printf("!%d\t",list2[i]); printf("\n"); }
		int result = 0;
		while ( left <= right ){
			int mid = (left + right)/2;
			if ( value == list2[ mid ] ){
//				printf("id: %d INTERSECTION %d\n",id,value);
				result = 1;
				break;
			}
			else if ( list2[mid] < value )
				left = mid + 1;
			else right = mid - 1;
		}
		V[id] = result;
//		printf("  algo2 search id: %d  %d get [%d] %d\n",id, value,right,list2[right]);
	}

	//Too many command
	__global__ void gallop_search_stage2(int *devL1,int *devL,int n){
		int id = threadIdx.x;
		int value = devL1[id];
		int pos = id;
		int left = 0,right = n;
		int bound;
		int result = 0;
		int len;
		//printf("id:%d get value:%d\n",id,value);
		int value2;

		int debugid = 20;
		int add_to_len;

		bool direction;
		//decide direction
		if ( devL[pos] <= value ) // right -->> direction
			direction = true,bound = right,len=1,add_to_len=0;
		else direction = false,bound = left,len=-1,add_to_len=1;
		printf(" (%d) value: %d  direction:%d  len:%d\n",id,value,direction,len);

		//len increase only one direction
		int pos2;
		while ( true ){
			value2 = devL[pos];
			if (id == debugid) printf(" compared with [%d]:%d,it move to %d for next time %d\n",pos,value2,pos,len);
			if ( value2 == value ) break;
			else if ( direction == ( value2 < value ) ){
				pos2 = pos + len;
			}
			else {
				len = (len+add_to_len)>>1;
				break;}
			if (!( left <= pos2 && pos2 <=right )) break;
			pos = pos2;
			len <<= 1;
		}

		if (!id) printf(" stage 1 over\n");
		printf(" (%d) value: %d  pos:%d  len:%d\n",id,value,pos,len);

		//len = 1;
		//len decrease, possible two directions
		while ( len!=0 ){
			len = (len+add_to_len)>>1;
			if (id == debugid) printf(" compared with %d,it move to %d for next time %d\n",value2,pos,len);
			if ( value2 == value ){
				result = 1;
				//add a break?
				break;
			}
			else if ( direction == (value2 < value )){
				pos += len;
			}
			else pos -= len;

			value2 = devL[pos];
		}

		if ( result )
			printf("(%d)  found %d at [%d]\n",id,value,pos);
		syncthreads();
		devL1[id] = result;
	}

	void test_gallop_search(){
		n = 1001;
		generate_random(1.0,1,1);
		//FOR_I(4,n)
//		host_lists[0][i] *=10;
		//host_lists[0][1] = 33;
		//host_lists[0][0] = 17;
		//generate_same(2);
		//generate_case3();
		debug_a(host_lists[0],n);outline;debug_a(host_lists[1],m);//debug
		init_data(128);
		init_device_variables();
		hipDeviceSynchronize();
		HERE;
		//n = 32;

		int * cpuresult = new int[n];
		FOR_I(0,n){
			cpuresult[i] = 0;
			FOR_J(0,n)
			if ( host_lists[0][i] == host_lists[1][j] ){
				cpuresult[i] = 1;
				break;
			}
		}
		gallop_search<<<1,n>>>(devL1,devL2,n-1);
		hipMemcpy(resultList,devL1,sizeof(int)*n,D_T_H);

		hipDeviceSynchronize();

		debug_a(resultList,n);
		debug_a(cpuresult,n);

		cu_checkError();

		FOR_I(0,n)
			if ( resultList[i] != cpuresult[i] ){
				printf("WRONG AT [%d]:%d\n",i,host_lists[0][i]);
				exit(0);
			}
	}

#endif


	/**
	 *
	 * To write the program use minimum command
	 * Three things to decide the next position:
	 * 1. position? out of boundary?
	 * 2. get value, direction? change or not?
	 * 3. length? how l
	 * L: position = what
	 *
	 *
	 * value: the value to search
	 * pos2: starting place, extrapolated position, it need to be return!
	 * return:result 0/1
	 *
	 */

	__device__ int gallop_search (int value,int & pos2,int *list,int left,int right){

		int pos = pos2;
		int result = 0;
		int len;
		int value2;

		char last_direction = 3;
		char direction = 0;
		char start_decrease = 0;

		len = 1;
		while ( len > 0){

			if (!( left <= pos2 && pos2 <= right )){
				start_decrease = 1;
			}
			else{
				value2 = list[pos2] - value;
				if ( 0 == value2 ){
					result = 1;break;
				}
				else direction = value2 < 0;
				start_decrease |= (direction + last_direction) ==1;// 0 + 1 or 1 + 0
				pos = pos2;
				last_direction = direction;
			}
			if ( start_decrease )
				len >>=1;
			else len <<=1;

			if ( direction )
				pos2 = pos + len;
			else pos2 = pos - len;
		}
		return result;
	}

	/*
	 * because the search_2 is done in one SM in GPU, no divergence occur
	 * begin and end are assigned from blockIdx, one block only have on value => no divergence
	 *
	 */

	template<bool use1>
	__device__ void search_2(int * V,int search_now,int offset,int begin1,int end1,int begin2,int end2){
		int id = threadIdx.x;
		// begin and end is closed interval [begin , end]

		int *list1 ;
		int *list2 ;
		int value;

		__shared__ int opposite_list[2048];

		int left(0),right;
		if ( use1 ){
			list2 = list_p[ search_now ][ 1 ];
			opposite_list[ id ] = list2[ begin2 + id ];
			if ( begin1 + id > end1 ) return;
			list1 = list_p[ search_now ][ 0 ];
			//left = begin2;
			//right = end2;
			right = end2 - begin2;
			value = list1[ begin1 + id ];
		}
		if ( !use1 ){
			list2 = list_p[ search_now ][ 0 ];
			if ( begin1 + id <= end1 ){
				opposite_list[ id ] = list2[begin1 + id];
				V[ begin1 + id ] = 0;
			}
			if ( begin2 + id > end2 ) return;
			list1 = list_p[ search_now ][ 1 ];
			//left = begin1;
			//right = end1;
			right = end1 - begin1;
			value = list1[ begin2 + id ];
		}
		syncthreads();

//		if ( !id )printf("block:%d flag:%d { %d  -> %d } [%d] -- [%d]\n",
//					blockIdx.x,use1,end1-begin1,end2 - begin2,list_p[ search_now ][ 0 ][end1],list_p[ search_now ][ 1 ][end2]);//debug



		int result = 0;
//		int mid = left;
//
//		while ( left <= right ){
//			mid = (left + right)/2;
//			if ( value == opposite_list[ mid ] ){
//				//printf("block:%d thread: %d INTERSECTION %d\n",blockIdx.x,threadIdx.x,value);
//				result = 1;
//				break;
//			}
//			else if ( opposite_list[mid] < value )
//				left = mid + 1;
//			else right = mid - 1;
//		}
		int pos = id;
		result = gallop_search(value,pos,opposite_list,0,right);


		if (use1){
			//printf("(%d,%d) set V[%d] %d to zero \n",blockIdx.x,threadIdx.x,begin1 + id,V[ begin1 + id ]);
			V[ begin1 + id ] = result;
			if (result){
				//printf("(%d,%d) set V[%d] %d to 1\n",blockIdx.x,threadIdx.x,begin1 + id,value);
			}
		}
		if (!use1){
			//printf("(%d,%d) set V[%d] %d to real_zero \n",blockIdx.x,threadIdx.x,begin1 + id,V[ begin1 + id ]);
			//V[ begin1 + id ] = 0;
			if (result){
				V[ begin1 + pos ] = 1;
				//printf("(%d,%d) set oV[%d] %d to 1\n",blockIdx.x,threadIdx.x,begin1 + mid,value);
			}
		}

	}

	__global__ void algo2_search(int * V,int search_now,int offset){
		//search_1(V,search_now,offset);
		int begin1 = partitions[search_now][blockIdx.x][0]+1;
		int end1 = partitions[search_now][blockIdx.x+1][0];
		int begin2 = partitions[search_now][blockIdx.x][1]+1;
		int end2 = partitions[search_now][blockIdx.x+1][1];
		if (end1 - begin1 <= end2 - begin2){
			search_2<true>(V,search_now,offset,begin1,end1,begin2,end2);
		}
		else {
			search_2<false>(V,search_now,offset,begin1,end1,begin2,end2);
		}
	}




