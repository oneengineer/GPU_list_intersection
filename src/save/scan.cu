#include "hip/hip_runtime.h"
#include "scan.cuh"
#include "../common_defines.h"
#include <assert.h>
#include <iostream>
#include <cstdio>

int *scan_buffer;
int SCAN_SMALL_SIZE = 2048;

hipStream_t *scan_stream;

	__device__ void scan_one(int *array,int n){
		int id = threadIdx.x;
		int id2 = id;
		__shared__ volatile int shared[ 2048*2  ];
		shared[ id ] = 0;
		id += n;
		shared[ id ] = array[ id2 ];
		for (int len = 1; len < n;len <<= 1){
			syncthreads();
			int a = shared[ id ] + shared[ id-len ];
			syncthreads();
			shared[ id ] = a;
		}
		array[ id2 ] = shared[id];
	}

	template<int loops,bool exclusive,bool directly>
	inline __device__ void scan_warp(volatile int * shared,int * src_data,int id,int idx){
		int val0;
		if ( !directly ){
			shared[idx] = 0;
			idx += WARP_SIZE;
			shared[idx] = src_data[ id ];
		}
		val0 = shared[idx];

		if ( 1 <= loops ) shared[idx] += shared[idx - 1];
		if ( 2 <= loops ) shared[idx] += shared[idx - 2];
		if ( 3 <= loops ) shared[idx] += shared[idx - 4];
		if ( 4 <= loops ) shared[idx] += shared[idx - 8];
		if ( 5 <= loops ) shared[idx] += shared[idx - 16];

		if ( exclusive )
			shared[idx] -= val0;
	}

	inline __device__ int scan_offset_share(int x){
		return (x<<1) - (x & (WARP_SIZE-1));
	}

	inline __device__ void scan_a_block2(int * src_data,int offset = 0){
		src_data += offset;
		int id = threadIdx.x;
		int id2 = threadIdx.x + blockDim.x;
		int idx = scan_offset_share(id);
		int idx2 = scan_offset_share(id2);

		volatile __shared__ int shared[SCAN_BLOCK_SIZE*2 + WARP_SIZE*2];
		syncthreads();
		scan_warp< LOG_WARP_SIZE,false,false>(shared,src_data,id,idx);//basic level
		syncthreads();
		scan_warp< LOG_WARP_SIZE,false,false>(shared,src_data,id2,idx2);//basic level part 2
		syncthreads();

		//move data
		if ( id < WARP_SIZE ){
			int id2 = scan_offset_share((id << LOG_WARP_SIZE) + WARP_SIZE-1) + WARP_SIZE;
			shared[ SCAN_BLOCK_SIZE*2 + id ] = 0;
			shared[ SCAN_BLOCK_SIZE*2 + WARP_SIZE+id ] = shared[ id2 ];
			scan_warp<LOG_SCAN_BLOCK_SIZE - LOG_WARP_SIZE ,true,true>(shared + SCAN_BLOCK_SIZE*2 + WARP_SIZE ,0,0,id);//second level, use calculated 32,in one block
		}
		syncthreads();
		//uniform update
		int a = shared[ (id >> LOG_WARP_SIZE) + SCAN_BLOCK_SIZE*2 + WARP_SIZE ];
		//printf("id: %d   += [%d] %d\n",id,(id >> LOG_WARP_SIZE) + SCAN_BLOCK_SIZE*2 + WARP_SIZE,a);
		shared[ idx+WARP_SIZE ] += a;
		a =  shared[ (id2 >> LOG_WARP_SIZE) + SCAN_BLOCK_SIZE*2 + WARP_SIZE ];
		shared[ idx2 + WARP_SIZE ] += a;
		//write back
		src_data[ id ] = shared [ idx + WARP_SIZE ];
		src_data[ id2 ] = shared [ idx2 + WARP_SIZE ];
	}

	__global__ void scan_small(int *array,int small_size){
		array += small_size * blockIdx.x;
		scan_one(array,small_size);
	}

	__global__ void scan_large(int *array,int *buffer,int blocks,int block_n){
		int id = threadIdx.x;
		int start = block_n-1;
		buffer[id] = array[start + id*block_n];
		scan_one(buffer,blocks);
	}

	__global__ void uniform_update(int *array,int *buffer){
		int id = threadIdx.x + blockIdx.x * blockDim.x;
		int addon = buffer[ blockIdx.x ];
		array[id] += addon;
	}

	__global__ void scan_x(int *array,int small_size){
		scan_a_block2(array,blockIdx.x*small_size);
	}


	void init_scan(hipStream_t * in_scan_stream = 0,int scan_small_size = 512){
		SCAN_SMALL_SIZE = scan_small_size;
		scan_stream = in_scan_stream;
		hipMalloc((void **)&scan_buffer,sizeof(int)*1024*8);
		int temp[] = {0,0,0,0};
		hipMemcpy(scan_buffer,temp,sizeof(temp),H_T_D);
	}

	void scan(int *devL,int n){
		int small_size = SCAN_SMALL_SIZE;
		int blocks = n / small_size;
		scan_small<<<blocks,small_size,0,*scan_stream>>>(devL,small_size);
		scan_large<<<1,blocks,0,*scan_stream>>>(devL,scan_buffer+4,blocks,small_size);
		uniform_update<<<blocks,small_size,0,*scan_stream>>>(devL,scan_buffer+3);
	}

	void scan2(int *devL,int n){
		int small_size = 1024;
		int blocks = n / small_size;
		scan_x<<<blocks,small_size/2,0,*scan_stream>>>(devL,small_size);
		scan_large<<<1,blocks,0,*scan_stream>>>(devL,scan_buffer+4,blocks,small_size);
		uniform_update<<<blocks,small_size,0,*scan_stream>>>(devL,scan_buffer+3);
	}

	void ScanSequence::run_scan(int step){
		assert(step < steps);
		int offset = step * step_size;
		scan_x<<< D1,D2,0,*scan_stream >>>( devL + offset , small_size);
	}

	__global__ void cu_printx(int *list,int size){
		printf("CUDA SHOWx:\n");
		if ( 0 == threadIdx.x && 0 == blockIdx.x ){
			FOR_I(0,size)
				printf("[%d]:%d\t",i,list[i]);
			printf("\n");
		}
	}

	void ScanSequence::run_large(){
		int blocks = n / small_size;
		scan_large<<<1,blocks,0,*scan_stream>>>(devL,scan_buffer+4,blocks,small_size);
	}






