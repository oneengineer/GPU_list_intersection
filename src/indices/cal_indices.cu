#include "hip/hip_runtime.h"


#include "cal_indices.cuh"
#include "../common_defines.h"
#include <cstdio>

	__device__ int next_queue_pos(int value){
			return (value+1) % QUEUE_SIZE;
	}

	// config should be <1,4>
	__device__ void cal_indx_1(int block_size,int block_2_size,int indices_now){
		int id = threadIdx.x;
		int group_id = id >> 1;
		int sub_id = id & 1;
		int opposite_sub_id = sub_id ^ 1; //!sub_id

		int indices_next = (indices_now + 1) % QUEUE_SIZE;
		int temp_blocks[] = {block_size, block_2_size};

		int *myList = list_p[indices_now][ group_id ^ sub_id ];
		int *oppositeList = list_p[indices_now][ group_id ^ opposite_sub_id ];
		int myLen = temp_blocks[sub_id];
		int oppositeLen = temp_blocks[ opposite_sub_id ];
		int value = myList[ myLen - 1 ];
		int left = 0, right = oppositeLen -1;

		while ( left < right ){
			int mid = (left + right + 1)/2;
			if ( value < oppositeList[mid] )
				right = mid - 1;
			else left = mid;
		}

		//printf("id:%d value %d get to %d\n",id,value,oppositeList[left]);//debug

		int next_opposite_offset = left + ( oppositeList[left] <= value );

		int *save_indices0 = calculated_indices_len[indices_now];
		int *save_indices = save_indices0 + (group_id << 1);
		int *opposite_indices = save_indices0 + ( (group_id^1) << 1);
		save_indices[sub_id] = next_opposite_offset ;

		int mysum = save_indices[ sub_id ] + save_indices[ opposite_sub_id ];
		int oppositeSum = opposite_indices[ sub_id ] + opposite_indices[ opposite_sub_id ];

		//decide opposite list offset and length
		if ( mysum + group_id > oppositeSum){

			//printf("cal TURN: id %d len: %d  \n",id,next_opposite_offset);//debug
			save_indices0[ opposite_sub_id ] = next_opposite_offset - 1;
			list_p[indices_now][sub_id] = myList;

			list_p[indices_next][ opposite_sub_id ] = oppositeList + next_opposite_offset;//set opposite pointer
			swapped[ indices_next ] = swapped[ indices_now ] ^ group_id;
		}
	}

	//config should be dim3 ths(16,2)
	__device__ void cal_indx_2(){

	}

	__global__ void cal_indx (int block_size,int block_2_size,int indices_now){
		cal_indx_1(block_size,block_2_size,indices_now);
	}

	//special case for cal_indices
	__global__ void move_indices(int len1,int len2,int block_size,int indices_now){
		int indices_next = (indices_now + 1) % QUEUE_SIZE;
		int swapflag = 0;
		//make sure that len1 is always the shorter one
		if ( len1 > len2 ){
			int *temp = list_p[indices_now][0];
			list_p[indices_now][0] = list_p[indices_now][1];
			list_p[indices_now][1] = temp;
			int temp2 = len1;
			len1 = len2;
			len2 = temp2;
			swapflag = 1;
		}
		int b1 = min(len1,block_size);
		list_p[ indices_next ][ 0 ] = list_p[ indices_now ][ 0 ] + b1;
		list_p[ indices_next ][ 1 ] = list_p[ indices_now ][ 1 ] ;//keep the same
		swapped[ indices_next ] = swapped[ indices_now ] ^ swapflag;
		calculated_indices_len[ indices_now ][ 0 ] = b1-1;// or b1
		calculated_indices_len[ indices_now ][ 1 ] = len2-1;// or b1
	}

