#include "hip/hip_runtime.h"
/*New feature:
	pre calculate indices, smart adaptive, fully pipeline, not fully hidden
	cudpp have streams, but number of device block/threads can not be controlled
	cudpp prefix sum for my algo
	pipeline ( cudpp pipelined) is employed
	pipeline ( myscan pipelined) is employed
*/

#include <iostream>
#include <fstream>
#include <algorithm>
#include <string>
#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#include <unistd.h>
#include <sys/time.h>

//------ for CUDA ------
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "common_defines.h"
#include "common_functions.h"
#include "others/bsearch.cuh"
#include "others/memory_data.cuh"
#include "indices/cal_indices.cuh"
#include "search/search.cuh"
#include "save/scan.cuh"
#include "save/save_result.h"

#include "others/generate_data.h"


using namespace std;

	template <typename T>
	void debug_a(T * data,int begin,int end){
		for (int i=begin;i<end;i++) cout<<"["<<i<<"]: "<<data[i]<<"\t";cout<<endl;
	}
	template <typename T>
	void debug_a(T * data,int end){
		debug_a(data,0,end);
	}
	template <typename T>
	void debug_a2(T * data,int end1,int end2){
		for (int i=0;i<end1;i++){cout<<"row "<<i<<endl; for (int j=0;j<end2;j++) cout<<"["<<i<<","<<j<<"] "<<data[i][j]<<"\t";cout<<endl;}
	}
	double get_sec(const struct timeval & tval){
		return ((double)(tval.tv_sec*1000*1000 + tval.tv_usec))/1000000.0;
	}

	template <typename T>
	T checkmin(T & data,T value){
		data = min(data,value);
		return data;
	}
struct Watch{
	timeval begin,end;
	void start(){gettimeofday(&begin,NULL);}
	double time(){return get_sec(end)-get_sec(begin);}
	double stop(){gettimeofday(&end,NULL);return time();}
};

	inline double rand_double(double range){
		return  ((double)rand()/(double)RAND_MAX)*range;
	}
//----------- cuda template ------------

struct CudaWatch{
	hipEvent_t start_event, stop_event;
	void start(){
	hipEventCreate(&start_event) ;
	hipEventCreate(&stop_event) ;
	hipEventRecord(start_event,0);
	}
	float stop(){
		float time_cost = -1.0;
		hipEventRecord(stop_event,0);
		hipEventSynchronize(stop_event);
		hipEventElapsedTime(&time_cost,start_event,stop_event);
		return time_cost;
	}
};
	void cu_checkError(){
		hipError_t error = hipGetLastError();
		if (error != hipSuccess){
			printf("ERROR: %s\n",hipGetErrorString(error));
			exit(-1);
		}
	}

	__global__ void cu_print(int *list,int size){
		printf("CUDA SHOW:\n");
		if ( 0 == threadIdx.x && 0 == blockIdx.x ){
			FOR_I(0,size)
				printf("[%d]:%d\t",i,list[i]);
			printf("\n");
		}
	}

	void cu_host_print(int *list,int size){
		hipDeviceSynchronize();
		cu_print<<<1,1>>>(list,size);
		hipDeviceSynchronize();
	}


//-------------------WRITE YOUR PROGRAM FROM HERE



int *host_lists[2];

int *devL1,*devL2;
bool *v;
int *devV[3],*devResult, *devMark;
int * devIndices;
int n,m;

int Lresult;
int *resultList;
int *cpuResult;

__device__ int *list_p0[2];// the origional   address
__device__ int *list_p[QUEUE_SIZE][2]; // save the starting position of either list
__device__ int calculated_indices_len[QUEUE_SIZE][4];
__device__ int swapped[QUEUE_SIZE];   // save swapped stage for each status
__device__ int *_result;
__device__ int _nm[2];

	inline void move_pos(int &pos){
		pos = (pos + 1) % QUEUE_SIZE;
	}

	void work(){
		int numStream;
		numStream = 4;
		hipStream_t *streams = (hipStream_t *) malloc(numStream * sizeof(hipStream_t));
		for (int i = 0; i < numStream; i++)
		{
			hipStreamCreate(&(streams[i]));
		}

		int D1,D2,D1save;
		int block_size;
		int block_2_size;
		//------ some settings ----------
		D1 = 256;
		D1save = 16;
		D2 = 512;
		//D1 = 32;
		//D2 = 32;
		//D1save = 8;

		int save_stream = 2;
		int search_stream = 1;

		block_size = D1 * D2;
		block_2_size = block_size;
		//block_2_size = block_size*1000;

		init_data(block_size);
		init_device_variables();
		init_scan(&streams[save_stream],1024);
		ScanSequence scanSeq(2,devV[0],block_size);
		scanSeq.init();

		SearchSettingQueue searchQue(2,&(streams[search_stream]));
		int searchConfig[][2] = { {64,512},{64,512},{64,512},{64,512} };
		int searchConfig8[][2] = { {32,512},{32,512},{32,512},{32,512},{32,512},{32,512},{32,512},{32,512} };
		int searchConfig1[][2] = { {256,512}};
		int searchConfig2[][2] = { {128,512},{128,512}};
		searchQue.setSettings(searchConfig2);
		searchQue.init();
		outln(searchQue.length());


		CudaWatch cudawatch;
		Watch cpuWatch;cpuWatch.start();
		cudawatch.start();

		int len1,len2;
		int cal_pos = 0, search_pos = 0, save_pos  =0; // they are for L1, L2 position
		while (true){
			int devVinc = 0; // use which devV to store data
			back_next_relative_len(len1,len2,cal_pos);
			cu_checkError();
			outln(len1);outln(len2);//debug
			if ( len1<=0 || len2 <= 0 ) break;
			int loops = min(len1,len2)/block_2_size;
			if ( loops >0 ){
				outln(loops);
				bool lastButOne = loops > 1;
				int *saveV; //pointer of saving result

				//-- stage 1
				cal_indx<<<1,4,0,streams[0]>>>(block_size,block_2_size,cal_pos);
				move_pos( cal_pos );

				//-- stage 2
				if ( lastButOne ){
					cal_indx<<<1,4,0,streams[0]>>>(block_size,block_2_size,cal_pos);
					searchQue.set(devV[devVinc],search_pos);
					searchQue.run_all();
					move_pos( cal_pos );move_pos( search_pos );
				}
				hipDeviceSynchronize();

				//-- stage middle
				for ( loops -= 2; loops> 0 ;loops -- ){
					//outln(loops);
					saveV = devV[ devVinc];devVinc = (devVinc+1) % 3;
					scanSeq.set(saveV,save_pos);searchQue.set(devV[devVinc],search_pos);


					scanSeq.run_scan(0);
					searchQue.run_search(0);
					scanSeq.run_scan(1);
					scanSeq.run_large();
					cal_indx<<<1,4,0,streams[0]>>>(block_size,block_2_size,cal_pos);
					scanSeq.run_save(0);
					searchQue.run_search(1);
					scanSeq.run_save(1);
					scanSeq.run_saveLarge();


//					scanSeq.run_all();
//					searchQue.run_all();


					move_pos( cal_pos );move_pos( search_pos );move_pos( save_pos );
					cu_checkError();
					//mssleep(10);
				}
				hipDeviceSynchronize();
				//-- stage last but one
				if (lastButOne){
					saveV = devV[ devVinc];devVinc = (devVinc+1) % 3;
					scanSeq.set(saveV,save_pos);
					scanSeq.run_all();
					move_pos( save_pos );
				}
				searchQue.set(devV[devVinc],search_pos);
				searchQue.run_all();
				move_pos( search_pos );
				hipDeviceSynchronize();
				//----- stage last --------
				saveV = devV[ devVinc];devVinc = (devVinc+1) % 3;
				scanSeq.set(saveV,save_pos);
				scanSeq.run_all();
				move_pos( save_pos );
			}
			else {
				cout<<"small seg "<<len1<<" "<<len2<<endl;
				move_indices<<<1,1,0,streams[save_stream]>>>(len1,len2,block_size,cal_pos);
				searchQue.set(devV[0],search_pos);
				searchQue.run_all();
				scanSeq.set( devV[0] ,save_pos);
				scanSeq.run_all();
				move_pos(cal_pos);move_pos(search_pos);move_pos(save_pos);
			}
			cu_checkError();
		}
		hipDeviceSynchronize();
		cout<<"MY Algo:"<<cudawatch.stop()/1000.0<<endl;
		cout<<"MY Algo cpu test: "<<cpuWatch.stop()<<endl;
	}

	int merge_algo(int *array1,int *array2, int begin1,int end1,int begin2,int end2){
		//return ;
		int i=begin1,j=begin2;
		int lasti,lastj;
		int cpuResultSize = 0;
		lasti=array1[i];
		lastj=array2[j];
		while ( i<end1 && j<end2){
			if ( lasti == lastj ){
				cpuResult[cpuResultSize++] = lasti;
				i++;j++;
				lasti=array1[i];
				lastj=array2[j];
			}
			else if ( lasti < lastj){
				i++;
				lasti = array1[i];
			}
			else {
				j++;
				lastj = array2[j];
			}
		}
		return cpuResultSize;
	}

	int check_correctness(int cpuResultSize){
			FOR_I(0,cpuResultSize)
				if (cpuResult[i] != resultList[i]){
					printf("Wrong at [%d] R:%d Yours:%d\n",i,cpuResult[i],resultList[i]);
					debug_a( cpuResult+i,-10,10 );
					debug_a( resultList+i,-10,10 );
					return i;
				}
		return -1;
	}

	void prepare_data(int n){
		host_lists[0] = new int [n];
		resultList = new int[n];
		host_lists[1] = new int [n];
		v = new bool [10*n];
		resultList = new int[n];
		cpuResult = new int [n];
	}

#if 0

	extern int * scan_buffer;

	__global__ void set_somethings(int *L,int n){
		calculated_indices_len[0][0] = n;
	}

	void test_scan_save(){
		n = 1024*128;
		generate_random(1.0,2.0,2.0);
		init_data(1024*256);
		init_device_variables();
		int *temp = new int [n];
		int *tempR = new int [n];
		FOR_I(0,n) temp[i] = (rand() % 50 )== 1;
		hipMemcpy(devV[0],temp,sizeof(int)*n,H_T_D);
		int num = 0;
		int count = 0;

		FOR_I(0,n) if ( temp[i]) tempR[num++] = host_lists[0][i];

		set_somethings<<<1,1>>>(devV[0],n);
		//cu_host_print(devV[0],n);
		hipStream_t stream;
		hipStreamCreate(&stream);
		init_scan(&stream,1024);
		ScanSequence seq(1,devV[0],n);
		seq.save_indices_now = 0;
		seq.init();
		seq.run_all();

//		seq.run_scan(0);
//		seq.run_large();
//		seq.run_save(0);

		seq.run_saveLarge();
		hipMemcpy(resultList,devResult,sizeof(int)*num,D_T_H);


		FOR_I(0,num){
			if ( tempR[i] != resultList[i] )
				cout<<"ERROR AT "<<i<<" "<<tempR[i]<<" "<<resultList[i]<<endl;
		}
//		debug_a(tempR,num);
//		debug_a(resultList,num);

		//cu_host_print(devResult,n);
		hipDeviceSynchronize();
		cout<<"Test over "<<endl;
	}
#endif

int main(){

	prepare_data(1024*1024*40);

//	test_scan_save();
//	return 0;

	FOR_I(103,10000){
	//r =1344532745 ;
	srand(i);
	n = 1024*1024*40;
	//n = 1024*1024;
	//n = 50;

	//generate_case5();
	generate_random(1.0,2.0,2.0);

//	FOR_I(0,n){ if ( 7592 == host_lists[1][i] ) printf("7592 = [%d]\n",i);
//	if ( 962329 == host_lists[1][i] ) printf("962329 = [%d]\n",i);
//	}

	cout<<"generate data over srand("<<i<<") n="<<n<<" m="<<m<<endl;
	printf("List 1 ( %d --- %d --- %d )\n",host_lists[0][0],host_lists[0][n/2],host_lists[0][n-1]);
	printf("List 2 ( %d --- %d --- %d )\n",host_lists[1][0],host_lists[1][m/2],host_lists[1][m-1]);


	Watch watch;watch.start();
	int cpuResultSize = merge_algo(host_lists[0],host_lists[1],0,n,0,m);
	cout<<"CPU ALGO time: "<<watch.stop()<<endl;

//	work3();//test bsearch
//	return 0;


	work();
	cuda_copyResult();cout<<"copied back"<<endl;
	cu_checkError();
	free_device_memory();
	printf(" results / elements = %d / %d %lf\n",Lresult,n,(1.0*Lresult)/(1.0*n));
	outln(Lresult);outln(cpuResultSize);

	//outln(Lresult);outln(cpuResultSize);
	//debug_a(resultList,Lresult);
	//debug_a(cpuResult,cpuResultSize);
	//break;

	//continue;// DO NOT CHECK correctness

	int error_index;
	if  ( (error_index = check_correctness(cpuResultSize)) >=0 ){
		outline;
		outln(i);
		printf("all length: cpu:%d gpu:%d  ration: %.5lf\n",cpuResultSize,Lresult,100.0*i/(1.0*Lresult));
		break;
	}
		outline;
		outline;
		break; //only execute once
	}//end FOR
	return 0;
}

