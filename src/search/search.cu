#include "hip/hip_runtime.h"
#include "../common_defines.h"
#include "search.cuh"
#include <cstdio>


#if DEPLETED

	//binary search lower bound in list2 for each element in list1
	//algo2 should not care about swap or not


	__device__ void search_1(int * V,int search_now,int offset){
		int id = CUID;
		id += offset;
		if (id > calculated_indices_len[search_now][0]){
//			printf("id %d quit!\n",id);
			return;
		}
		int left = 0;
		int right = calculated_indices_len[search_now][1];

		int *list1 = list_p[ search_now ][ 0 ];
		int *list2 = list_p[ search_now ][ 1 ];
		int value = list1[ id ];
//		if(!id) { printf("R:%d\n",right) ;FOR_I(0,right+1) printf("!%d\t",list2[i]); printf("\n"); }
		int result = 0;
		while ( left <= right ){
			int mid = (left + right)/2;
			if ( value == list2[ mid ] ){
//				printf("id: %d INTERSECTION %d\n",id,value);
				result = 1;
				break;
			}
			else if ( list2[mid] < value )
				left = mid + 1;
			else right = mid - 1;
		}
		V[id] = result;
//		printf("  algo2 search id: %d  %d get [%d] %d\n",id, value,right,list2[right]);
	}

	//Too many command
	__global__ void gallop_search_stage2(int *devL1,int *devL,int n){
		int id = threadIdx.x;
		int value = devL1[id];
		int pos = id;
		int left = 0,right = n;
		int bound;
		int result = 0;
		int len;
		//printf("id:%d get value:%d\n",id,value);
		int value2;

		int debugid = 20;
		int add_to_len;

		bool direction;
		//decide direction
		if ( devL[pos] <= value ) // right -->> direction
			direction = true,bound = right,len=1,add_to_len=0;
		else direction = false,bound = left,len=-1,add_to_len=1;
		printf(" (%d) value: %d  direction:%d  len:%d\n",id,value,direction,len);

		//len increase only one direction
		int pos2;
		while ( true ){
			value2 = devL[pos];
			if (id == debugid) printf(" compared with [%d]:%d,it move to %d for next time %d\n",pos,value2,pos,len);
			if ( value2 == value ) break;
			else if ( direction == ( value2 < value ) ){
				pos2 = pos + len;
			}
			else {
				len = (len+add_to_len)>>1;
				break;}
			if (!( left <= pos2 && pos2 <=right )) break;
			pos = pos2;
			len <<= 1;
		}

		if (!id) printf(" stage 1 over\n");
		printf(" (%d) value: %d  pos:%d  len:%d\n",id,value,pos,len);

		//len = 1;
		//len decrease, possible two directions
		while ( len!=0 ){
			len = (len+add_to_len)>>1;
			if (id == debugid) printf(" compared with %d,it move to %d for next time %d\n",value2,pos,len);
			if ( value2 == value ){
				result = 1;
				//add a break?
				break;
			}
			else if ( direction == (value2 < value )){
				pos += len;
			}
			else pos -= len;

			value2 = devL[pos];
		}

		if ( result )
			printf("(%d)  found %d at [%d]\n",id,value,pos);
		syncthreads();
		devL1[id] = result;
	}

	void test_gallop_search(){
		n = 1001;
		generate_random(1.0,1,1);
		//FOR_I(4,n)
//		host_lists[0][i] *=10;
		//host_lists[0][1] = 33;
		//host_lists[0][0] = 17;
		//generate_same(2);
		//generate_case3();
		debug_a(host_lists[0],n);outline;debug_a(host_lists[1],m);//debug
		init_data(128);
		init_device_variables();
		hipDeviceSynchronize();
		HERE;
		//n = 32;

		int * cpuresult = new int[n];
		FOR_I(0,n){
			cpuresult[i] = 0;
			FOR_J(0,n)
			if ( host_lists[0][i] == host_lists[1][j] ){
				cpuresult[i] = 1;
				break;
			}
		}
		gallop_search<<<1,n>>>(devL1,devL2,n-1);
		hipMemcpy(resultList,devL1,sizeof(int)*n,D_T_H);

		hipDeviceSynchronize();

		debug_a(resultList,n);
		debug_a(cpuresult,n);

		cu_checkError();

		FOR_I(0,n)
			if ( resultList[i] != cpuresult[i] ){
				printf("WRONG AT [%d]:%d\n",i,host_lists[0][i]);
				exit(0);
			}
	}

#endif



	//extern __device__ void scan_a_block_neat(int * src_data,volatile int shared[][WARP_SIZE*2]);

	template<int loops,bool exclusive>
	inline __device__ void scan_warp_neet(volatile int * shared,int value,int id){
		shared[id] = 0;
		id += WARP_SIZE;
		shared[id] = value;

		if ( 1 <= loops ) shared[id] += shared[id - 1];
		if ( 2 <= loops ) shared[id] += shared[id - 2];
		if ( 3 <= loops ) shared[id] += shared[id - 4];
		if ( 4 <= loops ) shared[id] += shared[id - 8];
		if ( 5 <= loops ) shared[id] += shared[id - 16];

		if ( exclusive )
			shared[id] -= value;
	}

	template< int id_range >
	inline __device__ void scan_a_block_neat(volatile int * src_data,volatile int  shared[][WARP_SIZE*2]){
		int id = threadIdx.x;
		int local_id = id&(WARP_SIZE-1);
		int warp_num = id >> LOG_WARP_SIZE ;
		volatile __shared__ int shared_level2[WARP_SIZE*2];

		scan_warp_neet< LOG_WARP_SIZE,false>(shared[warp_num],src_data[id],local_id );//basic level
		syncthreads();
		if ( id < WARP_SIZE){
			int myvalue;
			if ( id < id_range  )
				myvalue = shared[id][WARP_SIZE+WARP_SIZE-1];
			scan_warp_neet<LOG_WARP_SIZE ,true>(shared_level2,myvalue,id);//second level, use calculated 32,in one block
		}
		syncthreads();
		//uniform update
		int a = shared_level2[ warp_num+WARP_SIZE ];
		src_data[id] = shared[warp_num][ local_id + WARP_SIZE ] + a;
	}

	__global__ void scan_buffer_large(int save_indx){

#if DEF_D1*2/WARP_SIZE < 1
		__shared__ volatile int  shared[ 2 ][WARP_SIZE*2];
#else
		__shared__ volatile int  shared[ DEF_D1*2/WARP_SIZE ][WARP_SIZE*2];
#endif

		scan_a_block_neat<2*DEF_D1/WARP_SIZE>(_scan_buffers[save_indx],shared);
		//printf("scaned: <%d> %d\n",threadIdx.x,_scan_buffers[save_indx][threadIdx.x]);
	}



	/**
	 *
	 * To write the program use minimum command
	 * Three things decide the next position:
	 * 1. position out of boundary?
	 * 2. get value, direction? change or not?
	 * 3. length? how l
	 * L: position = what
	 *
	 *
	 * value: the value to search
	 * pos2: starting place, extrapolated position, it need to be return!
	 * return:result 0/1
	 *
	 */

	inline __device__ int gallop_search (int value,int & pos2,volatile int *list,int left,int right){

		int pos = pos2;
		int result = 0;
		int len;
		int value2;

		char last_direction = 3;
		char direction = 0;
		char start_decrease = 0;

		len = 1;
		while ( len > 0){

			if (!( left <= pos2 && pos2 <= right )){
				start_decrease = 1;
			}
			else{
				value2 = list[pos2] - value;
				if ( 0 == value2 ){
					result = 1;break;
				}
				else direction = value2 < 0;
				start_decrease |= (direction + last_direction) ==1;// 0 + 1 or 1 + 0
				pos = pos2;
				last_direction = direction;
			}
			if ( start_decrease )
				len >>=1;
			else len <<=1;

			if ( direction )
				pos2 = pos + len;
			else pos2 = pos - len;
		}
		return result;
	}

	inline __device__ int binary_search(int value,int & pos,volatile int *list,int left,int right){
		int result = 0;
		while ( left <= right ){
			if ( value == list[ pos ] ){
				//printf("block:%d thread: %d INTERSECTION %d\n",blockIdx.x,threadIdx.x,value);
				result = 1;
				break;
			}
			else if ( list[pos] < value )
				left = pos + 1;
			else right = pos - 1;
			pos = (left + right)/2;
		}
		return result;
	}

	/*
	 * because the search_2 is done in one SM in GPU, no divergence occur
	 * begin and end are assigned from blockIdx, one block only have on value => no divergence
	 *
	 */
	inline __device__ int search_one_value(int value,int & pos,volatile int *opposite_addr,const int &left,const int &right){

		int result = binary_search(value,pos,opposite_addr,left,right);

//
//		if (result){
//			int write_pos = atomicAdd(&gpu_result_size,1);
//			_result[write_pos] = value;
//		}
		return result;
	}

	/*
	 * search 4 integer stored in uint4, a,b,c,d
	 * use divide-and-conquer way a,c,b,d order search
	 *
	 */
	inline __device__ void search_uint4(const int & id,const uint4 &myvalue,volatile int *opposite_list,
			volatile int *shared_range,const struct partition_info &info,
			int &mask,int &uint4_sum){
		int pos_A,pos_X,pos_C,pos_D;
		pos_A = (info.left + info.right)/2;
		//pos_A = id<<2;//check search times
		if (search_one_value(myvalue.x,pos_A,opposite_list,info.left,info.right)){
			mask |=1,uint4_sum ++;
		}
		shared_range[id] = pos_A;
		syncthreads();
		int id_addone = id + 1;
		if ( id_addone != info.len )
			pos_D = shared_range[id_addone];
		else pos_D = info.right;

		pos_C = (pos_A + pos_D) /2 ;
		if (search_one_value(myvalue.z,pos_C,opposite_list,pos_A,pos_D)){
			mask |=4,uint4_sum ++;
		}
		pos_X = (pos_A + pos_C) /2 ;
		if (search_one_value(myvalue.y,pos_X,opposite_list,pos_A,pos_C)){
			mask |=2,uint4_sum ++;
		}
		pos_X = (pos_C + pos_D) /2 ;
		if (search_one_value(myvalue.w,pos_X,opposite_list,pos_C,pos_D)){
			mask |=8,uint4_sum ++;
		}
	}

#if 1
	/*
	 * Simple method, not use shared
	 * */
	inline __device__ void search_uint4_2(const int & id,const uint4 &myvalue,volatile int *opposite_list,
			volatile int *shared_range,const struct partition_info &info,
			int &mask,int &uint4_sum){
		int pos_A,pos_X,pos_D;
		//pos_A = id;
		pos_A = (info.left + info.right)/2;
		mask = 0,uint4_sum=0;
		if (search_one_value(myvalue.x,pos_A,opposite_list,info.left,info.right)){
			mask |=1,uint4_sum ++;
		}
		pos_D =  ( pos_A+ info.right)/2;
		if (search_one_value(myvalue.w,pos_D,opposite_list,pos_A,info.right)){
			mask |=8,uint4_sum ++;
		}
		pos_X = (pos_A + pos_D) /2 ;
		if (search_one_value(myvalue.y,pos_X,opposite_list,pos_A,pos_D)){
			mask |=2,uint4_sum ++;
		}
		if (search_one_value(myvalue.z,pos_X,opposite_list,pos_X,pos_D)){
			mask |=4,uint4_sum ++;
		}
	}
#endif



	inline __device__ void scan_and_save_buffer(volatile int *scan_array,volatile int *scan_shared,int * result_buffer,
			const int & id,uint4 &myvalue,int &mask,int &uint4_sum){
		scan_array[id] = uint4_sum;
		syncthreads();
		scan_a_block_neat<DEF_D2/4/WARP_SIZE>((int *)scan_array,(int (*)[WARP_SIZE*2])scan_shared);
		syncthreads();

		int position = scan_array[id] - uint4_sum; // exclusive scan
		int position_debug = position;

		if ( mask & 1 )
			result_buffer[position++] = myvalue.x;
		if ( mask & 2 )
			result_buffer[position++] = myvalue.y;
		if ( mask & 4 )
			result_buffer[position++] = myvalue.z;
		if ( mask & 8 )
			result_buffer[position++] = myvalue.w;

		//debug bad save method
//		FOR_I(position_debug,position){
//			//int write_pos = atomicAdd(&gpu_result_size,1);
//			if ( result_buffer[i] == 3836)
//			printf("%d is saved at %d by <%d,%d>\n",result_buffer[i],i,blockIdx.x,threadIdx.x);
//		}



	}

#if DEPLETED

	inline __device__ void brute_force(uint4 & myvalue,volatile int * list,const struct partition_info &info){
		FOR_I(info.left,info.right+1){
			if ( myvalue.x == list[i] ){
				int write_pos = atomicAdd(&gpu_result_size,1);
				_result[write_pos] = myvalue.x;
			}
			if ( myvalue.y == list[i] ){
				int write_pos = atomicAdd(&gpu_result_size,1);
				_result[write_pos] = myvalue.y;
			}
			if ( myvalue.z == list[i] ){
				int write_pos = atomicAdd(&gpu_result_size,1);
				_result[write_pos] = myvalue.z;
			}
			if ( myvalue.w == list[i] ){
				int write_pos = atomicAdd(&gpu_result_size,1);
				_result[write_pos] = myvalue.w;
			}
		}
	}
#endif

	inline __device__ void search_in_block(int * V,int search_now,const struct partition_info &info){
		int id = threadIdx.x;
		// begin and end is closed interval [begin , end]

		volatile __shared__ int opposite_list[DEF_D2];//scan use 2/4 times space
		volatile __shared__ int shared_range[DEF_D2/4];// only use once, the right most thread does not use it
		int mask,uint4_sum;
		uint4 myvalue ;
		bool flag1 = false;
		/*
		 * shared_range __shared__ use as src_data for scan
		 * opposite_list __shared__ use as
		 *  */

		// the thread which should copy opposited list elements
		if (id < info.len_opposite){

			myvalue = ((uint4 *)info.opposite_addr)[id];
			//atomicAdd(&debug1.wrong_1,myvalue.w);
			((uint4 *)opposite_list)[id] = myvalue;
			if ( id < info.len ){
				myvalue = ((uint4 *)info.addr)[id];
				flag1 = true;
				//search_uint4(id,myvalue,opposite_list,shared_range,info,mask,uint4_sum);
			}
		}
		if ( id < info.warp_len ){
			mask= 0;uint4_sum=0;
			syncthreads();
			if (flag1){
				search_uint4_2(id,myvalue,opposite_list,shared_range,info,mask,uint4_sum);
			}
			syncthreads();
			scan_and_save_buffer(shared_range,opposite_list,V+DEF_D2*blockIdx.x,id,myvalue,mask,uint4_sum);
			if ( info.warp_len-1 ==id ){
				//printf("<%d %d>'s last  is %d\n",blockIdx.x,threadIdx.x,shared_range[id]);
				_scan_buffers[search_now][blockIdx.x] = shared_range[id];//scan_array
			}
		}
	}

	__global__ void algo2_search(int * V,int search_now,int offset){
		if (partitions_info[search_now][blockIdx.x].len>0){
			search_in_block(V,search_now,partitions_info[search_now][ blockIdx.x ]);
		}
	}

#if DEPLETED

	template<bool use1>
	__device__ void search_2(int * V,int search_now,int offset,int begin1,int end1,int begin2,int end2){
		int id = threadIdx.x;
		// begin and end is closed interval [begin , end]

		int *list1 ;
		int *list2 ;
		int value;

		volatile __shared__ int opposite_list[2048];

		int left(0),right;
		if ( use1 ){
			list2 = list_p[ search_now ][ 1 ];
			opposite_list[ id ] = list2[ begin2 + id ];
			if ( begin1 + id > end1 ) return;
			list1 = list_p[ search_now ][ 0 ];
			//left = begin2;
			//right = end2;
			right = end2 - begin2;
			value = list1[ begin1 + id ];
		}
		if ( !use1 ){
			list2 = list_p[ search_now ][ 0 ];
			if ( begin1 + id <= end1 ){
				opposite_list[ id ] = list2[begin1 + id];
				V[ begin1 + id ] = 0;
			}
			if ( begin2 + id > end2 ) return;
			list1 = list_p[ search_now ][ 1 ];
			//left = begin1;
			//right = end1;
			right = end1 - begin1;
			value = list1[ begin2 + id ];
		}
		syncthreads();

//		if ( !id )printf("block:%d flag:%d { %d  -> %d } [%d] -- [%d]\n",
//					blockIdx.x,use1,end1-begin1,end2 - begin2,list_p[ search_now ][ 0 ][end1],list_p[ search_now ][ 1 ][end2]);//debug


		int result = 0;
		int pos = id;
		//result = binary_search(value,pos,opposite_list,left,right);
		result = gallop_search(value,pos,opposite_list,left,right);


		if (use1){
			//printf("(%d,%d) set V[%d] %d to zero \n",blockIdx.x,threadIdx.x,begin1 + id,V[ begin1 + id ]);
			V[ begin1 + id ] = result;
			if (result){
				//printf("(%d,%d) set V[%d] %d to 1\n",blockIdx.x,threadIdx.x,begin1 + id,value);
			}
		}
		if (!use1){
			//printf("(%d,%d) set V[%d] %d to real_zero \n",blockIdx.x,threadIdx.x,begin1 + id,V[ begin1 + id ]);
			//V[ begin1 + id ] = 0;
			if (result){
				V[ begin1 + pos ] = 1;
				//printf("(%d,%d) set oV[%d] %d to 1\n",blockIdx.x,threadIdx.x,begin1 + mid,value);
			}
		}

	}

	__global__ void algo2_search(int * V,int search_now,int offset){
		//search_1(V,search_now,offset);
		int begin1 = partitions[search_now][blockIdx.x][0]+1;
		int end1 = partitions[search_now][blockIdx.x+1][0];
		int begin2 = partitions[search_now][blockIdx.x][1]+1;
		int end2 = partitions[search_now][blockIdx.x+1][1];
		if (end1 - begin1 <= end2 - begin2){
			search_2<true>(V,search_now,offset,begin1,end1,begin2,end2);
		}
		else {
			search_2<false>(V,search_now,offset,begin1,end1,begin2,end2);
		}
	}
#endif




