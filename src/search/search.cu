#include "hip/hip_runtime.h"
#include "../common_defines.h"
#include "search.cuh"
#include <cstdio>

	//binary search lower bound in list2 for each element in list1
	//algo2 should not care about swap or not


	__device__ void search_1(int * V,int search_now,int offset){
		int id = CUID;
		id += offset;
		if (id > calculated_indices_len[search_now][0]){
//			printf("id %d quit!\n",id);
			return;
		}
		int left = 0;
		int right = calculated_indices_len[search_now][1];

		int *list1 = list_p[ search_now ][ 0 ];
		int *list2 = list_p[ search_now ][ 1 ];
		int value = list1[ id ];
//		if(!id) { printf("R:%d\n",right) ;FOR_I(0,right+1) printf("!%d\t",list2[i]); printf("\n"); }
		int result = 0;
		while ( left <= right ){
			int mid = (left + right)/2;
			if ( value == list2[ mid ] ){
//				printf("id: %d INTERSECTION %d\n",id,value);
				result = 1;
				break;
			}
			else if ( list2[mid] < value )
				left = mid + 1;
			else right = mid - 1;
		}
		V[id] = result;
//		printf("  algo2 search id: %d  %d get [%d] %d\n",id, value,right,list2[right]);
	}

	__device__ void use_shared(){}

	/*
	 * because the search_2 is done in one SM in GPU, no divergence occur
	 * begin and end are assigned from blockIdx, one block only have on value => no divergence
	 *
	 */

	template<bool use1>
	__device__ void search_2(int * V,int search_now,int offset,int begin1,int end1,int begin2,int end2){
		int id = threadIdx.x;
		// begin and end is closed interval [begin , end]

		int *list1 ;
		int *list2 ;
		int value;

		__shared__ int opposite_list[128];

		int left(0),right;
		if ( use1 ){
			list2 = list_p[ search_now ][ 1 ];
			opposite_list[ id ] = list2[ begin2 + id ];
			if ( begin1 + id > end1 ) return;
			list1 = list_p[ search_now ][ 0 ];
			//left = begin2;
			//right = end2;
			right = end2 - begin2;
			value = list1[ begin1 + id ];
		}
		if ( !use1 ){
			list2 = list_p[ search_now ][ 0 ];
			opposite_list[ id ] = list2[begin1 + id];
			if ( begin2 + id > end2 ) return;
			list1 = list_p[ search_now ][ 1 ];
			//left = begin1;
			//right = end1;
			right = end1 - begin1;
			value = list1[ begin2 + id ];
		}

//		if ( !id )printf("block:%d flag:%d { %d  -> %d } [%d] -- [%d]\n",
//					blockIdx.x,use1,end1-begin1,end2 - begin2,list_p[ search_now ][ 0 ][end1],list_p[ search_now ][ 1 ][end2]);//debug


		//TODO replace list2 as shared!
		int result = 0;
		int mid = left;

		while ( left <= right ){
			mid = (left + right)/2;
			if ( value == opposite_list[ mid ] ){
				//printf("block:%d thread: %d INTERSECTION %d\n",blockIdx.x,threadIdx.x,value);
				result = 1;
				break;
			}
			else if ( opposite_list[mid] < value )
				left = mid + 1;
			else right = mid - 1;
		}

		if ( result ){
			if (use1){
				V[ begin1 + id ] = 1;
//				printf("(%d,%d) set V[%d] %d to 1\n",blockIdx.x,threadIdx.x,begin1 + id,value);
			}
			if ( !use1 ){
				V[ begin1 + mid ] = 1;
//				printf("(%d,%d) set oV[%d] %d to 1\n",blockIdx.x,threadIdx.x,begin1 + mid,value);
			}
		}
	}

	__global__ void algo2_search(int * V,int search_now,int offset){
		//search_1(V,search_now,offset);
		int begin1 = partitions[search_now][blockIdx.x][0]+1;
		int end1 = partitions[search_now][blockIdx.x+1][0];
		int begin2 = partitions[search_now][blockIdx.x][1]+1;
		int end2 = partitions[search_now][blockIdx.x+1][1];
		if (end1 - begin1 <= end2 - begin2){
			if ( end1 < 128 ) search_2<true>(V,search_now,offset,begin1,end1,begin2,end2);
		}
		else {
			if (end2 < 128)
			search_2<false>(V,search_now,offset,begin1,end1,begin2,end2);
		}
	}




