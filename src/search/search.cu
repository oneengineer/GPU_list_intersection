#include "hip/hip_runtime.h"
#include "../common_defines.h"
#include "search.cuh"

	//binary search lower bound in list2 for each element in list1
	//algo2 should not care about swap or not
	__global__ void algo2_search(int * V,int search_now,int offset){
		int id = CUID;
		id += offset;
		if (id > calculated_indices_len[search_now][0]){
//			printf("id %d quit!\n",id);
			return;
		}
		int left = 0;
		int right = calculated_indices_len[search_now][1];

		int *list1 = list_p[ search_now ][ 0 ];
		int *list2 = list_p[ search_now ][ 1 ];
		int value = list1[ id ];
//		if(!id) { printf("R:%d\n",right) ;FOR_I(0,right+1) printf("!%d\t",list2[i]); printf("\n"); }
		int result = 0;
		while ( left <= right ){
			int mid = (left + right)/2;
			if ( value == list2[ mid ] ){
//				printf("id: %d INTERSECTION %d\n",id,value);
				result = 1;
				break;
			}
			else if ( list2[mid] < value )
				left = mid + 1;
			else right = mid - 1;
		}
		V[id] = result;
//		printf("  algo2 search id: %d  %d get [%d] %d\n",id, value,right,list2[right]);
	}
