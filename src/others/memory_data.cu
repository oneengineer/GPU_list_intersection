#include "hip/hip_runtime.h"

#include "../common_defines.h"
#include "memory_data.cuh"
#include <cstdio>

	void init_data(int block_size){
		block_size += 4 + 1 + WARP_SIZE*4;
		block_size *= 2;
		int sizeV = block_size +4;
		int *temp = new int [sizeV];
		FOR_I(0,sizeV) temp[i] = 0;

		hipMalloc( (void **)&devL1, sizeof(int)*(n+block_size) );
		hipMalloc( (void **)&devL2, sizeof(int)*(m+block_size) );
		FOR_I(0,QUEUE_SIZE){
			hipMalloc( (void **)&(devV[i]), sizeof(int)*sizeV );
			hipMemcpy( devV[i],temp,sizeof(int)*sizeV,H_T_D );
		}
		hipMalloc( (void **)&devResult, sizeof(int)*(n+block_size) );
		hipMemcpy( devL1,host_lists[0],sizeof(int)*(n),H_T_D );
		hipMemcpy( devL2,host_lists[1],sizeof(int)*(m),H_T_D );

		int max1 =  (1<<31-1) - 2 - block_size;
		FOR_I(0,block_size) temp[i] = max1+i;
		hipMemcpy((int *)(devL1+n),temp,sizeof(int)*block_size,H_T_D);
		FOR_I(0,block_size) temp[i] = max1 -1 -block_size +i;
		hipMemcpy((int *)(devL2+m),temp,sizeof(int)*block_size,H_T_D);

	}


	__global__ void init_device_var_kernel(){
		int id = CUID;
		if (id) return;
		list_p[0][0] = list_p0[0];
		list_p[0][1] = list_p0[1];
		FOR_I(0,QUEUE_SIZE){
			_result_addr[i] = _result;
			FOR_J(0,4+32) o_scan_buffers[i][j] = 0;
			_scan_buffers[i] = o_scan_buffers[i]+4;
		}
		swapped[0] = 0;//false
		gpu_result_size = 0;
		debug1.wrong_1 = debug1.wrong_2 = 0;
	}


	void init_device_variables(){
		hipMemcpyToSymbol(HIP_SYMBOL(list_p0),&devL1,sizeof(int *),0,H_T_D);
		hipMemcpyToSymbol(HIP_SYMBOL(list_p0),&devL2,sizeof(int *),0+sizeof(int *),H_T_D);
		int temp[] = {n,m};
		hipMemcpyToSymbol(HIP_SYMBOL(_nm),temp,sizeof(int)*2,0,H_T_D);
		hipMemcpyToSymbol(HIP_SYMBOL( _result),&devResult,sizeof(int*),0,H_T_D);
		init_device_var_kernel<<<1,1>>>();
		hipDeviceSynchronize();
		hipMalloc( (void **)&devIndices , sizeof(int )*2);
	}

	void free_device_memory(){
		hipFree(devL1);
		hipFree(devL2);
		hipFree(devResult);
	}

	// configure : <<<1,2>>> only and but only 2 threads
	__global__ void helpLen_relative(int *memory,int indices_now){
			int temp = indices_now; //next_queue_pos(indices_now);
			int id = threadIdx.x;
			int isSwapped = swapped[ temp ];
			int myindex = id ^ isSwapped;// id:1  !swapped
			memory[id] = list_p[temp][ id ]-list_p0[ myindex ];
			memory[id] = _nm[myindex] - memory[id];
		}

	void back_next_relative_len(int & len1,int & len2,int indices_now){
		int temp[2];
		helpLen_relative<<<1,2>>>(devIndices, indices_now);
		hipMemcpy(temp,devIndices,sizeof(int)*2, D_T_H);
		len1 = temp[0];len2 = temp[1];
	}

	void cuda_copyResult(){
		int *end;
		hipMemcpyFromSymbol(&end,HIP_SYMBOL(_result),sizeof(int *),0,D_T_H );
		Lresult = (int)(end - devResult );
		hipMemcpy( resultList,devResult, sizeof(int)*Lresult,D_T_H );
	}



