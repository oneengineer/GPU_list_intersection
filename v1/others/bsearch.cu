#include "hip/hip_runtime.h"
#include "memory_data.cuh"
#include "../common_defines.h"


	__global__ void algo_bsearch(
			int *list1,int *list2
			,int *mark
			,int n,int m){
		int i = CUID;
		int size = CUSIZE;
		while ( i < n){
			int value = list1[i];
			int left = 0,right = m-1;
			mark[i] = 0;
			while (left <= right){
				int mid = (left+right)/2;
				int difference = value-list2[mid];
				if ( difference <0 )
					right = mid-1;
				else if ( difference >0 )
					left = mid +1;
				else {
					mark[i] = 1;
					break;
				}
			}
			i += size;
		}
	}

	__global__ void bsearch_copy_result(int * list1,int *mark, int *result,int n){
		int id = CUID;
		int size = CUSIZE;
		while ( id < n ){
			if (mark[id] - mark[id-1] >0){
				result[ mark[id-1] ] = list1[ id ];
			}
			id += size;
		}
	}


