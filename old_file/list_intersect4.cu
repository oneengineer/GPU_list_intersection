#include "hip/hip_runtime.h"
/*New feature:
	different block_size for each list is employed.
	prefix sum algorithm is employed

*/

#include <iostream>
#include <fstream>
#include <algorithm>
//#include <string>
//#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#include <unistd.h>
#include <sys/time.h>
//#include <memory.h>

#define PI acos(-1)
#define eps 1e-9

#define out(x) (cout<<#x<<":"<<x<<" ")
#define outln(x) (cout<<#x<<":"<<x<<endl)
#define outs(x) (cout<<x)
#define outline (cout<<endl)
#define mssleep(time) usleep((time)*(10*1000))

#define FOR_I(begin,end) for (int i=begin;i<end;i++)
#define FOR_J(begin,end) for (int j=begin;j<end;j++)
#define FOR_K(begin,end) for (int k=begin;k<end;k++)
#define FOR_I_J(B1,E1,B2,E2) FOR_I(B1,E1) FOR_J(B2,E2)
#define FOR_I_J_K(B1,E1,B2,E2,B3,E3) FOR_I_J(B1,E1,B2,E2) FOR_K(B3,E3)
#define FOR(begin,end) FOR_I(begin,end)
#define FORN(end) FOR_I(0,end)

//------ for CUDA ------
#include <hip/hip_runtime.h>
//#include <cutil.h>
#include <hip/hip_runtime.h>
#define H_T_D hipMemcpyHostToDevice
#define D_T_H hipMemcpyDeviceToHost
#define D_T_D hipMemcpyDeviceToDevice
#define CUID (threadIdx.x + blockIdx.x*blockDim.x)
#define CUSIZE (blockDim.x*gridDim.x)

#define BANK_SIZE 32
#define BANK_SIZE_LOG 5

#define SHARE_OFFSET(id) ( (id) >> (BANK_SIZE_LOG))
using namespace std;

//for mpi
#define MCW MPI_COMM_WORLD

	template <typename T>
	void debug_a(T * data,int begin,int end){
		for (int i=begin;i<end;i++) cout<<"["<<i<<"]: "<<data[i]<<"\t";cout<<endl;
	}
	template <typename T>
	void debug_a(T * data,int end){
		debug_a(data,0,end);
	}
	template <typename T>
	void debug_a2(T * data,int end1,int end2){
		for (int i=0;i<end1;i++){cout<<"row "<<i<<endl; for (int j=0;j<end2;j++) cout<<"["<<i<<","<<j<<"] "<<data[i][j]<<"\t";cout<<endl;} 
	}
	double get_sec(const struct timeval & tval){
		return ((double)(tval.tv_sec*1000*1000 + tval.tv_usec))/1000000.0;
	}

	template <typename T>
	T checkmin(T & data,T value){
		data = min(data,value);
		return data;
	}
struct Watch{
	timeval begin,end;
	void start(){gettimeofday(&begin,NULL);}
	double time(){return get_sec(end)-get_sec(begin);}
	double stop(){gettimeofday(&end,NULL);return time();}
};

	inline double rand_double(double range){
		return  ((double)rand()/(double)RAND_MAX)*range;
	}
//----------- cuda template ------------

struct CudaWatch{
	hipEvent_t start_event, stop_event;
	void start(){
	hipEventCreate(&start_event) ;
	hipEventCreate(&stop_event) ;
	hipEventRecord(start_event,0);
	}
	float stop(){
		float time_cost = -1.0;
		hipEventRecord(stop_event,0);
		hipEventSynchronize(stop_event);
		hipEventElapsedTime(&time_cost,start_event,stop_event);
		return time_cost;
	}
};
	void cu_checkError(){
		hipError_t error = hipGetLastError();
		if (error != hipSuccess){
			printf("ERROR: %s\n",hipGetErrorString(error));
			exit(-1);
		}
	}

	__global__ void cu_print(int *list,int size){
		printf("CUDA SHOW:\n");
		if ( 0 == threadIdx.x && 0 == blockIdx.x ){
			FOR_I(0,size)
				printf("[%d]:%d\t",i,list[i]);
			printf("\n");
		}
	}

	void cu_host_print(int *list,int size){
		hipDeviceSynchronize();
		cu_print<<<1,1>>>(list,size);
		hipDeviceSynchronize();
	}
	

//-------------------WRITE YOUR PROGRAM FROM HERE


const int N = 50000100;
const int M = 1024*1024;

int list[2][N];
int *devL1,*devL2;
int length[2];

bool v[N*10];

int n,m;

int P[M],V[M],V2[M];
int resultList[N];
int cpuResult[N],cpuResultSize;

int *devP,*devV,*devResult;

int L2index,L1index,Lresult;
int *devL2index,*devL1index,*devResultIndex;

int block_size;


__shared__ int share[2048*2];

	void generate_different(int * array,int n,int mod){
		FOR_I(0,mod) v[i] = false;
		for (int i=0;i<n;i++){
			int j=-1;
			do
			{
				j = rand() % mod;
			}while ( v[j] );
			v[j] = true;
			array[i] = j;
		}
	}

	void generate_random(){
		int mod = n*7;
		generate_different(list[0],n,mod);
		sort(list[0],list[0]+n);
		generate_different(list[1],n,mod);
		sort(list[1],list[1]+n);
		m = n;
	}

	void generate_case2(){
		int temp1[] = {1,7,8,9,12};
		int temp2[] = {3,5,7,9,11};
		n = m = 5;
		FOR_I(0,n) list[0][i] = temp1[i];
		FOR_I(0,n) list[1][i] = temp2[i];
	}

	void generate_case3(){
		int temp1[] = {1,7,8,9,12};
		int temp2[] = {13,15,17,19,21};
		n = m = 5;
			FOR_I(0,n) list[0][i] = temp1[i];
			FOR_I(0,n) list[1][i] = temp2[i];
	}

	void generate_case4(){
		int temp1[] = {1,7,8,11,12,  25,29,30,55,90};
		int temp2[] = {13,15,17,19,21, 22,25,30,33,34};
		n = m = 10;
		FOR_I(0,n) list[0][i] = temp1[i];
		FOR_I(0,n) list[1][i] = temp2[i];
	}

	void generate_case5(){
		int temp1[] = {1,7,8,19,20, 21,29,30,35,40};
		int temp2[] = {13,15,17,19,21, 24,25,30,33,40};
		n = m = 10;
		FOR_I(0,n) list[0][i] = temp1[i];
		FOR_I(0,n) list[1][i] = temp2[i];
	}

	void generate_case6(){
		int temp1[] = {13,15,17,19,21, 24,25,30,33,40};
		int temp2[] = {71,77,78,79,80, 91,92,93,94,95};
		n = m = 10;
		FOR_I(0,n) list[0][i] = temp1[i];
		FOR_I(0,n) list[1][i] = temp2[i];
	}

	void init_data(){
		int sizeV = 2*block_size + 32;
		int *temp = new int [sizeV];
		FOR_I(0,sizeV) temp[i] = 0;

		hipMalloc( (void **)&devL1, sizeof(int)*n );
		hipMalloc( (void **)&devL2, sizeof(int)*m );
		hipMalloc( (void **)&devV, sizeof(int)*sizeV );
		hipMalloc( (void **)&devP, sizeof(int)*block_size );
		hipMalloc( (void **)&devResult, sizeof(int)*n );

		hipMalloc( (void **)&devL2index, sizeof(int) );
		hipMalloc( (void **)&devL1index, sizeof(int) );
		hipMalloc( (void **)&devResultIndex, sizeof(int) );

		hipMemcpy( devL1,list[0],sizeof(int)*n,H_T_D );
		hipMemcpy( devL2,list[1],sizeof(int)*m,H_T_D );
		hipMemcpy( devV,temp,sizeof(int)*sizeV,H_T_D );
		hipMemcpy( devP,temp,sizeof(int)*block_size,H_T_D );
		devV = devV + 32 + block_size;
	}

	void testCuda(){
		hipMemcpy( list[1],devL1,sizeof(int)*n,D_T_H);
		FOR_I(0,n)
			if (list[0][i] != list[1][i]){
				cout<<"CUDA ERROR"<<endl;
				exit(0);
			}
		cout<<"CUDA OK"<<endl;
		exit(0);
	}


	void cuda_copyback(){
		hipMemcpy( V,devV,sizeof(int)*block_size,D_T_H );
		hipMemcpy( P,devP,sizeof(int)*block_size,D_T_H );
	}

	void cuda_copyResult(){

		hipMemcpy( &Lresult,devResultIndex,sizeof(int),D_T_H );
		hipMemcpy( resultList,devResult, sizeof(int)*Lresult,D_T_H );
	}

//----------------- prefix sum part ------------------
	__global__ void prefix_sum_cuda(int *list){
		int id = threadIdx.x;
		int block_size = blockDim.x << 1;
		int offset = blockIdx.x * block_size;
		list += offset;
		
		share[ 2*id + SHARE_OFFSET(2*id) ] = list[ 2*id ];
		share[ 2*id + 1 + SHARE_OFFSET(2*id+1) ] = list[ 2*id+1 ];

		//printf("%d --> %d\n",2*id,2*id + SHARE_OFFSET(2*id));

		//reduce stage
		int len = 1;
		int threads = 1;
		for (threads = blockDim.x ; threads >0  ;len <<=1,threads >>= 1 ){
			syncthreads();
			if ( id < threads ){
				int from = len -1 + 2*len*id;
				int addTo =  from + len;
				from += SHARE_OFFSET(from);
				addTo += SHARE_OFFSET(addTo);
				//list[ addTo ] += list[from];
				share[addTo] += share[from];
				//printf("id %d to[%d] %d\n",id,addTo,share[addTo]);
			}
		}
		//map stage
		len >>= 2;
		for ( threads = 2;len >=1 ;len >>=1,threads <<= 1 ){
			syncthreads();
			if ( id < threads -1 ){
				int from = 2*len -1 + 2*len*id;
				int addTo = from + len;
				from += SHARE_OFFSET(from);
				addTo += SHARE_OFFSET(addTo);
				//list[ addTo ] += list[from];
				share[addTo] += share[from];
			}
		}

			syncthreads();
		list[ 2*id ] = share[ 2*id + SHARE_OFFSET(2*id) ] ;
		list[ 2*id+1 ] = share[ 2*id + 1 + SHARE_OFFSET(2*id+1) ] ;
	}


	//add the value to all elements in list
	__global__ void addup(int *list,int *valueList){
		int id = threadIdx.x;
		int block_size = blockDim.x << 1;
		int offset = (blockIdx.x+1) * block_size;
		list += offset;
		// the length of list should be double of CUSIZE

		int value = *(valueList + blockIdx.x);
		int temp = id << 1;
		list[ temp ] += value;
		list[ temp+1 ] += value;
	}

	inline int block_multiple(int value,int block_size){
		if ( value % block_size != 0 )
			return block_size - value % block_size + value;
		return value;
	}

	//do prefix sum for inputed list
	void prefix_sum_algo(int *devList,int size, int block_size,int deep){
		int num_blocks = size / block_size;
		prefix_sum_cuda<<< num_blocks ,block_size/2>>>( devList );
		if ( num_blocks <=1 ) return;
		int size2 = block_multiple(num_blocks,block_size);
		//prepare data stage
			//ATTENTION this part is sequential part
		int i=0;
		int *nextList = devList - size2 - 1;
		for (i=0;i < num_blocks ;i++){
			int *dst = nextList + i;/// NOTATION from the end and a zero between
			int *src = devList + block_size*i + block_size - 1;
			hipMemcpy( dst, src, sizeof(int) , D_T_D );//only copy last element
		}
		for (;i < size2 ;i++){
			int *dst = nextList + i;
			int temp = 0;
			hipMemcpy(dst,&temp,sizeof(int),H_T_D);
		}
		//cal higher level block
		prefix_sum_algo(nextList,size2,block_size,deep+1);
		addup<<< num_blocks-1 ,block_size/2>>>(devList,nextList);
	}

//------------------ prefix sum over ---------------


	/*find k where  L2[k]<= L1[i] < L2[k+1] for each i P[i] = k
		if  L2[k] == L1[i] then V[i] = 8;
		if  L2[k] < L1[i] then V[i] = 4;

		--- rull 3 and 4 ---
		if  any k L1[i] < L2[k] (no L2[k] <= L1[i])  Left edge
			then V[i] = 2;
		if  any k L2[k] < L1[i]  (no L1[i] < L2[k+1]) Right edge
			then V[i] = 16;

		---- because L2[k] < L2[k+1] ----
		L1[i] < L2[left] Left edge
		L2[i] > L2[right]   Right edge
	*/

	//binary search lower bound in list2 for each element in list1
	__global__ void algo2_search(int * list1, int * list2,
					int *index1,int * index2,
					int m,
					int *P,int *V,
					int block_2_size){
		list1 += *index1;
		list2 += *index2;
		//V += *index1;
		int i = threadIdx.x + blockIdx.x*blockDim.x;
		int block_size = blockDim.x * gridDim.x;
		int endIdx = block_size - 1;
		int left = 0;
		int right_edge = block_2_size - 1;
		int right = right_edge ;
		int value = list1[ i + left];
		while ( left < right ){
			int mid = (left + right + 1)/2;
			if ( value < list2[mid] )
				right = mid - 1;
			else left = mid;
		}


		P[i] = left ;
		if ( value == list2[left] ){
			V[i] = 8;
		}
		else if ( value < list2[left] ){
			//no lower bound
			V[i] = 2;P[i] = left -1;
		}
		else if ( value > list2[right_edge]){
			//bigger than all variables
			V[i] = 16;
		}
		else V[i] = 4;

		//decide index2
		if (i == endIdx){
			*index2 += P[i]+1;
		}
	}

	__global__ void prepare_prefix(int *V){
		int id = CUID;
		if (V[id] == 8){
			V[id] = 1;
		}
		else V[id] = 0;
	}

	__global__ void save_result(int *list1,int index1,int *result,int *index_r,int *V){
		int block_size = CUSIZE;
		int id = CUID;
		list1 += index1;
		result += (*index_r) - 1;
		if (V[id] - V[id-1] >0){
			result[ V[id] ] = list1[ id ];
		}
		syncthreads();
		//muse wait for others
		if ( id == block_size - 1 ){
			*index_r += V[id];
		}
	}

	//decide index1
	__global__ void cal_index(int *index1,int *index2,
				int *P,int *V){
		//V += *index1;
		int block_size = blockDim.x * gridDim.x;
		int i = threadIdx.x + blockIdx.x*blockDim.x;
		if ( i == block_size - 1 ){
			if (V[i] != 16 ){
				*index1 += block_size;
			}
		}
		else if ( V[i+1] - V[i] >=8 ){
			*index1 += i+1;
		}
	}

	__global__ void copyto(int * V,int *l){
		V[threadIdx.x] = l[threadIdx.x];
	}

	__global__ void cuda_index(int * devIndex,int index){
		*devIndex = index;
	}

	void back_index(){
		hipMemcpy(&L2index,devL2index,sizeof(int),D_T_H);
		hipMemcpy(&L1index,devL1index,sizeof(int),D_T_H);
	}

	__global__ void test(int *testv){
		*testv += 1;
	}

	void show_lists(int index1,int index2,int len1,int len2){
		printf("Two lists:\n");
		printf("L1 :\t");debug_a(list[0]+index1,len1);
		printf("L2 :\t");debug_a(list[1]+index2,len2);
		outline;
	}

	void show_vp(int len){
		cuda_copyback();
		cout<<"P\t";debug_a(P,len);
		cout<<"V\t";debug_a(V,len);
	}

	void show_vp(){
		show_vp(block_size);
	}

	void work(){
		int D1,D2,Dpre;// Dpre is for prefix sum
		int block_2_size;
		D1 = 64*2;
		D2 = 64*2;
		Dpre = 1024;

		block_size = D1 * D2;
		//block_size is only the length of looking list
		block_2_size = D1*D2*2;
		int block_sum = block_size + block_2_size;
		block_sum = block_size;

		outln(block_2_size);
		init_data();

		L1index = L2index = 0;
		cuda_index<<<1,1>>>(devL1index,0);
		cuda_index<<<1,1>>>(devL2index,0);
		cuda_index<<<1,1>>>(devResultIndex,0);

		int times=0;
		CudaWatch cudawatch;
		cudawatch.start();
		Watch watch;
		watch.start();

		bool DEB_VP = false;

		while (true){
			int len1,len2;
			len1 = n - L1index;
			len2 = m - L2index;

			//show_lists(L1index,L2index,len1,len2);//debug
			//out(len1);outln(len2);getchar();
			if ( len1<=0 || len2 <= 0 ) break;
			else if ( len1 < 1024 && ( len1 < block_sum || len2 < block_2_size) ) {
				cout<<"Small segment:";out(len1);outln(len2);out(L1index);outln(L2index);
				//getchar();
				len1 = min(block_size,len1);
				len2 = min(block_2_size,len2);
				//show_lists(L1index,L2index,len1,len2);//debug
				//algo1_each<<<len1,len2>>>(
				algo2_search<<<len1,1>>>(
					devL1,devL2,
					devL1index,devL2index,m,
					devP,devV,len2 );

				back_index();
				cal_index<<<1,len1>>>(devL1index,devL2index,devP,devV);
				int prefix_len = block_multiple(len1,Dpre);
				hipMemset(devV+len1,0,sizeof(int)*(prefix_len - len1));
				prepare_prefix<<<D1,D2>>>(devV);
				prefix_sum_algo(devV,block_size,Dpre,0);
				save_result<<<D1,D2>>>(devL1,L1index,devResult,devResultIndex,devV );


				if (DEB_VP){
					show_vp(len1);//debug
					back_index();
					out(L1index);outln(L2index);
					getchar();
				}



			}
			else if (len1 < block_sum || len2 < block_2_size){
				cout<<"go to 1024"<<endl;
				len2 = min(block_2_size,len2);
				algo2_search<<<32,32>>>(
					devL1,devL2,
					devL1index,devL2index,m,
					devP,devV,len2 );

				back_index();
				cal_index<<<32,32>>>(devL1index,devL2index,devP,devV);
				int prefix_len = block_multiple(len1,Dpre);
				prepare_prefix<<<32,32>>>(devV);
				prefix_sum_algo(devV,block_size,Dpre,0);
				save_result<<<32,32>>>(devL1,L1index,devResult,devResultIndex,devV );


			}
			else {
				int loops = min(len1/block_sum,len2/block_2_size);
				outln(loops);//debug
				if ( loops == 0) getchar();
				FOR_I(0,loops){
					//algo1_each<<<block_size,block_size>>>(
					algo2_search<<<D1,D2>>>(
						devL1,devL2,
						devL1index,devL2index,m,
						devP,devV,block_2_size);
					//cu_host_print(devV,block_size);

					back_index();
					cal_index<<<D1,D2>>>(devL1index,devL2index,devP,devV);

					prepare_prefix<<<D1,D2>>>(devV);
					prefix_sum_algo(devV,block_size,Dpre,0);

					save_result<<<D1,D2>>>(devL1,L1index,devResult,devResultIndex,devV );

					//getchar();

					if (DEB_VP){
						show_vp();//debug
						back_index();
						out(L1index);outln(L2index);
						getchar();
					}
					//back_index();//will be slow two times
				}//end for
			}//end else
			back_index();
			//cout<<"("<<L1index<<","<<L2index<<")"<<endl;//debug
			//printf("L1[%d] : %d   L2[%d] : %d\n",L1index,list[0][L1index],L2index,list[1][L2index]);

			//outline;
			//outln(times++);
		}
		outln(cudawatch.stop());
		outln(watch.stop());
	}

	void merge_algo(int *array1,int *array2, int begin1,int end1,int begin2,int end2){
		//return ;
		int i=begin1,j=begin2;
		int lasti,lastj;
		cpuResultSize = 0;
		lasti=array1[i];
		lastj=array2[j];
		while ( i<end1 && j<end2){
			if ( lasti == lastj ){
				//V2[i] = 1;
				cpuResult[cpuResultSize++] = lasti;
				i++;j++;
				lasti=array1[i];
				lastj=array2[j];
			}
			else if ( lasti < lastj){
				i++;
				lasti = array1[i];
			}
			else {
				j++;
				lastj = array2[j];
			}
		}
	}

	__global__ void GPU_merge(int *array1,int *array2,int *arrayV,int end1,int end2){
		int i=0,j=0;
		int lasti,lastj;
		lasti=array1[i];
		lastj=array2[j];
		while ( i<end1 && j<end2){
			if ( lasti == lastj ){
				arrayV[i] = 8;
				i++;j++;
				lasti=array1[i];
				lastj=array2[j];
			}
			else if ( lasti < lastj){
				i++;
				lasti = array1[i];
			}
			else {
				j++;
				lastj = array2[j];
			}
		}
	}
	void work2(){
		init_data();
		Watch watch;watch.start();
		GPU_merge<<<1,1>>>(devL1,devL2,devV,n,m);
		outln(watch.stop());
	}

	int check_correctness(){
			FOR_I(0,cpuResultSize)
				if (cpuResult[i] != resultList[i]){
					printf("Wrong at [%d] R:%d Yours:%d\n",i,cpuResult[i],resultList[i]);
					return i;
				}
		return -1;
	}

	__global__ void algo_bsearch(
			int *list1,int *list2
			,int *V
			,int n,int m){
		int i = threadIdx.x + blockIdx.x*blockDim.x;
		while ( i < n){
			int value = list1[i];
			int left = 0,right = m-1;
			while (left <= right){
				int mid = (left+right)/2;
				int difference = value-list2[mid];
				if ( difference <0 )
					right = mid-1;
				else if ( difference >0 )
					left = mid +1;
				else {
					V[i] = 8;break;
				}
			}
			//syncthreads();
			i += blockDim.x * gridDim.x;
		}
	}

	void work3(){
		init_data();
		Watch watch;CudaWatch cudawatch;
		watch.start();cudawatch.start();
		//best achieved at 128,128 64, 32, 16 
		// which is strange for 16 !
		algo_bsearch<<<32,512>>>(devL1,devL2,devV,n,m);
		outln(cudawatch.stop());
		outln(watch.stop());
	}

	void test_bsearch(){
		int value = 57;
		int left = 0;
		int right = 1;
		int list2[] = {16,16,30 };
		while ( left < right ){
			int mid = (left + right + 1)/2;
			outln(mid);
			if ( value < list2[mid] )
				right = mid - 1;
			else left = mid;
		}
		outln(left);
	}

	void test_prefix_sum(){
		n = 1024*2;
		generate_random();
		int m = n*2 +32;
		hipMalloc((void **)& devL1, sizeof(int)*m);
		hipMemcpy( devL1+n+32 ,list[0],sizeof(int)*n,H_T_D);
		hipMemcpy( devL1,list[0]+n,sizeof(int)*n,H_T_D);
		prefix_sum_algo(devL1+n+32,n,2,0);
		cu_checkError();
		hipDeviceSynchronize();
		cu_host_print(devL1,2*n);//debug
		hipDeviceSynchronize();
		hipMemcpy( list[1],devL1+n+32,sizeof(int)*n,D_T_H);
		FOR_I(1,n) {
			list[0][i] += list[0][i-1];
			if (list[0][i] != list[1][i]){
				cout<<"ERROR at"<<i<<endl;
				exit(123);
			}
		}

		cu_checkError();
		exit(0);
	}

int main(){
	//test_prefix_sum();

	//test_bsearch();return 0;

	int r;
	FOR_I(41,1000){
	r = rand() % 10013123 ;
	//r =1344532745 ;
	srand(i);
	n = 1024*1024*50;
	//n = 1000*1000*10;
	//n = 1000001;
	//n = 3412312;
	//n = 1024*50;
	//generate_case2();
	generate_random();
	cout<<"generate data over srand("<<i<<") n="<<n<<endl;

		//debug_a(list[0],n);
		//debug_a(list[1],n);

	memset(V2,0,sizeof(V2));
	memset(V,0,sizeof(V));

	Watch watch;watch.start();
	merge_algo(list[0],list[1],0,n,0,m);
	outln(watch.stop());

	work();
	cu_checkError();

	//work2();//test single merge
	//work3();//test bsearch
	//cuda_copyback();
	cuda_copyResult();
	outln(Lresult);outln(cpuResultSize);
	//debug_a(resultList,Lresult);
	cout<<"copied back"<<endl;
	//break;

	int error_index;
	if  ( (error_index = check_correctness()) >=0 ){
		outline;
		outln(i);
		break;
	}
		outline;
		outline;
		outline;
		outline;

		//break; //debug
	}//end FOR
	return 0;
}

