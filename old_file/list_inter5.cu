#include "hip/hip_runtime.h"
/*New feature:
	pre calculate indices, smart adaptive, fully pipeline, not fully hidden
	cudpp have streams, but number of device block/threads can not be controlled
	cudpp prefix sum for my algo
	pipeline ( cudpp pipelined) is employed
	pipeline ( myscan pipelined) is employed
*/

#include <iostream>
#include <fstream>
#include <algorithm>
//#include <string>
//#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#include <unistd.h>
#include <sys/time.h>
#include <cudpp.h>

#include "t_scan.h"

#define PI acos(-1)
#define eps 1e-9


#define out(x) (cout<<#x<<":"<<x<<" ")
#define outln(x) (cout<<#x<<":"<<x<<endl)
#define outs(x) (cout<<x)
#define outline (cout<<endl)
#define HERE (printf("GET HERE\n"))
#define THERE (printf("GET THERE\n"))
#define mssleep(time) usleep((time)*(10*1000))

#define FOR_I(begin,end) for (int i=begin;i<end;i++)
#define FOR_J(begin,end) for (int j=begin;j<end;j++)
#define FOR_K(begin,end) for (int k=begin;k<end;k++)
#define FOR_I_J(B1,E1,B2,E2) FOR_I(B1,E1) FOR_J(B2,E2)
#define FOR_I_J_K(B1,E1,B2,E2,B3,E3) FOR_I_J(B1,E1,B2,E2) FOR_K(B3,E3)
#define FOR(begin,end) FOR_I(begin,end)
#define FORN(end) FOR_I(0,end)

#define SWAP(a,b) { a=(a)-(b);b=(b)+(a);a=(b)-(a);}

//------ for CUDA ------
#include <hip/hip_runtime.h>
//#include <cutil.h>
#include <hip/hip_runtime.h>
#define H_T_D hipMemcpyHostToDevice
#define D_T_H hipMemcpyDeviceToHost
#define D_T_D hipMemcpyDeviceToDevice
#define CUID (threadIdx.x + blockIdx.x*blockDim.x)
#define CUSIZE (blockDim.x*gridDim.x)

using namespace std;

//for mpi
#define MCW MPI_COMM_WORLD

	template <typename T>
	void debug_a(T * data,int begin,int end){
		for (int i=begin;i<end;i++) cout<<"["<<i<<"]: "<<data[i]<<"\t";cout<<endl;
	}
	template <typename T>
	void debug_a(T * data,int end){
		debug_a(data,0,end);
	}
	template <typename T>
	void debug_a2(T * data,int end1,int end2){
		for (int i=0;i<end1;i++){cout<<"row "<<i<<endl; for (int j=0;j<end2;j++) cout<<"["<<i<<","<<j<<"] "<<data[i][j]<<"\t";cout<<endl;}
	}
	double get_sec(const struct timeval & tval){
		return ((double)(tval.tv_sec*1000*1000 + tval.tv_usec))/1000000.0;
	}

	template <typename T>
	T checkmin(T & data,T value){
		data = min(data,value);
		return data;
	}
struct Watch{
	timeval begin,end;
	void start(){gettimeofday(&begin,NULL);}
	double time(){return get_sec(end)-get_sec(begin);}
	double stop(){gettimeofday(&end,NULL);return time();}
};

	inline double rand_double(double range){
		return  ((double)rand()/(double)RAND_MAX)*range;
	}
//----------- cuda template ------------

struct CudaWatch{
	hipEvent_t start_event, stop_event;
	void start(){
	hipEventCreate(&start_event) ;
	hipEventCreate(&stop_event) ;
	hipEventRecord(start_event,0);
	}
	float stop(){
		float time_cost = -1.0;
		hipEventRecord(stop_event,0);
		hipEventSynchronize(stop_event);
		hipEventElapsedTime(&time_cost,start_event,stop_event);
		return time_cost;
	}
};
	void cu_checkError(){
		hipError_t error = hipGetLastError();
		if (error != hipSuccess){
			printf("ERROR: %s\n",hipGetErrorString(error));
			exit(-1);
		}
	}

	__global__ void cu_print(int *list,int size){
		printf("CUDA SHOW:\n");
		if ( 0 == threadIdx.x && 0 == blockIdx.x ){
			FOR_I(0,size)
				printf("[%d]:%d\t",i,list[i]);
			printf("\n");
		}
	}

	void cu_host_print(int *list,int size){
		hipDeviceSynchronize();
		cu_print<<<1,1>>>(list,size);
		hipDeviceSynchronize();
	}


//-------------------WRITE YOUR PROGRAM FROM HERE


const int N = 55000100;
const int M = 1024*1024;

int list[2][N];
int *devL1,*devL2;

bool v[N*10];

int n,m;

int V[M];
int resultList[N];
int cpuResult[N],cpuResultSize;

int *devV[2],*devResult, *devMark;

int L2index,L1index,Lresult;
int *devL2index,*devL1index,*devResultIndex;

int block_size;

const int QUEUE_SIZE= 4;

__device__ int *list_p0[2];
__device__ int *list_p[QUEUE_SIZE][2]; // save the starting position of either list
__device__ int calculated_length[QUEUE_SIZE];
__device__ int calculated_indices_len[QUEUE_SIZE][4];
__device__ int swapped[QUEUE_SIZE];   // save swapped stage for each status
__device__ int _nm[2];
__device__ int *_result;

__device__ double ratio_avg;
__device__ int cal_times;

int * devIndices;

__shared__ int share[2048*2];

	void generate_different(int * array,int n,int mod){
		FOR_I(0,mod) v[i] = false;
		for (int i=0;i<n;i++){
			int j=-1;
			do
			{
				j = rand() % mod;
			}while ( v[j] );
			v[j] = true;
			array[i] = j;
		}
	}

	void generate_random(double alpha=1.0,double scala1=1.0,double scala2=1.0){
		int mod = n*4;
		m = (int)((double)n*alpha + eps);
		int mod1 = (int)((double)mod*scala1 + eps);
		int mod2 = (int)((double)mod*scala2 + eps);
		generate_different(list[0],n,mod1);
		sort(list[0],list[0]+n);
		generate_different(list[1],m,mod2);
		sort(list[1],list[1]+m);
	}

	void generate_case2(){
		printf("a general test. mainly for the search bound\nPlease use D1 = D2 = 2,block2size = 5\n\n");
		int temp1[] = {1,2,3,4,           20,            29, 34,36,37,38 ,39,30,66
					,100,110,120,130,140,200,211,230,231,540 };
		int temp2[] = {1,2,3,4,6,7,10,12, 20,25,26,27,28 ,29                   ,60
				,100,110,120,130,140,200,210,220,230,540};

		n = sizeof(temp1)/sizeof(int);
		m = sizeof(temp2)/sizeof(int);
		FOR_I(0,n) list[0][i] = temp1[i];
		FOR_I(0,m) list[1][i] = temp2[i];
	}

	void generate_case3(){
		printf("a generate test. mainly for the search swap\nPlease use D1 = 1; D2 = 3,block2size = 5\n\n");
		int temp1[] = {1,  2,  9,   11,15,16,17,26,27
					,100,110,120,130,140,200,211,230,231,540 };
		int temp2[] = {1,2,5,6,9,          21,25,  27
					,100,110,120,130,140,200,210,220,230,540 };

		n = sizeof(temp1)/sizeof(int);
		m = sizeof(temp2)/sizeof(int);
		FOR_I(0,n) list[0][i] = temp1[i];
		FOR_I(0,m) list[1][i] = temp2[i];
	}

	void generate_case4(){
		printf("a end point test. Please use D1 = 2; D2 = 2,block2size = 6\n\n");
		int temp1[] = {196};
		int temp2[] = {195,196,200};

		n = sizeof(temp1)/sizeof(int);
		m = sizeof(temp2)/sizeof(int);
		FOR_I(0,n) list[0][i] = temp1[i];
		FOR_I(0,m) list[1][i] = temp2[i];
	}

	void generate_case5(){
		printf(" a total equal test \n\n");
		n = 64;
		m = 64;
		FOR_I(0,n) list[0][i] = list[1][i] = i*10 + rand() % 5;
	}



	void init_data(){
		int sizeV = block_size + 10;
		int *temp = new int [sizeV];
		FOR_I(0,sizeV) temp[i] = 0;

		hipMalloc( (void **)&devL1, sizeof(int)*n );
		hipMalloc( (void **)&devL2, sizeof(int)*m );
		hipMalloc( (void **)&(devV[0]), sizeof(int)*sizeV );
		hipMalloc( (void **)&(devV[1]), sizeof(int)*sizeV );
		hipMalloc( (void **)&devResult, sizeof(int)*n );
		hipMalloc( (void **)&devMark, sizeof(int)*(n+1) );
		cu_checkError();

		hipMalloc( (void **)&devL2index, sizeof(int) );
		hipMalloc( (void **)&devL1index, sizeof(int) );
		hipMalloc( (void **)&devResultIndex, sizeof(int) );
		cu_checkError();

		hipMemcpy( devL1,list[0],sizeof(int)*n,H_T_D );
		hipMemcpy( devL2,list[1],sizeof(int)*m,H_T_D );
		hipMemcpy( devMark,temp,sizeof(int),H_T_D );
		hipMemcpy( devV[0],temp,sizeof(int)*sizeV,H_T_D );
		hipMemcpy( devV[1],temp,sizeof(int)*sizeV,H_T_D );
		devV[0] +=4; // memeory allign for cudpp
		devV[1] +=4; // memeory allign for cudpp

		cu_checkError();
	}

	void show_lists(int index1,int index2,int len1=n,int len2=n){
		printf("Two lists:\n");
		printf("L1 :\t");debug_a(list[0]+index1,len1);
		printf("L2 :\t");debug_a(list[1]+index2,len2);
		outline;
	}

	void cuda_copyResult(){
		int *end;
		hipMemcpyFromSymbol(&end,HIP_SYMBOL(_result),sizeof(int *),0,D_T_H );
		cu_checkError();
		Lresult = (int)(end - devResult );
		hipMemcpy( resultList,devResult, sizeof(int)*Lresult,D_T_H );
		cu_checkError();
	}

	__device__ int next_queue_pos(int value){
			return (value+1) % QUEUE_SIZE;
	}

	__global__ void prefix_sum_oneCore(int *V,int block_size){
		FOR_I(1,block_size)
			V[i] += V[i-1];
	}

//------------------ prefix sum over ---------------

	//binary search lower bound in list2 for each element in list1
	//algo2 should not care about swap or not
	__global__ void algo2_search(int * V,int search_now,int offset = 0){
		int id = CUID;
		id += offset;
		if (id > calculated_indices_len[search_now][0]){
//			printf("id %d quit!\n",id);
			return;
		}
		int left = 0;
		int right = calculated_indices_len[search_now][1];

		int *list1 = list_p[ search_now ][ 0 ];
		int *list2 = list_p[ search_now ][ 1 ];
		int value = list1[ id ];
//		if(!id) { printf("R:%d\n",right) ;FOR_I(0,right+1) printf("!%d\t",list2[i]); printf("\n"); }
		int result = 0;
		while ( left <= right ){
			int mid = (left + right)/2;
			if ( value == list2[ mid ] ){
//				printf("id: %d INTERSECTION %d\n",id,value);
				result = 1;
				break;
			}
			else if ( list2[mid] < value )
				left = mid + 1;
			else right = mid - 1;
		}
		V[id] = result;
//		printf("  algo2 search id: %d  %d get [%d] %d\n",id, value,right,list2[right]);
	}

	__global__ void save_result(int *V,int indices_now){
		int _size = calculated_indices_len[indices_now][0];
		int id = CUID;
		int cusize = CUSIZE;
		int *list1 = list_p[indices_now][0];


		while (id <= _size ){
			int diff = V[id] - V[id-1];
			if ( diff >0){
				//if (diff !=1) printf("ERROR !=1\n");
				if ( V[id-1] > 256*512 ) {
					printf("ERROR > D1*D2 %d  id:[%d]  says save to V:%llx \n",V[id-1],id,V);
				}
				//if ( V[id-1] <0 ) printf("ERROR < 0\n");
				//printf("save result %d\n",list1[ id ]);
				_result[ V[id-1] ] = list1[ id ];
			}
			id += cusize;
		}
	}

	//can only be executed by one core
	__global__ void move_result_addr(int *V,int indices_now){
		int _size = calculated_indices_len[indices_now][0];
		_result += V[_size];
	}

	__global__ void init_device_var_kernel(){

		int id = CUID;
		if (id) return;
		list_p[0][0] = list_p0[0];
		list_p[0][1] = list_p0[1];
		swapped[0] = 0;//false
		ratio_avg = 0.0;//for debug
		cal_times = 0;//for debug
	}


	void init_device_variables(){
		hipMemcpyToSymbol(HIP_SYMBOL(list_p0),&devL1,sizeof(int *),0,H_T_D);
		cu_checkError();
		hipMemcpyToSymbol(HIP_SYMBOL(list_p0),&devL2,sizeof(int *),0+sizeof(int *),H_T_D);
		int temp[] = {n,m};
		hipMemcpyToSymbol(HIP_SYMBOL(_nm),temp,sizeof(int)*2,0,H_T_D);

		hipMemcpyToSymbol(HIP_SYMBOL( _result),&devResult,sizeof(int*),0,H_T_D);
		cu_checkError();
		init_device_var_kernel<<<1,1>>>();
		cu_checkError();
		hipDeviceSynchronize();
		hipMalloc( (void **)&devIndices , sizeof(int )*2);
	}

	void free_device_memory(){
		hipFree(devL1);
		hipFree(devL2);
		hipFree(devResult);
		hipFree(devMark);
		cu_checkError();
	}

	__global__ void cal_indx (int block_size,int block_2_size,int indices_now){
		int id = threadIdx.x;
		int group_id = id >> 1;
		int sub_id = id & 1;
		int opposite_sub_id = sub_id ^ 1; //!sub_id

		int indices_next = (indices_now + 1) % QUEUE_SIZE;
		int temp_blocks[] = {block_size, block_2_size};

		int *myList = list_p[indices_now][ group_id ^ sub_id ];
		int *oppositeList = list_p[indices_now][ group_id ^ opposite_sub_id ];
		int myLen = temp_blocks[sub_id];
		int oppositeLen = temp_blocks[ opposite_sub_id ];
		int value = myList[ myLen - 1 ];
		int left = 0, right = oppositeLen -1;

		while ( left < right ){
			int mid = (left + right + 1)/2;
			if ( value < oppositeList[mid] )
				right = mid - 1;
			else left = mid;
		}

		//printf("id:%d value %d get to %d\n",id,value,oppositeList[left]);//debug

		int next_opposite_offset = left + ( oppositeList[left] <= value );

		int *save_indices0 = calculated_indices_len[indices_now];
		int *save_indices = save_indices0 + (group_id << 1);
		int *opposite_indices = save_indices0 + ( (group_id^1) << 1);
		save_indices[sub_id] = next_opposite_offset ;

		int mysum = save_indices[ sub_id ] + save_indices[ opposite_sub_id ];
		int oppositeSum = opposite_indices[ sub_id ] + opposite_indices[ opposite_sub_id ];

		//decide opposite list offset and length
		if ( mysum + group_id > oppositeSum){

			//printf("cal TURN: id %d len: %d  \n",id,next_opposite_offset);//debug
			save_indices0[ opposite_sub_id ] = next_opposite_offset - 1;
			list_p[indices_now][sub_id] = myList;

			list_p[indices_next][ opposite_sub_id ] = oppositeList + next_opposite_offset;//set opposite pointer
			swapped[ indices_next ] = swapped[ indices_now ] ^ group_id;
		}
	}

	//special case for cal_indices
	__global__ void move_indices(int len1,int len2,int block_size,int indices_now){
		int indices_next = (indices_now + 1) % QUEUE_SIZE;
		int swapflag = 0;
		//make sure that len1 is always the shorter one
		if ( len1 > len2 ){
			int *temp = list_p[indices_now][0];
			list_p[indices_now][0] = list_p[indices_now][1];
			list_p[indices_now][1] = temp;
			SWAP(len1,len2);
			swapflag = 1;
		}
		int b1 = min(len1,block_size);
		list_p[ indices_next ][ 0 ] = list_p[ indices_now ][ 0 ] + b1;
		list_p[ indices_next ][ 1 ] = list_p[ indices_now ][ 1 ] ;//keep the same
		swapped[ indices_next ] = swapped[ indices_now ] ^ swapflag;
		calculated_indices_len[ indices_now ][ 0 ] = b1-1;// or b1
		calculated_indices_len[ indices_now ][ 1 ] = len2-1;// or b1
	}


	// configure : <<<1,2>>> only and but only 2 threads
	__global__ void helpLen_relative(int *memory,int indices_now){
			int temp = indices_now; //next_queue_pos(indices_now);
			int id = threadIdx.x;
			int isSwapped = swapped[ temp ];
			int myindex = id ^ isSwapped;// id:1  !swapped
			memory[id] = list_p[temp][ id ]-list_p0[ myindex ];
			memory[id] = _nm[myindex] - memory[id];
		}

	void back_next_relative_len(int & len1,int & len2,int indices_now){
		int temp[2];
		helpLen_relative<<<1,2>>>(devIndices, indices_now);
		hipMemcpy(temp,devIndices,sizeof(int)*2, D_T_H);
		len1 = temp[0];len2 = temp[1];
	}

	__global__ void show_cur_lists(int len1,int len2,int indices_now){
		FOR_I(0, calculated_indices_len[indices_now][0] )
				printf("{%d} %d\t",i,list_p[indices_now][0][i]);printf("\n");
		FOR_I(0, calculated_indices_len[indices_now][1])
				printf("{%d} %d\t",i,list_p[indices_now][1][i]);printf("\n");
	}

	CUDPPHandle prepare_prefixsum(int size, hipStream_t * stream){
		CUDPPConfiguration config;
		config.op = CUDPP_ADD;
		config.datatype = CUDPP_INT;
		config.algorithm = CUDPP_SCAN;
		config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;
		config.mystream = stream;

		CUDPPHandle theCudpp;
		cudppCreate(&theCudpp);
		CUDPPHandle scanplan = 0;
		CUDPPResult res = cudppPlan(theCudpp,&scanplan,config,size,1,0);
		if ( CUDPP_SUCCESS != res ){
			printf("ERROR in prepare_prefixsum\n");
			exit(-1);
		}
		return scanplan;
	}
	inline void move_pos(int &pos){
		pos = (pos + 1) % QUEUE_SIZE;
	}

struct SearchSetting{
	int D1,D2;
	int size(){
		return D1*D2;
	}
	void set(int D1,int D2){
		this->D1 = D1,this->D2 = D2;
	}
};

struct SearchSettingQueue{
	int parts;
	SearchSetting * settings;

	SearchSettingQueue(int parts){
		settings = new SearchSetting[parts];
	}
};

	void work(){

		int numStream;
		numStream = 4;
		hipStream_t *streams = (hipStream_t *) malloc(numStream * sizeof(hipStream_t));
		for (int i = 0; i < numStream; i++)
		{
			hipStreamCreate(&(streams[i]));
		}

		int D1,D2,D1save;
		int block_2_size;
		//------ some settings ----------
		D1 = 256;
		D1save = 8;
		D2 = 512;

		//D1 = 32;
		//D2 = 32;
		//D1save = 8;

		int save_stream = 2;

		block_size = D1 * D2;
		block_2_size = block_size;
		//block_2_size = block_size*1000;

		init_data();
		init_device_variables();
		CudaWatch cudawatch;
		Watch cpuWatch;cpuWatch.start();
		cudawatch.start();

		CUDPPHandle prefixsum_plan = prepare_prefixsum(block_size,&streams[save_stream]);
		//prefixsum_plan = prepare_prefixsum(block_size,0);

		init_scan(&streams[save_stream],1024);

		int len1,len2;
		int cal_pos = 0, search_pos = 0, save_pos  =0; // they are for L1, L2 position
		while (true){
			back_next_relative_len(len1,len2,cal_pos);
			cu_checkError();
			outln(len1);outln(len2);//debug
			if ( len1<=0 || len2 <= 0 ) break;
			int loops = min(len1,len2)/block_2_size;
			if ( loops >0 ){
				outln(loops);
				bool lastButOne = loops > 1;
				int *saveV; //pointer of saving result
				int devVinc = 0; // use which devV to store data

				//-- stage 1
				cal_indx<<<1,4,0,streams[0]>>>(block_size,block_2_size,cal_pos);
				move_pos( cal_pos );

				//-- stage 2
				if ( lastButOne ){
					cal_indx<<<1,4,0,streams[0]>>>(block_size,block_2_size,cal_pos);
					move_pos( cal_pos );
					algo2_search<<<D1,D2,0,streams[1]>>>( devV[devVinc],search_pos );
					move_pos( search_pos );
				}
				hipDeviceSynchronize();

				//-- stage middle
				for ( loops -= 2; loops> 0 ;loops -- ){
					//outln(loops);
					saveV = devV[ devVinc];
					devVinc = 1- devVinc;
					//scan(saveV,block_size);
					scan2(saveV,block_size);
					save_result<<<D1save,D2,0,streams[save_stream]>>>(saveV,save_pos);/*---- save part -----*/
					algo2_search<<<D1,D2,0,streams[1]>>>( devV[devVinc],search_pos );
					//hipDeviceSynchronize();
					cal_indx<<<1,4,0,streams[0]>>>(block_size,block_2_size,cal_pos);
					//cudppScan(prefixsum_plan,saveV,saveV,block_size);/*---- save part -----*/
					move_result_addr<<<1,1,0,streams[save_stream]>>>(saveV,save_pos);/*---- save part -----*/
					move_pos( cal_pos );move_pos( search_pos );move_pos( save_pos );
					cu_checkError();
				}
				hipDeviceSynchronize();
				//-- stage last but one
				if (lastButOne){
					saveV = devV[ devVinc];
					//scan(saveV,block_size);/*---- save part -----*/
					cudppScan(prefixsum_plan,saveV,saveV,block_size);/*---- save part -----*/
					devVinc = 1- devVinc;
					save_result<<<D1save,D2,0,streams[save_stream]>>>(saveV,save_pos);/*---- save part -----*/
					move_result_addr<<<1,1,0,streams[save_stream]>>>(saveV,save_pos);/*---- save part -----*/
					move_pos( save_pos );
				}
				algo2_search<<<D1,D2,0,streams[1]>>>( devV[devVinc],search_pos );
				move_pos( search_pos );
				hipDeviceSynchronize();
				//----- stage last --------
				saveV = devV[ devVinc];
				//scan(saveV,block_size);/*---- save part -----*/
				cudppScan(prefixsum_plan,saveV,saveV,block_size);/*---- save part -----*/
				devVinc = 1- devVinc;
				save_result<<<D1,D2,0,streams[save_stream]>>>(saveV,save_pos);/*---- save part -----*/
				move_result_addr<<<1,1,0,streams[save_stream]>>>(saveV,save_pos);/*---- save part -----*/
				move_pos( save_pos );
			}
			else {
				cout<<"small seg "<<len1<<" "<<len2<<endl;
				move_indices<<<1,1>>>(len1,len2,block_size,cal_pos);
				algo2_search<<<D1,D2>>>(devV[0],search_pos);
				cudppScan(prefixsum_plan,devV[0],devV[0],block_size);
				save_result<<<D1,D2>>>(devV[0],save_pos);
				move_result_addr<<<1,1>>>(devV[0],save_pos);
				move_pos(cal_pos);move_pos(search_pos);move_pos(save_pos);
			}
			//printf(" poss: %d %d %d\n",cal_pos,search_pos,save_pos);
		}
		cu_checkError();
		hipDeviceSynchronize();
		cout<<"MY Algo:"<<cudawatch.stop()/1000.0<<endl;
		cout<<"MY Algo cpu test: "<<cpuWatch.stop()<<endl;
	}

	void merge_algo(int *array1,int *array2, int begin1,int end1,int begin2,int end2){
		//return ;
		int i=begin1,j=begin2;
		int lasti,lastj;
		cpuResultSize = 0;
		lasti=array1[i];
		lastj=array2[j];
		while ( i<end1 && j<end2){
			if ( lasti == lastj ){
				cpuResult[cpuResultSize++] = lasti;
				i++;j++;
				lasti=array1[i];
				lastj=array2[j];
			}
			else if ( lasti < lastj){
				i++;
				lasti = array1[i];
			}
			else {
				j++;
				lastj = array2[j];
			}
		}
	}

	int check_correctness(){
			FOR_I(0,cpuResultSize)
				if (cpuResult[i] != resultList[i]){
					printf("Wrong at [%d] R:%d Yours:%d\n",i,cpuResult[i],resultList[i]);
					debug_a( cpuResult+i,-10,10 );
					debug_a( resultList+i,-10,10 );
					return i;
				}
		return -1;
	}

	__global__ void algo_bsearch(
			int *list1,int *list2
			,int *mark
			,int n,int m){
		int i = CUID;
		int size = CUSIZE;
		while ( i < n){
			int value = list1[i];
			int left = 0,right = m-1;
			mark[i] = 0;
			while (left <= right){
				int mid = (left+right)/2;
				int difference = value-list2[mid];
				if ( difference <0 )
					right = mid-1;
				else if ( difference >0 )
					left = mid +1;
				else {
					mark[i] = 1;
					break;
				}
			}
			i += size;
		}
	}

	__global__ void bsearch_copy_result(int * list1,int *mark, int *result,int n){
		int id = CUID;
		int size = CUSIZE;
		while ( id < n ){
			if (mark[id] - mark[id-1] >0){
				result[ mark[id-1] ] = list1[ id ];
			}
			id += size;
		}
	}

	void work3(){
		init_data();
		hipStream_t oneStream;
		hipStreamCreate(&oneStream);
		//outln(n);
		CUDPPHandle prefixsum_plan = prepare_prefixsum(n+1,&oneStream);
		CudaWatch cudawatch,w2;cudawatch.start();
		int * markFrom = devMark+1;

		w2.start();
		algo_bsearch<<<32,512,0,oneStream>>>(devL1,devL2,markFrom,n,m);
		cout<<"naive bsearch: part1 :"<<w2.stop()/1000.0<<endl;

		cudppScan(prefixsum_plan,devMark,devMark,n+1);
		bsearch_copy_result<<<32,512,0,oneStream>>>(devL1,markFrom,devResult,n+1);
		cout<<"NAIVE Bsearch:"<<cudawatch.stop()/1000.0<<endl;
		hipMemcpy(&Lresult,devMark+n,sizeof(int),D_T_H);cu_checkError();
		hipMemcpy(resultList,devResult,sizeof(int)*Lresult,D_T_H);cu_checkError();
		free_device_memory();
	}

int main(){
	int r;
	FOR_I(103,10000){
	r = rand() % 10013123 ;
	//r =1344532745 ;
	srand(i);
	n = 1024*1024*40;
	//n = 1024*1024*5;
	//n = 50;

	//generate_case5();
	generate_random(1.0,2.0,2.0);

	cout<<"generate data over srand("<<i<<") n="<<n<<" m="<<m<<endl;
	printf("List 1 ( %d --- %d --- %d )\n",list[0][0],list[0][n/2],list[0][n-1]);
	printf("List 2 ( %d --- %d --- %d )\n",list[1][0],list[1][m/2],list[1][m-1]);

//		debug_a(list[0],n);outline;debug_a(list[1],m);

	memset(V,0,sizeof(V));

	Watch watch;watch.start();
	merge_algo(list[0],list[1],0,n,0,m);
	cout<<"CPU ALGO time: "<<watch.stop()<<endl;

//	work3();//test bsearch
//	return 0;


	work();
	cuda_copyResult();cout<<"copied back"<<endl;
	cu_checkError();
	free_device_memory();
	printf(" results / elements = %d / %d %lf\n",Lresult,n,(1.0*Lresult)/(1.0*n));
	outln(Lresult);outln(cpuResultSize);


	//outln(Lresult);outln(cpuResultSize);
	//debug_a(resultList,Lresult);
	//debug_a(cpuResult,cpuResultSize);
	//break;

	//continue;// DO NOT CHECK correctness

	int error_index;
	if  ( (error_index = check_correctness()) >=0 ){
		outline;
		outln(i);
		printf("all length: cpu:%d gpu:%d  ration: %.5lf\n",cpuResultSize,Lresult,100.0*i/(1.0*Lresult));
		break;
	}
		outline;
		outline;
		outline;
		outline;

		break; //only execute once
	}//end FOR
	return 0;
}
